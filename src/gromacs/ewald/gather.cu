#include "hip/hip_runtime.h"
#include "pme.h"

#include "gromacs/utility/basedefinitions.h"
#include "gromacs/utility/real.h"
#include "gromacs/math/vectypes.h"
#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/gpu_utils/cuda_arch_utils.cuh"
#include <hip/hip_runtime.h>

#include "th-a.cuh"
#include "check.h"

#include "pme-internal.h"
#include "pme-cuda.h"

typedef real *splinevec[DIM];
#ifdef DEBUG_PME_GPU
extern gpu_flags gather_gpu_flags;
#endif
#ifdef DEBUG_PME_TIMINGS_GPU
extern gpu_events gpu_events_gather;
#endif

#define DO_FSPLINE(order)                      \
    for (int ithx = 0; (ithx < order); ithx++)              \
    {                                              \
        int index_x = (i0[i]+ithx)*pny*pnz;               \
        real tx      = thx[iorder+ithx];                       \
        real dx      = dthx[iorder+ithx];                      \
                                               \
        for (int ithy = 0; (ithy < order); ithy++)          \
        {                                          \
            int index_xy = index_x+(j0[i]+ithy)*pnz;      \
            real ty       = thy[iorder+ithy];                  \
            real dy       = dthy[iorder+ithy];                 \
            real fxy1     = 0, fz1 = 0;		   \
                                               \
            for (int ithz = 0; (ithz < order); ithz++)      \
            {                                      \
                /*printf(" INDEX %d %d %d\n", (i0[i] + ithx), (j0[i]+ithy), (k0[i]+ithz));*/\
                real gval  = grid[index_xy+(k0[i]+ithz)];  \
                fxy1 += thz[iorder+ithz]*gval;            \
                fz1  += dthz[iorder+ithz]*gval;           \
            }                                      \
            fx += dx*ty*fxy1;                      \
            fy += tx*dy*fxy1;                      \
            fz += tx*ty*fz1;                       \
        }                                          \
    }


static __global__ void gather_f_bsplines_kernel
(real *grid, int order, int n,
 int nx, int ny, int nz, int pnx, int pny, int pnz,
 real rxx, real ryx, real ryy, real rzx, real rzy, real rzz,
 real *thx, real *thy, real *thz, real *dthx, real *dthy, real *dthz,
 real *atc_f, real *coefficient_v, int *i0, int *j0, int *k0)
{
  /* sum forces for local particles */
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n)
  {
    real coefficient = coefficient_v[i];
    real fx     = 0;
    real fy     = 0;
    real fz     = 0;
    int iorder = i*order;
    int idim = i * DIM;

    switch (order)
    {
    case 4:
      DO_FSPLINE(4);
      break;
    case 5:
      DO_FSPLINE(5);
      break;
    default:
      DO_FSPLINE(order);
      break;
    }


    atc_f[idim + XX] += -coefficient*( fx*nx*rxx );
    atc_f[idim + YY] += -coefficient*( fx*nx*ryx + fy*ny*ryy );
    atc_f[idim + ZZ] += -coefficient*( fx*nx*rzx + fy*ny*rzy + fz*nz*rzz );

    /*printf("kernel coeff=%f f=%f,%f,%f\n",
	   (double) coefficient,
	   (double) fx, (double) fy, (double) fz);*/

    /* Since the energy and not forces are interpolated
     * the net force might not be exactly zero.
     * This can be solved by also interpolating F, but
     * that comes at a cost.
     * A better hack is to remove the net force every
     * step, but that must be done at a higher level
     * since this routine doesn't see all atoms if running
     * in parallel. Don't know how important it is?  EL 990726
     */
  }
}

void gather_f_bsplines_gpu_2_pre
(gmx_bool bClearF,
 int *spline_ind, int spline_n,
 real *atc_coefficient, rvec *atc_f,
 real scale, int thread
 )
{
    // compact atc_f before cpu calcucation

    int size_forces = DIM * spline_n * sizeof(real);
    real *atc_f_compacted = th_a(TH_ID_F, thread, size_forces, TH_LOC_HOST); //yupinov fixed allocation size - not actually compacted, same for i_compacted
    int size_indices = spline_n * sizeof(int);
    int *atc_i_compacted = th_i(TH_ID_I, thread, size_indices, TH_LOC_HOST);

    int oo = 0;
    for (int ii = 0; ii < spline_n; ii++)
    {
        int i           = spline_ind[ii];
        real coefficient_i = scale*atc_coefficient[i];
        if (bClearF)
        {
            atc_f[i][XX] = 0;
            atc_f[i][YY] = 0;
            atc_f[i][ZZ] = 0;
        }

        if (coefficient_i != 0.0)
        {
            atc_f_compacted[oo * DIM + XX] = atc_f[i][XX];
            atc_f_compacted[oo * DIM + YY] = atc_f[i][YY];
            atc_f_compacted[oo * DIM + ZZ] = atc_f[i][ZZ];
            atc_i_compacted[oo] = i;  // indices of uncompacted particles stored in a compacted array
            oo++;
        }
    }
    //oo is a real size of compacted stuff now
}

void gather_f_bsplines_gpu_2
(real *grid, gmx_bool bClearF,
 int order,
 int nx, int ny, int nz, int pnx, int pny, int pnz,
 real rxx, real ryx, real ryy, real rzx, real rzy, real rzz,
 int *spline_ind, int spline_n,
 real *atc_coefficient, rvec *atc_f, ivec *atc_idx,
 splinevec *spline_theta, splinevec *spline_dtheta,
 real scale,
 gmx_pme_t *pme,
 int thread
 )
{
    hipStream_t s = pme->gpu->pmeStream;
    int ndatatot = pnx*pny*pnz;

    if (!spline_n)
        return;

    int size_grid = ndatatot * sizeof(real);
    real *grid_d = th_a_cpy(TH_ID_GRID, thread, grid, size_grid, TH_LOC_CUDA, s);

    //copy order?
    //compacting, and size....
    int n = spline_n;
    int size_indices = n * sizeof(int);
    int size_coefficients = n * sizeof(real);
    int size_forces = DIM * n * sizeof(real);
    int size_splines = order * n * sizeof(int);


    real *atc_f_compacted = th_a(TH_ID_F, thread, -1, TH_LOC_HOST); //but that's wrong! realloc

    int *atc_i_compacted = th_i(TH_ID_I, thread, -1, TH_LOC_HOST);  //way to get sizes from th-a?
    real *coefficients_compacted = th_a(TH_ID_COEFFICIENT, thread, size_coefficients, TH_LOC_HOST);
    //yupinov reuse H_ID_COEFFICIENT and other stuff from before solve?

    int *i0_compacted = th_i(TH_ID_I0, thread, size_indices, TH_LOC_HOST); //yupinov these are IDXPTR, actually. maybe split it?
    int *j0_compacted = th_i(TH_ID_J0, thread, size_indices, TH_LOC_HOST);
    int *k0_compacted = th_i(TH_ID_K0, thread, size_indices, TH_LOC_HOST);

    real *theta_x_compacted = th_a(TH_ID_THX, thread, size_splines, TH_LOC_HOST);
    real *theta_y_compacted = th_a(TH_ID_THY, thread, size_splines, TH_LOC_HOST);
    real *theta_z_compacted = th_a(TH_ID_THZ, thread, size_splines, TH_LOC_HOST);
    real *dtheta_x_compacted = th_a(TH_ID_DTHX, thread, size_splines, TH_LOC_HOST);
    real *dtheta_y_compacted = th_a(TH_ID_DTHY, thread, size_splines, TH_LOC_HOST);
    real *dtheta_z_compacted = th_a(TH_ID_DTHZ, thread, size_splines, TH_LOC_HOST);

    int oo = 0;
    for (int ii = 0; ii < spline_n; ii++)
    {
        int i           = spline_ind[ii];
        real coefficient_i = scale*atc_coefficient[i];
        if (bClearF)
        {
            atc_f[i][XX] = 0; //yupinov memeset?
            atc_f[i][YY] = 0;
            atc_f[i][ZZ] = 0;
        }

        if (coefficient_i != 0)
        {
            coefficients_compacted[oo] = coefficient_i;
            int *idxptr = atc_idx[i];
            //Mattias: atc_f_h force-copying is in gather_f_bsplines_gpu_2_pre()
            //yupinov: the fuck is it doing there?
            atc_i_compacted[oo] = i;
            i0_compacted[oo] = idxptr[XX];
            j0_compacted[oo] = idxptr[YY];
            k0_compacted[oo] = idxptr[ZZ];
            int iiorder = ii*order;
            int ooorder = oo*order;
            for (int o = 0; o < order; ++o)
            {
                theta_x_compacted[ooorder + o] = (*spline_theta)[XX][iiorder + o];
                theta_y_compacted[ooorder + o] = (*spline_theta)[YY][iiorder + o];
                theta_z_compacted[ooorder + o] = (*spline_theta)[ZZ][iiorder + o];
                dtheta_x_compacted[ooorder + o] = (*spline_dtheta)[XX][iiorder + o];
                dtheta_y_compacted[ooorder + o] = (*spline_dtheta)[YY][iiorder + o];
                dtheta_z_compacted[ooorder + o] = (*spline_dtheta)[ZZ][iiorder + o];
            }
            ++oo;
        }
    }

    n = oo;
    if (!n)
        return;

    //copypasted
    size_indices = n * sizeof(int);
    size_coefficients = n * sizeof(real);
    size_forces = DIM * n * sizeof(real);
    size_splines = order * n * sizeof(int);

    real *atc_f_d = th_a_cpy(TH_ID_F, thread, atc_f_compacted, size_forces, TH_LOC_CUDA, s);
    real *coefficients_d = th_a_cpy(TH_ID_COEFFICIENT, thread, coefficients_compacted, size_coefficients, TH_LOC_CUDA, s);

    int *i0_d = th_i_cpy(TH_ID_I0, thread, i0_compacted, size_indices, TH_LOC_CUDA, s);
    int *j0_d = th_i_cpy(TH_ID_J0, thread, j0_compacted, size_indices, TH_LOC_CUDA, s);
    int *k0_d = th_i_cpy(TH_ID_K0, thread, k0_compacted, size_indices, TH_LOC_CUDA, s);

    real *theta_x_d = th_a_cpy(TH_ID_THX, thread, theta_x_compacted, size_splines, TH_LOC_CUDA, s);
    real *theta_y_d = th_a_cpy(TH_ID_THY, thread, theta_y_compacted, size_splines, TH_LOC_CUDA, s);
    real *theta_z_d = th_a_cpy(TH_ID_THZ, thread, theta_z_compacted, size_splines, TH_LOC_CUDA, s);
    real *dtheta_x_d = th_a_cpy(TH_ID_DTHX, thread, dtheta_x_compacted, size_splines, TH_LOC_CUDA, s);
    real *dtheta_y_d = th_a_cpy(TH_ID_DTHY, thread, dtheta_y_compacted, size_splines, TH_LOC_CUDA, s);
    real *dtheta_z_d = th_a_cpy(TH_ID_DTHZ, thread, dtheta_z_compacted, size_splines, TH_LOC_CUDA, s);

    int block_size = 2 * warp_size;
    int n_blocks = (n + block_size - 1) / block_size;
#ifdef DEBUG_PME_TIMINGS_GPU
    events_record_start(gpu_events_gather, s);
#endif
    gather_f_bsplines_kernel<<<n_blocks, block_size, 0, s>>>
      (grid_d,
       order, n,
       nx, ny, nz, pnx, pny, pnz,
       rxx, ryx, ryy, rzx, rzy, rzz,
       theta_x_d, theta_y_d, theta_z_d,
       dtheta_x_d, dtheta_y_d, dtheta_z_d,
       atc_f_d, coefficients_d,
       i0_d, j0_d, k0_d);
    CU_LAUNCH_ERR("gather_f_bsplines_kernel");
#ifdef DEBUG_PME_TIMINGS_GPU
    events_record_stop(gpu_events_gather, s, ewcsPME_GATHER, 0);
#endif

    th_cpy(atc_f_compacted, atc_f_d, size_forces, TH_LOC_HOST, s);

    for (int ii = 0; ii < n; ii++)  // iterating over compacted particles
    {
        int i = atc_i_compacted[ii]; //index of uncompacted particle
        atc_f[i][XX] = atc_f_compacted[ii * DIM + XX];
        atc_f[i][YY] = atc_f_compacted[ii * DIM + YY];
        atc_f[i][ZZ] = atc_f_compacted[ii * DIM + ZZ];
    }
}
