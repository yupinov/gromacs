#include "hip/hip_runtime.h"
/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 2016, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */

/*! \internal \file
 *  \brief Implements common PME GPU routines in CUDA.
 *
 *  \author Aleksei Iupinov <a.yupinov@gmail.com>
 */

#include "gmxpre.h"

/* GPU initialization includes */
#include "gromacs/gpu_utils/gpu_utils.h"
#include "gromacs/utility/cstringutil.h"
#include "gromacs/hardware/hw_info.h"
#include "gromacs/utility/logger.h"

/* The rest */
#include <assert.h>

#include "gromacs/gpu_utils/pmalloc_cuda.h"
#include "gromacs/math/units.h"
#include "gromacs/utility/smalloc.h"
#include "pme.cuh"
#include "pme-gpu.h"

/*! \brief \internal
 *
 * Allocates the energy and virial memory both on GPU and CPU (7 floats).
 *
 * \param[in] pme            The PME structure.
 */
void pme_gpu_alloc_energy_virial(const gmx_pme_t *pme)
{
    pme->gpu->energyAndVirialSizeBytes = 7 * sizeof(float); /* 6 virial components + energy */
    hipError_t stat = hipMalloc((void **)&pme->gpu->energyAndVirial, pme->gpu->energyAndVirialSizeBytes);
    CU_RET_ERR(stat, "hipMalloc failed on PME energy and virial");
    pmalloc((void **)&pme->gpu->energyAndVirialHost, pme->gpu->energyAndVirialSizeBytes);
}

/*! \brief \internal
 * Frees the energy and virial memory both on GPU and CPU (7 floats).
 *
 * \param[in] pme            The PME structure.
 */
void pme_gpu_free_energy_virial(const gmx_pme_t *pme)
{
    hipError_t stat = hipFree(pme->gpu->energyAndVirial);
    CU_RET_ERR(stat, "hipFree failed on PME energy and virial");
    pme->gpu->energyAndVirial = NULL;
    pfree(pme->gpu->energyAndVirialHost);
    pme->gpu->energyAndVirialHost = NULL;
}

/*! \brief
 *
 * Clears the energy and virial memory on GPU with 0.
 * Should be called at the end of the energy/virial calculation step.
 */
void pme_gpu_clear_energy_virial(const gmx_pme_t *pme)
{
    hipError_t stat = hipMemsetAsync(pme->gpu->energyAndVirial, 0, pme->gpu->energyAndVirialSizeBytes, pme->gpu->pmeStream);
    CU_RET_ERR(stat, "PME energies/virial hipMemsetAsync error");
}

/*! \brief \internal
 * Copies the precalculated reciprocal box to the GPU constants structure.
 *
 * \param[in] pme            The PME structure.
 */
void pme_gpu_copy_recipbox(const gmx_pme_t *pme)
{
    const float3 box[3] =
    {
        {pme->recipbox[XX][XX], pme->recipbox[YY][XX], pme->recipbox[ZZ][XX]},
        {                  0.0, pme->recipbox[YY][YY], pme->recipbox[ZZ][YY]},
        {                  0.0,                   0.0, pme->recipbox[ZZ][ZZ]}
    };
    assert(pme->recipbox[XX][XX] != 0.0f);
    memcpy(pme->gpu->kernelParams.step.recipbox, box, sizeof(box));
}

/*! \brief \internal
 *
 * Reallocates and copies the pre-computed B-spline values to the GPU.
 * FIXME: currently uses just a global memory, could be using texture memory/ldg.
 *
 * \param[in] pme            The PME structure.
 */
void pme_gpu_realloc_and_copy_bspline_values(const gmx_pme_t *pme)
{
    const int splineValuesOffset[DIM] = {0, pme->nkx, pme->nkx + pme->nky}; //?replace nkx
    memcpy(&pme->gpu->kernelParams.grid.splineValuesOffset, &splineValuesOffset, sizeof(splineValuesOffset));

    const int newSplineValuesSize  = pme->nkx + pme->nky + pme->nkz;
    cu_realloc_buffered((void **)&pme->gpu->kernelParams.grid.splineValuesArray, NULL, sizeof(float),
                        &pme->gpu->splineValuesSize, &pme->gpu->splineValuesSizeAlloc, newSplineValuesSize, pme->gpu->pmeStream, true);

    for (int i = 0; i < DIM; i++)
    {
        size_t       gridSize;
        switch (i)
        {
            case XX:
                gridSize = pme->nkx;
                break;

            case YY:
                gridSize = pme->nky;
                break;

            case ZZ:
                gridSize = pme->nkz;
                break;
        }
        size_t  modSize  = gridSize * sizeof(float);
        /* reallocate the host buffer */
        if ((pme->gpu->splineValuesHost[i] == NULL) || (pme->gpu->splineValuesHostSizes[i] < modSize))
        {
            pfree(pme->gpu->splineValuesHost[i]);
            pmalloc((void **)&pme->gpu->splineValuesHost[i], modSize);
        }
        memcpy(pme->gpu->splineValuesHost[i], pme->bsp_mod[i], modSize);
        //yupinov instead use pinning here as well!
        cu_copy_H2D_async(pme->gpu->kernelParams.grid.splineValuesArray + splineValuesOffset[i], pme->gpu->splineValuesHost[i], modSize, pme->gpu->pmeStream);
    }
}

/*! \brief \internal
 * Frees the pre-computed B-spline values on the GPU (and the transfer CPU buffers).
 *
 * \param[in] pme            The PME structure.
 */
void pme_gpu_free_bspline_values(const gmx_pme_t *pme)
{
    for (int i = 0; i < DIM; i++)
    {
        pfree(pme->gpu->splineValuesHost[i]);
    }
    cu_free_buffered(pme->gpu->kernelParams.grid.splineValuesArray, &pme->gpu->splineValuesSize, &pme->gpu->splineValuesSizeAlloc);
}

/*! \brief \internal
 * Copies the grid sizes for overlapping (used in the PME wrap/unwrap).
 *
 * \param[in] pme            The PME structure.
 */
void pme_gpu_copy_wrap_zones(const gmx_pme_t *pme)
{
    const int nx      = pme->gpu->kernelParams.grid.localGridSize.x;
    const int ny      = pme->gpu->kernelParams.grid.localGridSize.y;
    const int nz      = pme->gpu->kernelParams.grid.localGridSize.z;
    const int overlap = pme->pme_order - 1;

    /* Cell counts in the 7 overlapped grid parts */
    /* Is this correct? No Z alignment changes? */
    const int3 zoneSizes_h[OVERLAP_ZONES] =
    {
        {     nx,        ny,   overlap},
        {     nx,   overlap,        nz},
        {overlap,        ny,        nz},
        {     nx,   overlap,   overlap},
        {overlap,        ny,   overlap},
        {overlap,   overlap,        nz},
        {overlap,   overlap,   overlap}
    };
    /* The X is never used on the GPU, actually */
    int2 zoneSizesYZ_h[OVERLAP_ZONES];
    for (int i = 0; i < OVERLAP_ZONES; i++)
    {
        zoneSizesYZ_h[i].x = zoneSizes_h[i].y;
        zoneSizesYZ_h[i].y = zoneSizes_h[i].z;
    }
    int cellsAccumCount_h[OVERLAP_ZONES];
    for (int i = 0; i < OVERLAP_ZONES; i++)
    {
        cellsAccumCount_h[i] = zoneSizes_h[i].x * zoneSizes_h[i].y * zoneSizes_h[i].z;
    }
    /* Accumulation */
    for (int i = 1; i < OVERLAP_ZONES; i++)
    {
        cellsAccumCount_h[i] = cellsAccumCount_h[i] + cellsAccumCount_h[i - 1];
    }
    memcpy(pme->gpu->kernelParams.grid.overlapSizes, zoneSizesYZ_h, sizeof(zoneSizesYZ_h));
    memcpy(pme->gpu->kernelParams.grid.overlapCellCounts, cellsAccumCount_h, sizeof(cellsAccumCount_h));
}

/*! \brief
 * Reallocates the GPU buffer for the resulting PME forces.
 *
 *
 */
void pme_gpu_realloc_forces(const gmx_pme_t *pme)
{
    const int newForcesSize = pme->gpu->kernelParams.atoms.nAtoms * DIM;
    assert(pme->gpu->kernelParams.atoms.nAtoms > 0);
    cu_realloc_buffered((void **)&pme->gpu->kernelParams.atoms.forces, NULL, sizeof(float),
                        &pme->gpu->forcesSize, &pme->gpu->forcesSizeAlloc, newForcesSize, pme->gpu->pmeStream, true);
}

void pme_gpu_free_forces(const gmx_pme_t *pme)
{
    cu_free_buffered(pme->gpu->kernelParams.atoms.forces, &pme->gpu->forcesSize, &pme->gpu->forcesSizeAlloc);
}

/*! \brief
 * Reallocates the buffer on the GPU and copies the coordinates from the CPU buffer (pme->gpu->coordinatesHost).
 *
 * \param[in] pme            The PME structure.
 *
 * Needs to be called every MD step. The coordinates are then used in the spline calculation.
 * Should probably be split into realloc (on DD) and copy (every step) parts...
 */
void pme_gpu_realloc_and_copy_coordinates(const gmx_pme_t *pme)
{
    assert(pme->gpu->kernelParams.atoms.nAtoms > 0);
    assert(pme->gpu->coordinatesHost);
    const size_t newCoordinatesSize = pme->gpu->kernelParams.atoms.nAtoms * DIM;
    cu_realloc_buffered((void **)&pme->gpu->kernelParams.atoms.coordinates, pme->gpu->coordinatesHost, sizeof(float),
                        &pme->gpu->coordinatesSize, &pme->gpu->coordinatesSizeAlloc, newCoordinatesSize, pme->gpu->pmeStream, true);
}

/*! \brief
 * Frees the coordinates on the GPU.
 *
 * \param[in] pme            The PME structure.
 */
void pme_gpu_free_coordinates(const gmx_pme_t *pme)
{
    cu_free_buffered((void **)&pme->gpu->kernelParams.atoms.coordinates, &pme->gpu->coordinatesSize, &pme->gpu->coordinatesSizeAlloc);
}

/*! \brief
 * Reallocates the buffer on the GPU and copies the charges (sometimes also called coefficients) from the CPU buffer (pme->gpu->coefficientsHost).
 *
 * \param[in] pme            The PME structure.
 *
 * Does not need to be done every MD step, only whenever the local charges change.
 * (So, in the beginning of the run, or on DD step).
 */
void pme_gpu_realloc_and_copy_charges(const gmx_pme_t *pme)
{
    assert(pme->gpu->kernelParams.atoms.nAtoms > 0);
    assert(pme->gpu->coefficientsHost);
    const size_t newCoefficientSize = pme->gpu->kernelParams.atoms.nAtoms;
    cu_realloc_buffered((void **)&pme->gpu->kernelParams.atoms.coefficients, pme->gpu->coefficientsHost, sizeof(float),
                        &pme->gpu->coefficientsSize, &pme->gpu->coefficientsSizeAlloc, newCoefficientSize, pme->gpu->pmeStream, true);
}

/*! \brief
 * Frees the charges on the GPU.
 *
 * \param[in] pme            The PME structure.
 */
void pme_gpu_free_charges(const gmx_pme_t *pme)
{
    cu_free_buffered((void **)&pme->gpu->kernelParams.atoms.coefficients, &pme->gpu->coefficientsSize, &pme->gpu->coefficientsSizeAlloc);
}

/*! \brief
 * Reallocates the buffers on the GPU for the particle spline data.
 *
 * \param[in] pme            The PME structure.
 */
void pme_gpu_realloc_spline_data(const gmx_pme_t *pme)
{
    const int    order     = pme->pme_order;
    const int    alignment = PME_SPREADGATHER_PARTICLES_PER_WARP;
    /* Probably needs to be particlesPerBlock for full padding */
    const size_t nAtomsPadded      = ((pme->gpu->kernelParams.atoms.nAtoms + alignment - 1) / alignment) * alignment;
    const size_t newSplineDataSize = DIM * order * nAtomsPadded;
    assert(newSplineDataSize > 0);

    /* Two arrays of the same size */
    int currentSizeTemp      = pme->gpu->splineDataSize;
    int currentSizeTempAlloc = pme->gpu->splineDataSizeAlloc;
    cu_realloc_buffered((void **)&pme->gpu->kernelParams.atoms.theta, NULL, sizeof(float),
                        &currentSizeTemp, &currentSizeTempAlloc, newSplineDataSize, pme->gpu->pmeStream, true);
    cu_realloc_buffered((void **)&pme->gpu->kernelParams.atoms.dtheta, NULL, sizeof(float),
                        &pme->gpu->splineDataSize, &pme->gpu->splineDataSizeAlloc, newSplineDataSize, pme->gpu->pmeStream, true);
}

/*! \brief
 * Frees the buffers on the GPU for the particle spline data.
 *
 * \param[in] pme            The PME structure.
 */
void pme_gpu_free_spline_data(const gmx_pme_t *pme)
{
    /* Two arrays of the same size */
    int currentSizeTemp      = pme->gpu->splineDataSize;
    int currentSizeTempAlloc = pme->gpu->splineDataSizeAlloc;
    cu_free_buffered((void **)&pme->gpu->kernelParams.atoms.theta, &currentSizeTemp, &currentSizeTempAlloc);
    cu_free_buffered((void **)&pme->gpu->kernelParams.atoms.dtheta, &pme->gpu->splineDataSize, &pme->gpu->splineDataSizeAlloc);
}

/*! \brief \internal
 * Reallocates the buffer on the GPU for the particle gridline indices.
 *
 * \param[in] pme            The PME structure.
 */
void pme_gpu_realloc_grid_indices(const gmx_pme_t *pme)
{
    const size_t newIndicesSize = DIM * pme->gpu->kernelParams.atoms.nAtoms;
    assert(newIndicesSize > 0);
    cu_realloc_buffered((void **)&pme->gpu->kernelParams.atoms.gridlineIndices, NULL, sizeof(int),
                        &pme->gpu->gridlineIndicesSize, &pme->gpu->gridlineIndicesSizeAlloc, newIndicesSize, pme->gpu->pmeStream, true);
}

/*! \brief
 * Frees the buffer on the GPU for the particle gridline indices.
 *
 * \param[in] pme            The PME structure.
 */
void pme_gpu_free_grid_indices(const gmx_pme_t *pme)
{
    cu_free_buffered((void **)&pme->gpu->kernelParams.atoms.gridlineIndices, &pme->gpu->gridlineIndicesSize, &pme->gpu->gridlineIndicesSizeAlloc);
}

void pme_gpu_realloc_grids(const gmx_pme_t *pme)
{
    const int pnx         = pme->pmegrid_nx; //?
    const int pny         = pme->pmegrid_ny;
    const int pnz         = pme->pmegrid_nz;
    const int newGridSize = pnx * pny * pnz;

    if (pme->gpu->bOutOfPlaceFFT)
    {
        /* Allocate a separate complex grid */
        int tempGridSize      = pme->gpu->gridSize;
        int tempGridSizeAlloc = pme->gpu->gridSizeAlloc;
        cu_realloc_buffered((void **)&pme->gpu->kernelParams.grid.fourierGrid, NULL, sizeof(float),
                            &tempGridSize, &tempGridSizeAlloc, newGridSize, pme->gpu->pmeStream, true);
    }
    cu_realloc_buffered((void **)&pme->gpu->kernelParams.grid.realGrid, NULL, sizeof(float),
                        &pme->gpu->gridSize, &pme->gpu->gridSizeAlloc, newGridSize, pme->gpu->pmeStream, true);
    if (!pme->gpu->bOutOfPlaceFFT)
    {
        /* Using the same grid */
        pme->gpu->kernelParams.grid.fourierGrid = (float2 *)(pme->gpu->kernelParams.grid.realGrid);
    }
}

void pme_gpu_free_grids(const gmx_pme_t *pme)
{
    if (pme->gpu->bOutOfPlaceFFT)
    {
        /* Free a separate complex grid of the same size */
        int tempGridSize      = pme->gpu->gridSize;
        int tempGridSizeAlloc = pme->gpu->gridSizeAlloc;
        cu_free_buffered((void **)&pme->gpu->kernelParams.grid.fourierGrid, &tempGridSize, &tempGridSizeAlloc);
    }
    cu_free_buffered((void **)&pme->gpu->kernelParams.grid.realGrid, &pme->gpu->gridSize, &pme->gpu->gridSizeAlloc);
}

void pme_gpu_clear_grids(const gmx_pme_t *pme)
{
    hipStream_t s = pme->gpu->pmeStream;

    hipError_t  stat = hipMemsetAsync(pme->gpu->kernelParams.grid.realGrid, 0, pme->gpu->gridSize * sizeof(float), s);
    /* Should the complex grid be cleared in some weird case? */
    CU_RET_ERR(stat, "hipMemsetAsync on the PME grid error");
}

/*! \brief
 * The PME GPU reinitialization function that is called both at the end of any MD step and on any load balancing step.
 *
 * \param[in] pme            The PME structure.
 */
void pme_gpu_step_reinit(const gmx_pme_t *pme)
{
    pme_gpu_clear_grids(pme);
    pme_gpu_clear_energy_virial(pme);
}

/*! \brief
 * The PME GPU initialization function that is called in the beginning of the run and on any load balancing step.
 *
 * \param[in] pme            The PME structure.
 * ......
 */
void pme_gpu_init(gmx_pme_t *pme, const gmx_hw_info_t *hwinfo, const gmx_gpu_opt_t *gpu_opt)
{
    if (!pme_gpu_enabled(pme))
    {
        return;
    }

    const gmx_bool firstInit = !pme->gpu;
    if (firstInit)
    {
        snew(pme->gpu, 1);
        hipError_t stat;

        /* GPU selection copied from non-bondeds */
        const int PMEGPURank = pme->nodeid;
        char      gpu_err_str[STRLEN];
        assert(hwinfo);
        assert(hwinfo->gpu_info.gpu_dev);
        assert(gpu_opt->dev_use);

        int   forcedGPUId       = -1;
        char *forcedGPUIdString = getenv("GMX_PME_GPU_ID");
        if (forcedGPUIdString)
        {
            forcedGPUId = atoi(forcedGPUIdString);
            printf("PME rank %d trying to use GPU %d\n", PMEGPURank, forcedGPUId);
            stat = hipSetDevice(forcedGPUId);
            CU_RET_ERR(stat, "PME failed to set the GPU device");
        }
        else
        {
            pme->gpu->deviceInfo = &hwinfo->gpu_info.gpu_dev[gpu_opt->dev_use[PMEGPURank]];
            const gmx::MDLogger temp;
            if (!init_gpu(temp, PMEGPURank, gpu_err_str, &hwinfo->gpu_info, gpu_opt))
            {
                gmx_fatal(FARGS, "Could not select GPU %d for PME rank %d\n", pme->gpu->deviceInfo->id, PMEGPURank);
            }
        }

        /* Some permanent settings are set here */

        pme->gpu->bGPUSingle = pme_gpu_enabled(pme) && (pme->nnodes == 1);
        /* A convenience variable. */

        pme->gpu->bGPUFFT = pme->gpu->bGPUSingle && !getenv("GMX_PME_GPU_FFTW");
        /* cuFFT will only used for a single rank. */

        pme->gpu->bGPUSolve = true;
        /* pme->gpu->bGPUFFT - CPU solve with the CPU FFTW is definitely broken at the moment - 20160511 */

        pme->gpu->bGPUGather = true;
        /* CPU gather has got to be broken as well due to different theta/dtheta layout. */

        pme->gpu->bOutOfPlaceFFT = true;
        /* This should give better performance, according to the cuFFT documentation.
         * The performance seems to be the same though.
         * Perhaps the limiting factor is using paddings/overlaps in the grid, which is also frowned upon.
         * PME could also try to pick up nice grid sizes (with factors of 2, 3, 5, 7)
         */

        pme->gpu->bTiming = (getenv("GMX_DISABLE_CUDA_TIMING") == NULL); /* This should also check for NB GPU being launched, and NB should check for PME GPU! */

        //pme->gpu->bUseTextureObjects = (pme->gpu->deviceInfo->prop.major >= 3);
        //yupinov - have to fix the GPU id selection, forced GPUIdHack?

        /* Creating a PME CUDA stream */
#if GMX_CUDA_VERSION >= 5050
        int highest_priority;
        int lowest_priority;
        stat = hipDeviceGetStreamPriorityRange(&lowest_priority, &highest_priority);
        CU_RET_ERR(stat, "PME hipDeviceGetStreamPriorityRange failed");
        stat = hipStreamCreateWithPriority(&pme->gpu->pmeStream,
                                            hipStreamDefault, //hipStreamNonBlocking,
                                            highest_priority);

        CU_RET_ERR(stat, "hipStreamCreateWithPriority on the PME stream failed");
#else
        stat = hipStreamCreate(&pme->gpu->pmeStream);
        CU_RET_ERR(stat, "PME hipStreamCreate error");
#endif

        /* Creating synchronization events */
        stat = hipEventCreateWithFlags(&pme->gpu->syncEnerVirD2H, hipEventDisableTiming);
        CU_RET_ERR(stat, "hipEventCreate on syncEnerVirH2D failed");
        stat = hipEventCreateWithFlags(&pme->gpu->syncForcesD2H, hipEventDisableTiming);
        CU_RET_ERR(stat, "hipEventCreate on syncForcesH2D failed");
        stat = hipEventCreateWithFlags(&pme->gpu->syncSpreadGridD2H, hipEventDisableTiming);
        CU_RET_ERR(stat, "hipEventCreate on syncSpreadGridH2D failed");
        stat = hipEventCreateWithFlags(&pme->gpu->syncSolveGridD2H, hipEventDisableTiming);
        CU_RET_ERR(stat, "hipEventCreate on syncSolveGridH2D failed");

        pme_gpu_init_timings(pme);

        pme_gpu_alloc_energy_virial(pme);

        GMX_RELEASE_ASSERT(pme->epsilon_r != 0.0f, "PME GPU: erroneous electostatic factor");
        pme->gpu->kernelParams.constants.elFactor = ONE_4PI_EPS0 / pme->epsilon_r;
    }

    const bool gridSizeChanged = true; /* This function is called on DLB steps as well */
    if (gridSizeChanged)               /* The need for reallocation is checked for inside, might do a redundant grid size increase check here anyway?... */
    {
        /* The grid size variants */
        const int3   localGridSize = {pme->nkx, pme->nky, pme->nkz};
        memcpy(&pme->gpu->kernelParams.grid.localGridSize, &localGridSize, sizeof(localGridSize));
        const float3 localGridSizeFP = {(float)localGridSize.x, (float)localGridSize.y, (float)localGridSize.z};
        memcpy(&pme->gpu->kernelParams.grid.localGridSizeFP, &localGridSizeFP, sizeof(localGridSizeFP));
        const int3   localGridSizePadded = {pme->pmegrid_nx, pme->pmegrid_ny, pme->pmegrid_nz};
        memcpy(&pme->gpu->kernelParams.grid.localGridSizePadded, &localGridSizePadded, sizeof(localGridSizePadded));

        pme_gpu_copy_wrap_zones(pme);
        pme_gpu_realloc_and_copy_fract_shifts(pme);
        pme_gpu_realloc_and_copy_bspline_values(pme);
        pme_gpu_realloc_grids(pme);

        if (pme->gpu->bGPUFFT)
        {
            snew(pme->gpu->pfft_setup_gpu, pme->ngrids); //yupinov - memory leaking?
            for (int i = 0; i < pme->ngrids; ++i)
            {
                gmx_parallel_3dfft_init_gpu(&pme->gpu->pfft_setup_gpu[i], (int *)&localGridSize, pme);
            }
        }
    }

    pme_gpu_step_reinit(pme);
}

void pme_gpu_deinit(gmx_pme_t *pme)
{
    if (!pme_gpu_enabled(pme))
    {
        return;
    }

    stopGpuProfiler();

    hipError_t stat;

    /* Free lots of dynamic data */
    pme_gpu_free_energy_virial(pme);
    pme_gpu_free_bspline_values(pme);
    pme_gpu_free_forces(pme);
    pme_gpu_free_coordinates(pme);
    pme_gpu_free_charges(pme);
    pme_gpu_free_spline_data(pme);
    pme_gpu_free_grid_indices(pme);
    pme_gpu_free_fract_shifts(pme);
    pme_gpu_free_grids(pme);

    /* cuFFT cleanup */
    if (pme->gpu->pfft_setup_gpu)
    {
        for (int i = 0; i < pme->ngrids; i++)
        {
            gmx_parallel_3dfft_destroy_gpu(pme->gpu->pfft_setup_gpu[i]);
        }
        sfree(pme->gpu->pfft_setup_gpu);
    }

    /* Free the synchronization events */
    stat = hipEventDestroy(pme->gpu->syncEnerVirD2H);
    CU_RET_ERR(stat, "hipEventDestroy failed on syncEnerVirH2D");
    stat = hipEventDestroy(pme->gpu->syncForcesD2H);
    CU_RET_ERR(stat, "hipEventDestroy failed on syncForcesH2D");
    stat = hipEventDestroy(pme->gpu->syncSpreadGridD2H);
    CU_RET_ERR(stat, "hipEventDestroy failed on syncpreadGridH2D");
    stat = hipEventDestroy(pme->gpu->syncSolveGridD2H);
    CU_RET_ERR(stat, "hipEventDestroy failed on syncSolveGridH2D");

    /* Free the timing events */
    pme_gpu_destroy_timings(pme);

    /* Destroy the CUDA stream */
    stat = hipStreamDestroy(pme->gpu->pmeStream);
    CU_RET_ERR(stat, "PME hipStreamDestroy error");

    /* Finally free the GPU structure itself */
    sfree(pme->gpu);
    pme->gpu = NULL;
}

void pme_gpu_set_constants(const gmx_pme_t *pme, const matrix box, const float ewaldCoeff)
{
    if (!pme_gpu_enabled(pme))
    {
        return;
    }

    /* Assuming the recipbox is calculated already */
    pme_gpu_copy_recipbox(pme); // could use some boolean checks to know if it should run each time, like pressure coupling?
    // actually, just compare the memory

    pme->gpu->kernelParams.step.boxVolume = box[XX][XX] * box[YY][YY] * box[ZZ][ZZ];
    assert(pme->gpu->kernelParams.step.boxVolume != 0.0f);

    pme->gpu->kernelParams.grid.ewaldFactor = (M_PI * M_PI) / (ewaldCoeff * ewaldCoeff);
}

void pme_gpu_set_io_ranges(const gmx_pme_t *pme, rvec *coordinates, rvec *forces)
{
    if (!pme_gpu_enabled(pme))
    {
        return;
    }

    pme->gpu->forcesHost       = reinterpret_cast<float *>(forces);
    pme->gpu->coordinatesHost  = reinterpret_cast<float *>(coordinates);
    /* TODO: should the hipHostRegister be called for the *Host pointers under some condition/policy? */
}

void pme_gpu_step_init(const gmx_pme_t *pme)
{
    if (!pme_gpu_enabled(pme))
    {
        return;
    }

    pme_gpu_realloc_and_copy_coordinates(pme);
}

void pme_gpu_reinit_atoms(const gmx_pme_t *pme, const int nAtoms, float *coefficients)
{
    if (!pme_gpu_enabled(pme))
    {
        return;
    }

    const gmx_bool haveToRealloc = (pme->gpu->kernelParams.atoms.nAtoms < nAtoms);
    pme->gpu->kernelParams.atoms.nAtoms = nAtoms;

    pme->gpu->coefficientsHost = reinterpret_cast<float *>(coefficients);
    pme_gpu_realloc_and_copy_charges(pme);

    if (haveToRealloc) /* This check might be redundant, but is logical */
    {
        pme_gpu_realloc_forces(pme);
        pme_gpu_realloc_spline_data(pme);
        pme_gpu_realloc_grid_indices(pme);
    }
}

void pme_gpu_step_end(const gmx_pme_t *pme, const gmx_bool bCalcF, const gmx_bool bCalcEnerVir)
{
    if (!pme_gpu_enabled(pme))
    {
        return;
    }

    hipError_t stat = hipStreamSynchronize(pme->gpu->pmeStream); /* Needed for copy back/timing events */
    CU_RET_ERR(stat, "Failed to synchronize the PME GPU stream!");

    if (bCalcF)
    {
        pme_gpu_sync_output_forces(pme);
    }
    if (bCalcEnerVir)
    {
        pme_gpu_sync_energy_virial(pme);
    }

    pme_gpu_update_timings(pme);

    pme_gpu_step_reinit(pme);
}

/* FIXME: this function does not actually seem to be used when it should be, with CPU FFT? */
void pme_gpu_sync_grid(const gmx_pme_t *pme, const gmx_fft_direction dir)
{
    if (!pme_gpu_enabled(pme))
    {
        return;
    }

    gmx_bool syncGPUGrid = ((dir == GMX_FFT_REAL_TO_COMPLEX) ? true : pme->gpu->bGPUSolve);
    if (syncGPUGrid)
    {
        hipError_t stat = hipStreamWaitEvent(pme->gpu->pmeStream,
                                               (dir == GMX_FFT_REAL_TO_COMPLEX) ? pme->gpu->syncSpreadGridD2H : pme->gpu->syncSolveGridD2H, 0);
        CU_RET_ERR(stat, "Error while waiting for the PME GPU grid to be copied to CPU");
    }
}

// TODO: use gmx_inline for small functions

// wrappers just for the pme.cpp host calls - a PME GPU code that should ideally be in this file as well
// C++11 not supported in CUDA host code by default => the code stays there for now

gmx_bool pme_gpu_performs_gather(const gmx_pme_t *pme)
{
    return pme_gpu_enabled(pme) && pme->gpu->bGPUGather;
}

gmx_bool pme_gpu_performs_FFT(const gmx_pme_t *pme)
{
    return pme_gpu_enabled(pme) && pme->gpu->bGPUFFT;
}

gmx_bool pme_gpu_performs_wrapping(const gmx_pme_t *pme)
{
    return pme_gpu_enabled(pme) && pme->gpu->bGPUSingle;
}

gmx_bool pme_gpu_performs_solve(const gmx_pme_t *pme)
{
    return pme_gpu_enabled(pme) && pme->gpu->bGPUSolve;
}
