#include "hip/hip_runtime.h"
#include "pme.h"
#include "pme-internal.h"

#include "gromacs/fft/fft.h"
#include "gromacs/fft/parallel_3dfft.h"

#include "gromacs/utility/basedefinitions.h"
#include "gromacs/utility/gmxmpi.h"
#include "gromacs/utility/real.h"
#include "gromacs/math/vectypes.h"

#include "check.h"

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/gpu_utils/cuda_arch_utils.cuh"

#ifdef DEBUG_PME_TIMINGS_GPU
extern gpu_events gpu_events_fft_r2c;
extern gpu_events gpu_events_fft_c2r;
#endif

#include "pme-internal.h" //yupinov pme passed everywhere
#include "pme-cuda.h"

#include "th-a.cuh"


struct gmx_parallel_3dfft_gpu
{
    real *real_data;
    t_complex *complex_data;

    /* unused */
    MPI_Comm                  comm[2];
    gmx_bool                  bReproducible;
    int                       nthreads;

    ivec                      complex_order;
    ivec                      local_ndata;
    ivec                      local_offset;
    ivec                      local_size;

    int n[3];
    hipfftHandle planR2C;
    hipfftHandle planC2R;
    hipfftReal *rdata;
    hipfftComplex *cdata;
};

void gmx_parallel_3dfft_init_gpu(gmx_parallel_3dfft_gpu_t *pfft_setup,
                                   ivec                      ndata,
                                   real **real_data,
                                   t_complex **complex_data,
                                   MPI_Comm                  comm[2],
gmx_bool                  bReproducible,
int                       nthreads,
gmx_pme_t *pme)
{
    gmx_parallel_3dfft_gpu_t setup = new gmx_parallel_3dfft_gpu();

    //yupinov FIXME: this copies the already setup pointer, to check them after execute

    setup->real_data = *real_data;

    setup->complex_data = *complex_data;

    setup->comm[0] = comm[0];
    setup->comm[1] = comm[1];
    setup->bReproducible = bReproducible;
    setup->nthreads = nthreads;

    /*
    // (local pme and fft differs only by overlap (and pme > fft))
    pmeidx = ix*(local_pme[YY]*local_pme[ZZ])+iy*(local_pme[ZZ])+iz;
    fftidx = ix*(local_fft[YY]*local_fft[ZZ])+iy*(local_fft[ZZ])+iz;
    fftgrid[fftidx] = pmegrid[pmeidx];
    // TODO: align cufft minor dim to 128 bytes
   */
    setup->n[0] = ndata[0];
    setup->n[1] = ndata[1];
    setup->n[2] = ndata[2]; //yupinov ZZ

    int x = setup->n[0], y = setup->n[1], z = setup->n[2];

    hipError_t stat = hipMalloc((void **) &setup->rdata, x * y * (z / 2 + 1) * 2 * sizeof(hipfftReal));
    CU_RET_ERR(stat, "fft init hipMalloc error");
    stat = hipMalloc((void **) &setup->cdata, x * y * (z / 2 + 1) * 2 * sizeof(hipfftComplex)); //yupinov: there are 2 complex planes here - for transposing
    CU_RET_ERR(stat, "fft init hipMalloc error"); //yupinov check all cuFFT errors

    *pfft_setup = setup;


    hipfftResult_t result;
    /*
    result = hipfftPlan3d(&setup->planR2C, setup->n[0], setup->n[1], setup->n[2], HIPFFT_R2C);
    if (result != HIPFFT_SUCCESS)
    {
        fprintf(stderr, "cufft planR2C error %d\n", result);
        setup = NULL; //yupinov FIX
    }

    result = hipfftPlan3d(&setup->planC2R, setup->n[0], setup->n[1], setup->n[2], HIPFFT_C2R);
    if (result != HIPFFT_SUCCESS)
    {
        fprintf(stderr, "cufft planC2R error %d\n", result);
        setup = NULL; // FIX
    }
    */

    int rembed[3];
    rembed[0] = setup->n[XX];
    rembed[1] = setup->n[YY];
    rembed[2] = setup->n[ZZ];
    rembed[2] = (rembed[2] / 2 + 1) * 2;
    int cembed[3];
    cembed[0] = setup->n[XX];
    cembed[1] = setup->n[YY];
    cembed[2] = setup->n[ZZ];
    cembed[2] = (cembed[2] / 2 + 1);

    int rank = 3, batch = 1;

    result = hipfftPlanMany(&setup->planR2C, rank, setup->n,
                                       rembed, 1, rembed[0] * rembed[1] * rembed[2],
                                       cembed, 1, cembed[0] * cembed[1] * cembed[2],
                                       HIPFFT_R2C,
                                      batch);
    if (result != HIPFFT_SUCCESS)
    {
        fprintf(stderr, "cufft planR2RC error %d\n", result); //yupinov throw fatalerror
        setup = NULL; // FIX
    }

    result = hipfftPlanMany(&setup->planC2R, rank, setup->n,
                                       cembed, 1, cembed[0] * cembed[1] * cembed[2],
                                       rembed, 1, rembed[0] * rembed[1] * rembed[2],
                                       HIPFFT_C2R,
                                       batch);
    if (result != HIPFFT_SUCCESS)
    {
        fprintf(stderr, "cufft planC2R error %d\n", result);
        setup = NULL; // FIX
    }

    hipStream_t s = pme->gpu->pmeStream;
    result = hipfftSetStream(setup->planR2C, s);
    if (result != HIPFFT_SUCCESS)
    {
        fprintf(stderr, "cufft planR2RC error %d\n", result);
        setup = NULL;
    }
    result = hipfftSetStream(setup->planC2R, s);
    if (result != HIPFFT_SUCCESS)
    {
        fprintf(stderr, "cufft planR2RC error %d\n", result);
        setup = NULL;
    }
}

void gmx_parallel_3dfft_real_limits_gpu(gmx_parallel_3dfft_gpu_t      pfft_setup,
                                       ivec                      local_ndata,
                                       ivec                      local_offset,
                                       ivec                      local_size)
{
    //fprintf(stderr, "3dfft_real_limits_gpu\n");
    gmx_parallel_3dfft_gpu_t setup = pfft_setup;
    setup->local_ndata[0] = local_ndata[0];
    setup->local_ndata[1] = local_ndata[1];
    setup->local_ndata[2] = local_ndata[2];
    setup->local_offset[0] = local_offset[0];
    setup->local_offset[1] = local_offset[1];
    setup->local_offset[2] = local_offset[2];
    setup->local_size[0] = local_size[0];
    setup->local_size[1] = local_size[1];
    setup->local_size[2] = local_size[2];
}

void gmx_parallel_3dfft_complex_limits_gpu(gmx_parallel_3dfft_gpu_t      pfft_setup,
                                          ivec                      complex_order,
                                          ivec                      local_ndata,
                                          ivec                      local_offset,
                                          ivec                      local_size)
{
    //yupinov why are they here
    //fprintf(stderr, "3dfft_complex_limits_gpu\n");
    gmx_parallel_3dfft_gpu_t setup = pfft_setup;
    setup->complex_order[0] = complex_order[0];
    setup->complex_order[1] = complex_order[1];
    setup->complex_order[2] = complex_order[2];
    setup->local_ndata[0] = local_ndata[0];
    setup->local_ndata[1] = local_ndata[1];
    setup->local_ndata[2] = local_ndata[2];
    setup->local_offset[0] = local_offset[0];
    setup->local_offset[1] = local_offset[1];
    setup->local_offset[2] = local_offset[2];
    setup->local_size[0] = local_size[0];
    setup->local_size[1] = local_size[1];
    setup->local_size[2] = local_size[2];
}

__global__ void transpose_xyz_yzx_kernel(int nx, int ny, int nz,
                                         hipfftComplex *cdata,
                                         bool forward)
{
    // transpose cdata to be contiguous in a y z x loop
    // z-dim has nz/2 + 1 elems
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    if ((x < nx) && (y < ny) && (z < (nz / 2 + 1)))
    {
        int idx1 = (x * ny + y) * (nz / 2 + 1) + z; //XYZ index into the first complex plane.
        int idx2 = ((ny + y) * (nz / 2 + 1) + z) * nx + x; //YZX-index into second complex plane

        //if (idx1 == 2)
        //    printf ("index %d %d %d %d %d %f %f\n", x, y, z, idx1, idx2, cdata[idx1].x, cdata[idx1].y);

        if (forward)
        {
            cdata[idx2] = cdata[idx1];
        }
        else
        {
            cdata[idx1] = cdata[idx2];
        }
    }
}

void transpose_xyz_yzx(int nx, int ny, int nz,
                       hipfftComplex *cdata,
                       bool forward, gmx_pme_t *pme)
{
    hipStream_t s = pme->gpu->pmeStream;
    int block_size = warp_size;
    dim3 dimGrid((nx + block_size - 1) / block_size, ny, nz / 2 + 1);
    dim3 dimBlock(block_size, 1, 1);
    transpose_xyz_yzx_kernel<<<dimGrid, dimBlock, 0, s>>>(nx, ny, nz, cdata, forward);
    CU_LAUNCH_ERR("transpose_xyz_yzx_kernel");
    //hipMemcpy(cdata, cdata + nx * ny * (nz/2+1), nx * ny * (nz/2+1) * sizeof(hipfftComplex), hipMemcpyDeviceToDevice);
}

void gmx_parallel_3dfft_execute_gpu(gmx_parallel_3dfft_gpu_t    pfft_setup,
                                   enum gmx_fft_direction  dir,
                                   int                     thread,
                                   gmx_wallcycle_t         wcycle,
                                   gmx_pme_t *pme,
                                   t_complex **complexFFTGridSavedOnDevice)
{
    hipStream_t s = pme->gpu->pmeStream;

    //fprintf(stderr, "3dfft_execute_gpu\n");
    gmx_parallel_3dfft_gpu_t setup = pfft_setup;

    int x = setup->n[0], y = setup->n[1], z = setup->n[2];

    //int rank = 3, batch = 1;
    /*fprintf(stderr, "FFT plan %dx%dx%d %s %p(%d)->%p(%d)\n", x, y, z,
      dir == GMX_FFT_REAL_TO_COMPLEX ? "HIPFFT_R2C" : "HIPFFT_C2R",
      setup->real_data, (int) sizeof(real),
      setup->complex_data, (int) sizeof(t_complex));*/
    /* FIX plan in advance (in init)
  if (hipfftPlanMany(&setup->plan, rank, setup->n,
            NULL, 0, 0,
            NULL, 0, 0,
            dir == GMX_FFT_REAL_TO_COMPLEX ? HIPFFT_R2C : HIPFFT_C2R,
            batch)
      != HIPFFT_SUCCESS) {
    fprintf(stderr, "PLAN_MANY FAIL!!! %d %p\n", thread, &wcycle);
    setup = NULL; // FIX
  }
  */

    if (dir == GMX_FFT_REAL_TO_COMPLEX)
    {
        //yupinov hack for padded data
        th_cpy(setup->rdata, setup->real_data, x * y * (z / 2 + 1) * 2 * sizeof(real), TH_LOC_CUDA, s);
        /*
        if (thread == 0) // redundant - called in thread 0 already though
        {
            hipfftReal *dest = setup->rdata;
            real *src = setup->real_data;
            for (int xi = 0; xi < x; xi++)
                for (int yi = 0; yi < y; yi++)
                 {
                    int size = z;
                    int stripe = (z / 2 + 1) * 2;
                    stat = hipMemcpy(dest, src, size * sizeof(real), hipMemcpyHostToDevice);
                    CU_RET_ERR(stat, "hipMemcpy R2C error");
                    dest += size;
                    src += stripe;
                 }
        }
        */


        #ifdef DEBUG_PME_TIMINGS_GPU
        events_record_start(gpu_events_fft_r2c, s);
        #endif
        hipfftResult_t result = hipfftExecR2C(setup->planR2C, setup->rdata, setup->cdata);
        if (result)
            fprintf(stderr, "cufft R2C error %d\n", result);
        // FIXME: -> y major, z middle, x minor or continuous
        transpose_xyz_yzx(x, y, z, setup->cdata, true, pme);
        #ifdef DEBUG_PME_TIMINGS_GPU
        events_record_stop(gpu_events_fft_r2c, s, ewcsPME_FFT_R2C, 0);
        #endif
    }
    else
    {
        //yupinov no second transfer
        th_cpy(setup->cdata + x * y * (z / 2 + 1), setup->complex_data, x * y * (z / 2 + 1) * sizeof(t_complex), TH_LOC_CUDA, s);
        // FIXME: y major, z middle, x minor or continuous ->
        #ifdef DEBUG_PME_TIMINGS_GPU
        events_record_start(gpu_events_fft_c2r, s);
        #endif
        transpose_xyz_yzx(x, y, z, setup->cdata, false, pme);
        hipfftResult_t result = hipfftExecC2R(setup->planC2R, setup->cdata, setup->rdata);
        if (result)
            fprintf(stderr, "cufft C2R error %d\n", result);
        #ifdef DEBUG_PME_TIMINGS_GPU
        events_record_stop(gpu_events_fft_c2r, s, ewcsPME_FFT_C2R, 0);
        #endif
    }

    if (dir == GMX_FFT_REAL_TO_COMPLEX)
    {
        hipfftComplex *complexFFTGrid = setup->cdata + x * y * (z / 2 + 1);
        if (!complexFFTGridSavedOnDevice)
            th_cpy(setup->complex_data, complexFFTGrid, x * y * (z / 2 + 1) * sizeof(t_complex), TH_LOC_HOST, s);
        else
            *complexFFTGridSavedOnDevice = (t_complex *)complexFFTGrid;
    }
    else
    {
        //yupinov hack for padded data

        th_cpy(setup->real_data, setup->rdata, x * y * (z / 2 + 1) * 2 * sizeof(real), TH_LOC_HOST, s);

        /*
        if (thread == 0) // redundant - called in thread 0 already though
        {
            real *dest = setup->real_data;
            hipfftReal *src = setup->rdata;
            for (int xi = 0; xi < x; xi++)
                for (int yi = 0; yi < y; yi++)
                 {
                    int size = z;
                    int stripe = (z / 2 + 1) * 2;
                    stat = hipMemcpy(dest, src, size * sizeof(real), hipMemcpyDeviceToHost);
                    CU_RET_ERR(stat, "hipMemcpy C2R error");
                    dest += stripe;
                    src += size;
                 }
        }
        */

    }
    // FIX destroy plans after
    //hipfftDestroy(setup->plan);
}

void gmx_parallel_3dfft_destroy_gpu(gmx_parallel_3dfft_gpu_t pfft_setup)
{
  //fprintf(stderr, "3dfft_destroy_gpu\n");
  gmx_parallel_3dfft_gpu_t setup = pfft_setup;

  hipfftDestroy(setup->planR2C);
  hipfftDestroy(setup->planC2R);

  hipError_t stat = hipFree((void **)setup->rdata);
  CU_RET_ERR(stat, "hipFree error");
  stat = hipFree((void **)setup->cdata);
  CU_RET_ERR(stat, "hipFree error");

  delete setup;
}
