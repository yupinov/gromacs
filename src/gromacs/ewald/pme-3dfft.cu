/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 2016, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */

/*! \internal \file
 *  \brief Implements CUDA FFT routines for PME GPU.
 *
 *  \author Aleksei Iupinov <a.yupinov@gmail.com>
 */

#include "gmxpre.h"

#include <assert.h>
#include <hipfft/hipfft.h>
#include "pme-gpu.h"
#include "pme-timings.cuh"
#include "pme-cuda.cuh"

#include "gromacs/utility/gmxassert.h"
#include "gromacs/utility/smalloc.h"
#include "gromacs/gpu_utils/cudautils.cuh"

struct gmx_parallel_3dfft_gpu
{
    ivec          ndata_real;
    ivec          size_real;
    ivec          size_complex;

    hipfftHandle   planR2C;
    hipfftHandle   planC2R;
    hipfftReal    *realGrid;
    hipfftComplex *complexGrid;

    /* unused */
    ivec                      local_offset;
};

void gmx_parallel_3dfft_init_gpu(gmx_parallel_3dfft_gpu_t *pfft_setup, ivec ndata, gmx_pme_t *pme)
{
    hipfftResult_t            result;
    gmx_parallel_3dfft_gpu_t setup;
    snew(setup, 1);

    setup->ndata_real[0] = ndata[XX];
    setup->ndata_real[1] = ndata[YY];
    setup->ndata_real[2] = ndata[ZZ];

    *pfft_setup = setup;

    if (pme->gpu->bGPUSingle)
    {
        ndata[XX] = pme->pmegrid_nx;
        ndata[YY] = pme->pmegrid_ny;
        ndata[ZZ] = pme->pmegrid_nz;
    }
    else
    {
        gmx_fatal(FARGS, "FFT size choice not implemented");
    }

    memcpy(setup->size_real, ndata, sizeof(setup->size_real));

    memcpy(setup->size_complex, setup->size_real, sizeof(setup->size_real));
    GMX_RELEASE_ASSERT(setup->size_complex[ZZ] % 2 == 0, "odd inplace cuFFT dimension size");
    setup->size_complex[ZZ] /= 2;
    // this is alright because Z includes overlap

    const int gridSizeComplex = setup->size_complex[XX] * setup->size_complex[YY] * setup->size_complex[ZZ];
    const int gridSizeReal    = setup->size_real[XX] * setup->size_real[YY] * setup->size_real[ZZ];

    memset(setup->local_offset, 0, sizeof(setup->local_offset)); //!

    setup->realGrid = (hipfftReal *)pme->gpu->grid;
    assert(setup->realGrid);
    setup->complexGrid = (hipfftComplex *)pme->gpu->fourierGrid;

    /*
       result = hipfftPlan3d(&setup->planR2C, setup->ndata_real[XX], setup->ndata_real[YY], setup->ndata_real[ZZ], HIPFFT_R2C);
       if (result != HIPFFT_SUCCESS)
        gmx_fatal(FARGS, "hipfftPlan3d R2C error %d\n", result);

       result = hipfftPlan3d(&setup->planC2R, setup->ndata_real[XX], setup->ndata_real[YY], setup->ndata_real[ZZ], HIPFFT_C2R);
       if (result != HIPFFT_SUCCESS)
        gmx_fatal(FARGS, "hipfftPlan3d C2R error %d\n", result);
     */


    const int rank = 3, batch = 1;
    result = hipfftPlanMany(&setup->planR2C, rank, setup->ndata_real,
                           setup->size_real, 1, gridSizeReal,
                           setup->size_complex, 1, gridSizeComplex,
                           HIPFFT_R2C,
                           batch);
    if (result != HIPFFT_SUCCESS)
    {
        gmx_fatal(FARGS, "hipfftPlanMany R2C error %d\n", result);
    }

    result = hipfftPlanMany(&setup->planC2R, rank, setup->ndata_real,
                           setup->size_complex, 1, gridSizeComplex,
                           setup->size_real, 1, gridSizeReal,
                           HIPFFT_C2R,
                           batch);
    if (result != HIPFFT_SUCCESS)
    {
        gmx_fatal(FARGS, "hipfftPlanMany C2R error %d\n", result);
    }

    hipStream_t s = pme->gpu->pmeStream;
    assert(s);
    result = hipfftSetStream(setup->planR2C, s);
    if (result != HIPFFT_SUCCESS)
    {
        gmx_fatal(FARGS, "hipfftSetStream R2C error %d\n", result);
    }

    result = hipfftSetStream(setup->planC2R, s);
    if (result != HIPFFT_SUCCESS)
    {
        gmx_fatal(FARGS, "hipfftSetStream C2R error %d\n", result);
    }
}

void gmx_parallel_3dfft_real_limits_gpu(gmx_parallel_3dfft_gpu_t      setup,
                                        ivec                          local_ndata,
                                        ivec                          local_offset,
                                        ivec                          local_size)
{
    if (local_ndata)
    {
        memcpy(local_ndata, setup->ndata_real, sizeof(setup->ndata_real));
    }
    if (local_size)
    {
        memcpy(local_size, setup->size_real, sizeof(setup->size_real));
    }
    if (local_offset)
    {
        memcpy(local_offset, setup->local_offset, sizeof(setup->local_offset));
    }
}

void gmx_parallel_3dfft_complex_limits_gpu(gmx_parallel_3dfft_gpu_t      setup,
                                           ivec                          local_ndata,
                                           ivec                          local_offset,
                                           ivec                          local_size)
{
    if (local_ndata)
    {
        memcpy(local_ndata, setup->ndata_real, sizeof(setup->ndata_real));
        local_ndata[ZZ] = local_ndata[ZZ] / 2 + 1;
    }
    if (local_size)
    {
        memcpy(local_size, setup->size_complex, sizeof(setup->size_complex));
    }
    if (local_offset)
    {
        memcpy(local_offset, setup->local_offset, sizeof(setup->local_offset));
    }
}

void gmx_parallel_3dfft_execute_gpu(gmx_pme_t        *pme,
                                    gmx_fft_direction dir,
                                    const int         grid_index)
{
    /*
       const int gridSizeComplex = setup->size_complex[XX] * setup->size_complex[YY] * setup->size_complex[ZZ] * sizeof(hipfftComplex);
       const int gridSizeReal = setup->size_real[XX] * setup->size_real[YY] * setup->size_real[ZZ] * sizeof(hipfftReal);
     */
    gmx_parallel_3dfft_gpu_t setup = pme->gpu->pfft_setup_gpu[grid_index];

    if (dir == GMX_FFT_REAL_TO_COMPLEX)
    {
        //if (!pme->gpu->keepGPUDataBetweenSpreadAndR2C)
        //    cu_copy_H2D(setup->realGrid, setup->hostRealGrid, gridSizeReal);
        // CPU spread and GPU FFT? unlikely, only for debug

        pme_gpu_timing_start(pme, ewcsPME_FFT_R2C);

        hipfftResult_t result = hipfftExecR2C(setup->planR2C, setup->realGrid, setup->complexGrid);

        pme_gpu_timing_stop(pme, ewcsPME_FFT_R2C);

        if (result)
        {
            fprintf(stderr, "cufft R2C error %d\n", result);
        }
    }
    else
    {
        //if (!pme->gpu->keepGPUDataBetweenSolveAndC2R)
        //    cu_copy_H2D(setup->complexGrid, setup->hostComplexGrid, gridSizeComplex);
        // CPU solve and GPU FFT? unlikely, only for debug

        pme_gpu_timing_start(pme, ewcsPME_FFT_C2R);

        hipfftResult_t result = hipfftExecC2R(setup->planC2R, setup->complexGrid, setup->realGrid);

        pme_gpu_timing_stop(pme, ewcsPME_FFT_C2R);

        if (result)
        {
            fprintf(stderr, "cufft C2R error %d\n", result);
        }
    }
    /*
       if (dir == GMX_FFT_REAL_TO_COMPLEX)
       {
        // GPU FFT and CPU solve - unlikely, only for debug
        if (!pme->gpu->keepGPUDataBetweenR2CAndSolve)
            cu_copy_D2H(setup->hostComplexGrid, setup->complexGrid, gridSizeComplex);
       }
       else
       {
        //GPU FFT and CPU gather - unlikely, only for debug
        if (!pme->gpu->keepGPUDataBetweenC2RAndGather)
            cu_copy_D2H(setup->hostRealGrid, setup->realGrid, gridSizeReal);
       }
     */
}

void gmx_parallel_3dfft_destroy_gpu(const gmx_parallel_3dfft_gpu_t &pfft_setup)
{
    if (pfft_setup)
    {
        hipfftResult_t result;

        result = hipfftDestroy(pfft_setup->planR2C);
        if (result != HIPFFT_SUCCESS)
        {
            gmx_fatal(FARGS, "hipfftDestroy R2C error %d\n", result);
        }
        result = hipfftDestroy(pfft_setup->planC2R);
        if (result != HIPFFT_SUCCESS)
        {
            gmx_fatal(FARGS, "hipfftDestroy C2R error %d\n", result);
        }

        sfree(pfft_setup);
    }
}
