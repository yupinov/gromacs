#include <assert.h>
#include <hipfft/hipfft.h>
#include "pme-timings.cuh"
#include "pme-cuda.cuh"

#include "gromacs/utility/gmxassert.h"
#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/gpu_utils/cuda_arch_utils.cuh"

struct gmx_parallel_3dfft_gpu
{
    real *hostRealGrid;
    t_complex *hostComplexGrid;

    /* unused */
    MPI_Comm                  comm[2];
    gmx_bool                  bReproducible;

    ivec                      complex_order;
    ivec                      local_offset;

    ivec ndata_real;
    ivec size_real;
    ivec size_complex;

    hipfftHandle planR2C;
    hipfftHandle planC2R;
    hipfftReal *realGrid;
    hipfftComplex *complexGrid;
};

//yupinov warn against double precision

void gmx_parallel_3dfft_init_gpu(gmx_parallel_3dfft_gpu_t *pfft_setup,
                                   ivec                      ndata,
                                   real **real_data,
                                   t_complex **complex_data,
                                   MPI_Comm                  comm[2],
gmx_bool                  bReproducible,
gmx_pme_t *pme)
{
    hipfftResult_t result;
    gmx_parallel_3dfft_gpu_t setup = new gmx_parallel_3dfft_gpu;

    //yupinov FIXME: this copies the already setup pointer, to check them after execute

    setup->hostRealGrid = *real_data;

    setup->hostComplexGrid = *complex_data;

    setup->comm[0] = comm[0];
    setup->comm[1] = comm[1];
    setup->bReproducible = bReproducible;

    setup->ndata_real[0] = ndata[XX];
    setup->ndata_real[1] = ndata[YY];
    setup->ndata_real[2] = ndata[ZZ];

    *pfft_setup = setup;

    if (pme->bGPUSingle)
    {
        ndata[XX] = pme->pmegrid_nx;
        ndata[YY] = pme->pmegrid_ny;
        ndata[ZZ] = pme->pmegrid_nz;
    }
    else
        gmx_fatal(FARGS, "FFT size choice not implemented");

    memcpy(setup->size_real, ndata, sizeof(setup->size_real));

    memcpy(setup->size_complex, setup->size_real, sizeof(setup->size_real));
    GMX_RELEASE_ASSERT(setup->size_complex[ZZ] % 2 == 0, "odd inplace cuFFT dimension size");
    setup->size_complex[ZZ] /= 2;
    //this is alright because Z includes overlap

    const int gridSizeComplex = setup->size_complex[XX] * setup->size_complex[YY] * setup->size_complex[ZZ];
    const int gridSizeReal = setup->size_real[XX] * setup->size_real[YY] * setup->size_real[ZZ];

    setup->realGrid = (hipfftReal *)pme->gpu->grid;
    assert(setup->realGrid);
    setup->complexGrid = (hipfftComplex *)PMEMemoryFetch(PME_ID_COMPLEX_GRID, gridSizeComplex * sizeof(hipfftComplex), ML_DEVICE);

    /*
    result = hipfftPlan3d(&setup->planR2C, setup->ndata_real[XX], setup->ndata_real[YY], setup->ndata_real[ZZ], HIPFFT_R2C);
    if (result != HIPFFT_SUCCESS)
        gmx_fatal(FARGS, "hipfftPlan3d R2C error %d\n", result);

    result = hipfftPlan3d(&setup->planC2R, setup->ndata_real[XX], setup->ndata_real[YY], setup->ndata_real[ZZ], HIPFFT_C2R);
    if (result != HIPFFT_SUCCESS)
        gmx_fatal(FARGS, "hipfftPlan3d C2R error %d\n", result);
    */


    const int rank = 3, batch = 1;
    result = hipfftPlanMany(&setup->planR2C, rank, setup->ndata_real,
                                       setup->size_real, 1, gridSizeReal,
                                       setup->size_complex, 1, gridSizeComplex,
                                       HIPFFT_R2C,
                                       batch);
    if (result != HIPFFT_SUCCESS)
        gmx_fatal(FARGS, "hipfftPlanMany R2C error %d\n", result);

    result = hipfftPlanMany(&setup->planC2R, rank, setup->ndata_real,
                                       setup->size_complex, 1, gridSizeComplex,
                                       setup->size_real, 1, gridSizeReal,
                                       HIPFFT_C2R,
                                       batch);
    if (result != HIPFFT_SUCCESS)
        gmx_fatal(FARGS, "hipfftPlanMany C2R error %d\n", result);

    hipStream_t s = pme->gpu->pmeStream;
    assert(s);
    result = hipfftSetStream(setup->planR2C, s);
    if (result != HIPFFT_SUCCESS)
        gmx_fatal(FARGS, "hipfftSetStream R2C error %d\n", result);

    result = hipfftSetStream(setup->planC2R, s);
    if (result != HIPFFT_SUCCESS)
        gmx_fatal(FARGS, "hipfftSetStream C2R error %d\n", result);
}

void gmx_parallel_3dfft_real_limits_gpu(gmx_parallel_3dfft_gpu_t      setup,
                                       ivec                      local_ndata,
                                       ivec                      local_offset,
                                       ivec                      local_size)
{
    if (local_ndata)
        memcpy(local_ndata, setup->ndata_real, sizeof(setup->ndata_real));
    if (local_size)
        memcpy(local_size, setup->size_real, sizeof(setup->size_real));

    //yupinov
    setup->local_offset[0] = local_offset[0];
    setup->local_offset[1] = local_offset[1];
    setup->local_offset[2] = local_offset[2];
}

void gmx_parallel_3dfft_complex_limits_gpu(gmx_parallel_3dfft_gpu_t      setup,
                                          ivec                      complex_order,
                                          ivec                      local_ndata,
                                          ivec                      local_offset,
                                          ivec                      local_size)
{
    if (local_ndata)
    {
        memcpy(local_ndata, setup->ndata_real, sizeof(setup->ndata_real));
        local_ndata[ZZ] = local_ndata[ZZ] / 2 + 1;
    }
    if (local_size)
        memcpy(local_size, setup->size_complex, sizeof(setup->size_complex));

    //yupinov why are they here
    setup->complex_order[0] = complex_order[0];
    setup->complex_order[1] = complex_order[1];
    setup->complex_order[2] = complex_order[2];
    setup->local_offset[0] = local_offset[0];
    setup->local_offset[1] = local_offset[1];
    setup->local_offset[2] = local_offset[2];
}

void gmx_parallel_3dfft_execute_gpu(gmx_parallel_3dfft_gpu_t    pfft_setup,
                                   enum gmx_fft_direction  dir,
                                   gmx_pme_t *pme)
{
    hipStream_t s = pme->gpu->pmeStream;

    gmx_parallel_3dfft_gpu_t setup = pfft_setup;

    const int gridSizeComplex = setup->size_complex[XX] * setup->size_complex[YY] * setup->size_complex[ZZ] * sizeof(hipfftComplex);
    const int gridSizeReal = setup->size_real[XX] * setup->size_real[YY] * setup->size_real[ZZ] * sizeof(hipfftReal);

    if (dir == GMX_FFT_REAL_TO_COMPLEX)
    {      
        if (!pme->gpu->keepGPUDataBetweenSpreadAndR2C)
            cu_copy_H2D_async(setup->realGrid, setup->hostRealGrid, gridSizeReal, s);

        pme_gpu_timing_start(pme, ewcsPME_FFT_R2C);

        hipfftResult_t result = hipfftExecR2C(setup->planR2C, setup->realGrid, setup->complexGrid);

        pme_gpu_timing_stop(pme, ewcsPME_FFT_R2C);

        if (result)
            fprintf(stderr, "cufft R2C error %d\n", result);
    }
    else
    {
        if (!pme->gpu->keepGPUDataBetweenSolveAndC2R)
            cu_copy_H2D_async(setup->complexGrid, setup->hostComplexGrid, gridSizeComplex, s);

        pme_gpu_timing_start(pme, ewcsPME_FFT_C2R);

        hipfftResult_t result = hipfftExecC2R(setup->planC2R, setup->complexGrid, setup->realGrid);

        pme_gpu_timing_stop(pme, ewcsPME_FFT_C2R);

        if (result)
            fprintf(stderr, "cufft C2R error %d\n", result);
    }

    if (dir == GMX_FFT_REAL_TO_COMPLEX)
    {
        hipDeviceSynchronize();
        if (!pme->gpu->keepGPUDataBetweenR2CAndSolve)
            cu_copy_D2H/*_async*/(setup->hostComplexGrid, setup->complexGrid, gridSizeComplex);//, s);
    }
    else
    {
        if (!pme->gpu->keepGPUDataBetweenC2RAndGather)
            cu_copy_D2H_async(setup->hostRealGrid, setup->realGrid, gridSizeReal, s);
    }
}

void gmx_parallel_3dfft_destroy_gpu(const gmx_parallel_3dfft_gpu_t &pfft_setup)
{
    if (pfft_setup)
    {
        hipfftResult_t result;

        result = hipfftDestroy(pfft_setup->planR2C);
        if (result != HIPFFT_SUCCESS)
            gmx_fatal(FARGS, "hipfftDestroy R2C error %d\n", result);
        result = hipfftDestroy(pfft_setup->planC2R);
        if (result != HIPFFT_SUCCESS)
            gmx_fatal(FARGS, "hipfftDestroy C2R error %d\n", result);

        delete pfft_setup;
    }
}
