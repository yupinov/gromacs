#include "hip/hip_runtime.h"
/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 2016, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */

/*! \internal \file
 *  \brief Implements PME GPU Fourier grid solving in CUDA.
 *
 *  \author Aleksei Iupinov <a.yupinov@gmail.com>
 */

#include "gmxpre.h"

#include "gromacs/utility/gmxassert.h"
#include "pme.cuh"
#include "pme-gpu.h"   //?
#include "pme-internal.h"
#include "pme-solve.h" //? some work structure reliance?

void pme_gpu_alloc_energy_virial(const gmx_pme_t *pme, const int gmx_unused grid_index)
{
    pme->gpu->energyAndVirialSize = 7 * sizeof(real); /* 6 virial components + energy */
    pme->gpu->energyAndVirial     = (real *)PMEMemoryFetch(pme, PME_ID_ENERGY_AND_VIRIAL, pme->gpu->energyAndVirialSize, ML_DEVICE);
}

void pme_gpu_clear_energy_virial(const gmx_pme_t *pme, const int gmx_unused grid_index)
{
    hipError_t stat = hipMemsetAsync(pme->gpu->energyAndVirial, 0, pme->gpu->energyAndVirialSize, pme->gpu->pmeStream);
    CU_RET_ERR(stat, "PME solve energies/virial hipMemsetAsync");
}

/*! \brief
 *
 * Copies the pre-computed B-spline modules to the GPU
 */
void pme_gpu_copy_bspline_moduli(const gmx_pme_t *pme)
{
    for (int i = 0; i < DIM; i++)
    {
        int       gridSize;
        PMEDataID id;
        switch (i)
        {
            case XX:
                gridSize = pme->nkx;
                id       = PME_ID_BSP_MOD_XX;
                break;

            case YY:
                gridSize = pme->nky;
                id       = PME_ID_BSP_MOD_YY;
                break;

            case ZZ:
                gridSize = pme->nkz;
                id       = PME_ID_BSP_MOD_ZZ;
                break;
        }
        int   modSize  = gridSize * sizeof(real);
        real *bspMod_h = (real *)PMEMemoryFetch(pme, id, modSize, ML_HOST);
        memcpy(bspMod_h, pme->bsp_mod[i], modSize);
        real *bspMod_d = (real *)PMEMemoryFetch(pme, id, modSize, ML_DEVICE);
        cu_copy_H2D_async(bspMod_d, bspMod_h, modSize, pme->gpu->pmeStream);
    }
}


#define THREADS_PER_BLOCK (4 * warp_size)

template<
    const gmx_bool bEnerVir,
    // should the energy/virial be computed
    const gmx_bool YZXOrdering
    // false - GPU solve works in a XYZ ordering (after a single-rank cuFFT)
    // true - GPU solve works in a YZX ordering, like the CPU one (after FFTW)
    >
__global__ void pme_solve_kernel
    (const int localCountMajor, const int localCountMiddle, const int localCountMinor,
    const int localOffsetMinor, const int localOffsetMajor, const int localOffsetMiddle,
    const int localSizeMinor, /*const int localSizeMajor,*/ const int localSizeMiddle,
    const real * __restrict__ BSplineModuleMinor,
    const real * __restrict__ BSplineModuleMajor,
    const real * __restrict__ BSplineModuleMiddle,
    float2 * __restrict__ globalGrid,
    const struct pme_gpu_const_parameters constants,
    real * __restrict__ virialAndEnergy)
{
    // this is a PME solve kernel
    // each thread works on one cell of the Fourier space complex 3D grid (float2 * __restrict__ grid)
    // each block handles THREADS_PER_BLOCK cells - depending on the grid contiguous dimension size,
    // that can range from a part of a single gridline to several complete gridlines
    // the minor dimension index is (YZXOrdering ? XX : ZZ)
    const int threadLocalId = (threadIdx.y * blockDim.x) + threadIdx.x;
    //const int blockSize = (blockDim.x * blockDim.y * blockDim.z); // == cellsPerBlock
    const int blockSize = THREADS_PER_BLOCK;
    //const int threadId = blockId * blockSize + threadLocalId;

    const int nMinor  = !YZXOrdering ? constants.localGridSize.z : constants.localGridSize.x; //yupinov fix all pme->nkx and such
    const int nMajor  = !YZXOrdering ? constants.localGridSize.x : constants.localGridSize.y;
    const int nMiddle = !YZXOrdering ? constants.localGridSize.y : constants.localGridSize.z;

    int       maxkMajor  = (nMajor + 1) / 2;  //X or Y
    int       maxkMiddle = (nMiddle + 1) / 2; //Y OR Z => only check for !YZX
    int       maxkMinor  = (nMinor + 1) / 2;  //Z or X => only check for YZX

    const int enerVirSize = 7;

    real      energy = 0.0f;
    real      virxx  = 0.0f, virxy = 0.0f, virxz = 0.0f, viryy = 0.0f, viryz = 0.0f, virzz = 0.0f;

    const int indexMinor  = blockIdx.x * blockDim.x + threadIdx.x;
    const int indexMiddle = blockIdx.y * blockDim.y + threadIdx.y;
    const int indexMajor  = blockIdx.z * blockDim.z + threadIdx.z;

    if ((indexMajor < localCountMajor) && (indexMiddle < localCountMiddle) && (indexMinor < localCountMinor))
    {
        /* The offset should be equal to the global thread index */
        float2    *globalGridPtr = globalGrid + (indexMajor * localSizeMiddle + indexMiddle) * localSizeMinor + indexMinor;

        const int  kMajor = indexMajor + localOffsetMajor;
        /* Checking either X in XYZ, or Y in YZX cases */
        const real mMajor = (kMajor < maxkMajor) ? kMajor : (kMajor - nMajor);

        const int  kMiddle = indexMiddle + localOffsetMiddle;
        real       mMiddle = kMiddle;
        /* Checking Y in XYZ case */
        if (!YZXOrdering)
        {
            mMiddle = (kMiddle < maxkMiddle) ? kMiddle : (kMiddle - nMiddle);
        }
        /* We should skip the k-space point (0,0,0) */

        const int      kMinor       = localOffsetMinor + indexMinor;
        const gmx_bool notZeroPoint = (kMinor > 0 || kMajor > 0 || kMiddle > 0);
        real           mMinor       = kMinor, mhxk, mhyk, mhzk, m2k;

        /* Checking X in YZX case */
        if (YZXOrdering)
        {
            mMinor = (kMinor < maxkMinor) ? kMinor : (kMinor - nMinor);
        }

        real mX, mY, mZ;
        if (YZXOrdering)
        {
            mX = mMinor;
            mY = mMajor;
            mZ = mMiddle;
        }
        else
        {
            mX = mMajor;
            mY = mMiddle;
            mZ = mMinor;
        }

        /* 0.5 correction for corner points of a minor dimension */
        real corner_fac = 1.0f;
        if (YZXOrdering)
        {
            if (kMiddle == 0 || kMiddle == maxkMiddle)
            {
                corner_fac = 0.5f;
            }
        }
        else
        {
            if (kMinor == 0 || kMinor == maxkMinor)
            {
                corner_fac = 0.5f;
            }
        }

        if (notZeroPoint)
        {
            mhxk       = mX * constants.recipbox[XX].x;
            mhyk       = mX * constants.recipbox[XX].y + mY * constants.recipbox[YY].y;
            mhzk       = mX * constants.recipbox[XX].z + mY * constants.recipbox[YY].z + mZ * constants.recipbox[ZZ].z;

            m2k        = mhxk * mhxk + mhyk * mhyk + mhzk * mhzk;
            real denom = m2k * real(M_PI) * constants.volume * BSplineModuleMajor[kMajor] * BSplineModuleMiddle[kMiddle] * BSplineModuleMinor[kMinor];
            real tmp1  = -constants.ewaldFactor * m2k;

            denom = 1.0f / denom;
            tmp1  = expf(tmp1);
            real   etermk = constants.elFactor * tmp1 * denom;

            float2 gridValue    = *globalGridPtr;
            float2 oldGridValue = gridValue;
            gridValue.x   *= etermk;
            gridValue.y   *= etermk;
            *globalGridPtr = gridValue;

            if (bEnerVir)
            {
                real tmp1k = 2.0f * (gridValue.x * oldGridValue.x + gridValue.y * oldGridValue.y);

                real vfactor = (constants.ewaldFactor + 1.0f / m2k) * 2.0f;
                real ets2    = corner_fac * tmp1k;
                energy = ets2;

                real ets2vf  = ets2 * vfactor;

                virxx   = ets2vf * mhxk * mhxk - ets2;
                virxy   = ets2vf * mhxk * mhyk;
                virxz   = ets2vf * mhxk * mhzk;
                viryy   = ets2vf * mhyk * mhyk - ets2;
                viryz   = ets2vf * mhyk * mhzk;
                virzz   = ets2vf * mhzk * mhzk - ets2;
            }
        }
    }

    if (bEnerVir)
    {
        /* The energy and virial reduction */

#if (GMX_PTX_ARCH >= 300)
        /* There really should be a shuffle reduction here!
         * (only for orders of power of 2)
         */
        /*
           if (!(blockSize & (blockSize - 1)))
           {

           }
           else
         */
#endif
        {
            __shared__ real virialAndEnergyShared[enerVirSize * blockSize];
            // 3.5k smem per block - a serious limiter!

            /*  a 7-thread reduction in shared memory inspired by reduce_force_j_generic */
            if (threadLocalId < blockSize)
            {
                virialAndEnergyShared[threadLocalId + 0 * blockSize] = virxx;
                virialAndEnergyShared[threadLocalId + 1 * blockSize] = viryy;
                virialAndEnergyShared[threadLocalId + 2 * blockSize] = virzz;
                virialAndEnergyShared[threadLocalId + 3 * blockSize] = virxy;
                virialAndEnergyShared[threadLocalId + 4 * blockSize] = virxz;
                virialAndEnergyShared[threadLocalId + 5 * blockSize] = viryz;
                virialAndEnergyShared[threadLocalId + 6 * blockSize] = energy;
            }
            __syncthreads();

            /* Reducing every component to fit into warp_size */
            for (int s = blockSize >> 1; s >= warp_size; s >>= 1)
            {
#pragma unroll
                for (int i = 0; i < enerVirSize; i++)
                {
                    if (threadLocalId < s) // split per threads?
                    {
                        virialAndEnergyShared[i * blockSize + threadLocalId] += virialAndEnergyShared[i * blockSize + threadLocalId + s];
                    }
                }
                __syncthreads();
            }

            const int threadsPerComponent    = warp_size / enerVirSize;         // this is also the stride, will be 32 / 7 = 4
            const int contributionsPerThread = warp_size / threadsPerComponent; // will be 32 / 4 = 8
            if (threadLocalId < enerVirSize * threadsPerComponent)
            {
                const int componentIndex        = threadLocalId / threadsPerComponent;
                const int threadComponentOffset = threadLocalId - componentIndex * threadsPerComponent;

                float     sum = 0.0f;
#pragma unroll
                for (int j = 0; j < contributionsPerThread; j++)
                {
                    sum += virialAndEnergyShared[componentIndex * blockSize + j * threadsPerComponent + threadComponentOffset];
                }
                // write to global memory
                atomicAdd(virialAndEnergy + componentIndex, sum);
            }

            /* A naive reduction */
            /*
               if (threadLocalId < blockSize)
               {
                virialAndEnergyShared[sizing * threadLocalId + 0] = virxx;
                virialAndEnergyShared[sizing * threadLocalId + 1] = viryy;
                virialAndEnergyShared[sizing * threadLocalId + 2] = virzz;
                virialAndEnergyShared[sizing * threadLocalId + 3] = virxy;
                virialAndEnergyShared[sizing * threadLocalId + 4] = virxz;
                virialAndEnergyShared[sizing * threadLocalId + 5] = viryz;
                virialAndEnergyShared[sizing * threadLocalId + 6] = energy;
               }
               __syncthreads();
               #pragma unroll
               for (unsigned int stride = 1; stride < blockSize; stride <<= 1)
               {
                if ((threadLocalId % (stride << 1) == 0))
                {
               #pragma unroll
                    for (int i = 0; i < sizing; i++)
                        virialAndEnergyShared[sizing * threadLocalId + i] += virialAndEnergyShared[sizing * (threadLocalId + stride) + i];
                }
                __syncthreads();
               }
               if (threadLocalId < sizing)
               {
                atomicAdd(virialAndEnergy + threadLocalId, virialAndEnergyShared[threadLocalId]);
               }
             */
        }
    }
}

void solve_pme_gpu(struct gmx_pme_t *pme, t_complex *grid,
                   gmx_bool bEnerVir)
{
    /* do recip sum over local cells in grid */

    const gmx_bool YZXOrdering = !pme->gpu->bGPUFFT;
    /* true: y major, z middle, x minor or continuous - the CPU FFTW way */
    /* false: x major, y middle, z minor - the single rank GPU cuFFT way */

    hipStream_t s = pme->gpu->pmeStream;

    ivec         local_ndata, local_offset, local_size, complex_order;
    /* Dimensions should be identical for A/B grid, so we just use A here */

    if (pme->gpu->bGPUFFT)
    {
        gmx_parallel_3dfft_complex_limits_gpu(pme->gpu->pfft_setup_gpu[PME_GRID_QA], local_ndata, local_offset, local_size);
    }
    else
    {
        gmx_parallel_3dfft_complex_limits(pme->pfft_setup[PME_GRID_QA], complex_order, local_ndata, local_offset, local_size);
    }

    const int minorDim  = !YZXOrdering ? ZZ : XX;
    const int middleDim = !YZXOrdering ? YY : ZZ;
    const int majorDim  = !YZXOrdering ? XX : YY;

    /*
       const int nMinor =  local_ndata[minorDim]; //!YZXOrdering ? pme->nkz : pme->nkx;
       const int nMajor = local_ndata[majorDim];
       const int nMiddle = local_ndata[middleDim]; //these are basic sizes, so what
     */
    const real *bspModMinor_d  = (real *)PMEMemoryFetch(pme, !YZXOrdering ? PME_ID_BSP_MOD_ZZ : PME_ID_BSP_MOD_XX, 0, ML_DEVICE);
    const real *bspModMiddle_d = (real *)PMEMemoryFetch(pme, !YZXOrdering ? PME_ID_BSP_MOD_YY : PME_ID_BSP_MOD_ZZ, 0, ML_DEVICE);
    const real *bspModMajor_d  = (real *)PMEMemoryFetch(pme, !YZXOrdering ? PME_ID_BSP_MOD_XX : PME_ID_BSP_MOD_YY, 0, ML_DEVICE);

    const int   gridSize = local_size[XX] * local_size[YY] * local_size[ZZ] * sizeof(float2);

    float2     *grid_d = (float2 *)pme->gpu->fourierGrid;
    if (!pme->gpu->bGPUFFT)
    {
        cu_copy_H2D_async(grid_d, grid, gridSize, s);
    }

    // Z-dimension is too small in CUDA limitations (64 on CC30?), so instead of major-middle-minor sizing we do minor-middle-major
    const int maxBlockSize      = THREADS_PER_BLOCK;
    const int gridLineSize      = local_size[minorDim];
    const int gridLinesPerBlock = max(maxBlockSize / gridLineSize, 1);
    const int blocksPerGridLine = (gridLineSize + maxBlockSize - 1) / maxBlockSize; // rounded up
    dim3 threads((maxBlockSize + gridLinesPerBlock - 1) / gridLinesPerBlock, gridLinesPerBlock);
    const int blockSize = threads.x * threads.y * threads.z;
    GMX_RELEASE_ASSERT(blockSize >= maxBlockSize, "wrong PME GPU solve launch parameters");
    // we want to have spare threads to zero all the shared memory which we use in CC2.0 shared mem reduction

    dim3 blocks(blocksPerGridLine,
                (local_ndata[middleDim] + gridLinesPerBlock - 1) / gridLinesPerBlock, // rounded up middle dimension block number
                local_ndata[majorDim]);

    pme_gpu_timing_start(pme, ewcsPME_SOLVE);

    if (YZXOrdering)
    {
        if (bEnerVir)
        {
            pme_solve_kernel<TRUE, TRUE> <<< blocks, threads, 0, s>>>
            (local_ndata[majorDim], local_ndata[middleDim], local_ndata[minorDim],
             local_offset[minorDim], local_offset[majorDim], local_offset[middleDim],
             local_size[minorDim], /*local_size[majorDim],*/ local_size[middleDim],
             bspModMinor_d, bspModMajor_d, bspModMiddle_d,
             grid_d,
             pme->gpu->constants,
             pme->gpu->energyAndVirial);
        }
        else
        {
            pme_solve_kernel<FALSE, TRUE> <<< blocks, threads, 0, s>>>
            (local_ndata[majorDim], local_ndata[middleDim], local_ndata[minorDim ],
             local_offset[minorDim], local_offset[majorDim], local_offset[middleDim],
             local_size[minorDim], /*local_size[majorDim],*/ local_size[middleDim],
             bspModMinor_d, bspModMajor_d, bspModMiddle_d,
             grid_d,
             pme->gpu->constants,
             pme->gpu->energyAndVirial);
        }
    }
    else
    {
        if (bEnerVir)
        {
            pme_solve_kernel<TRUE, FALSE> <<< blocks, threads, 0, s>>>
            (local_ndata[majorDim], local_ndata[middleDim], local_ndata[minorDim],
             local_offset[minorDim], local_offset[majorDim], local_offset[middleDim],
             local_size[minorDim], /*local_size[majorDim],*/ local_size[middleDim],
             bspModMinor_d, bspModMajor_d, bspModMiddle_d,
             grid_d,
             pme->gpu->constants,
             pme->gpu->energyAndVirial);
        }
        else
        {
            pme_solve_kernel<FALSE, FALSE> <<< blocks, threads, 0, s>>>
            (local_ndata[majorDim], local_ndata[middleDim], local_ndata[minorDim ],
             local_offset[minorDim], local_offset[majorDim], local_offset[middleDim],
             local_size[minorDim], /*local_size[majorDim],*/ local_size[middleDim],
             bspModMinor_d, bspModMajor_d, bspModMiddle_d,
             grid_d,
             pme->gpu->constants,
             pme->gpu->energyAndVirial);
        }
    }
    CU_LAUNCH_ERR("pme_solve_kernel");

    pme_gpu_timing_stop(pme, ewcsPME_SOLVE);

    if (bEnerVir)
    {
        real       *energyAndVirial_h = (real *)PMEMemoryFetch(pme, PME_ID_ENERGY_AND_VIRIAL, pme->gpu->energyAndVirialSize, ML_HOST);
        cu_copy_D2H_async(energyAndVirial_h, pme->gpu->energyAndVirial, pme->gpu->energyAndVirialSize, s);
        hipError_t stat = hipEventRecord(pme->gpu->syncEnerVirD2H, s);
        CU_RET_ERR(stat, "PME solve energy/virial sync fail");
    }

    if (!pme->gpu->bGPUFFT)
    {
        cu_copy_D2H_async(grid, grid_d, gridSize, s);
        hipError_t stat = hipEventRecord(pme->gpu->syncSolveGridD2H, s);
        CU_RET_ERR(stat, "PME solve grid sync fail");
    }
}

void pme_gpu_get_energy_virial(const gmx_pme_t *pme)
{
    hipStream_t             s = pme->gpu->pmeStream;

    struct pme_solve_work_t *work          = &pme->solve_work[0];
    real                    *work_energy_q = &(work->energy_q);
    matrix                  &work_vir_q    = work->vir_q;

    hipError_t              stat = hipStreamWaitEvent(s, pme->gpu->syncEnerVirD2H, 0);
    CU_RET_ERR(stat, "error while waiting for PME solve");
    real                    *energyAndVirial_h = (real *)PMEMemoryFetch(pme, PME_ID_ENERGY_AND_VIRIAL, pme->gpu->energyAndVirialSize, ML_HOST);
    real                     energy            = 0.0;
    real                     virxx             = 0.0, virxy = 0.0, virxz = 0.0, viryy = 0.0, viryz = 0.0, virzz = 0.0;

    int j = 0;
    virxx  += energyAndVirial_h[j++];
    viryy  += energyAndVirial_h[j++];
    virzz  += energyAndVirial_h[j++];
    virxy  += energyAndVirial_h[j++];
    virxz  += energyAndVirial_h[j++];
    viryz  += energyAndVirial_h[j++];
    energy += energyAndVirial_h[j++];
    for (j = 0; j < 7; j++)
    {
        GMX_RELEASE_ASSERT(!isnan(energyAndVirial_h[j]), "PME GPU is broken - NaN reduction result");
    }

    work_vir_q[XX][XX] = 0.25 * virxx;
    work_vir_q[YY][YY] = 0.25 * viryy;
    work_vir_q[ZZ][ZZ] = 0.25 * virzz;
    work_vir_q[XX][YY] = work_vir_q[YY][XX] = 0.25 * virxy;
    work_vir_q[XX][ZZ] = work_vir_q[ZZ][XX] = 0.25 * virxz;
    work_vir_q[YY][ZZ] = work_vir_q[ZZ][YY] = 0.25 * viryz;

    /* This energy should be corrected for a charged system */
    *work_energy_q = 0.5 * energy;
}
