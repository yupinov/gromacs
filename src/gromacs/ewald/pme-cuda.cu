#include <vector>
#include <stdio.h>
#include "gromacs/gpu_utils/cudautils.cuh"

#include "pme-cuda.cuh"

#include <assert.h>

void pme_gpu_update_flags(
        gmx_pme_gpu_t *pmeGPU,
        gmx_bool keepGPUDataBetweenSpreadAndR2C,
        gmx_bool keepGPUDataBetweenR2CAndSolve,
        gmx_bool keepGPUDataBetweenSolveAndC2R,
        gmx_bool keepGPUDataBetweenC2RAndGather
        )
{
    pmeGPU->keepGPUDataBetweenSpreadAndR2C = keepGPUDataBetweenSpreadAndR2C;
    pmeGPU->keepGPUDataBetweenR2CAndSolve = keepGPUDataBetweenR2CAndSolve;
    pmeGPU->keepGPUDataBetweenSolveAndC2R = keepGPUDataBetweenSolveAndC2R;
    pmeGPU->keepGPUDataBetweenC2RAndGather = keepGPUDataBetweenC2RAndGather;
}

void pme_gpu_init(gmx_pme_gpu_t **pmeGPU)
{
    *pmeGPU = new gmx_pme_gpu_t;
    hipError_t stat;
//yupinov dealloc@

#if GMX_CUDA_VERSION >= 5050
    int highest_priority;
    int lowest_priority;
    stat = hipDeviceGetStreamPriorityRange(&lowest_priority, &highest_priority);
    CU_RET_ERR(stat, "PME hipDeviceGetStreamPriorityRange failed");
    stat = hipStreamCreateWithPriority(&(*pmeGPU)->pmeStream,
                                            //hipStreamNonBlocking,
                                            hipStreamDefault,
                                            highest_priority);
    //yupinov: fighting with nbnxn non-local for highest priority - check on MPI!
    CU_RET_ERR(stat, "hipStreamCreateWithPriority on PME stream failed");
#else
    stat = hipStreamCreate(&(*pme)->pmeStream);
    CU_RET_ERR(stat, "PME hipStreamCreate error");
#endif

    pme_gpu_update_flags(*pmeGPU, false, false, false, false);
}

#define MAXTAGS 1

static std::vector<int> PMEStorageSizes(ML_END_INVALID * PME_ID_END_INVALID * MAXTAGS);
static std::vector<void *> PMEStoragePointers(ML_END_INVALID * PME_ID_END_INVALID * MAXTAGS);

static bool debugMemoryPrint = false;

template <typename T>
T *PMEFetch(PMEDataID id, int unusedTag, int size, MemLocType location)
{
    //yupinov grid resize mistake!
    assert(unusedTag == 0);
    hipError_t stat;
    int i = (location * PME_ID_END_INVALID + id) * MAXTAGS + unusedTag;

    if ((PMEStorageSizes[i] > 0) && (size > 0) && (size > PMEStorageSizes[i]))
        printf("asked to realloc %d into %d with ID %d\n", PMEStorageSizes[i], size, id);

    if (PMEStorageSizes[i] < size || size == 0) //delete
    {
        if (PMEStoragePointers[i])
        {
            if (debugMemoryPrint)
                fprintf(stderr, "free! %p\n", PMEStoragePointers[i]);
            if (location == ML_DEVICE)
            {
                stat = hipFree(PMEStoragePointers[i]);
                CU_RET_ERR(stat, "PME hipFree error");
            }
            else
            {
                delete[] (T *) PMEStoragePointers[i];
            }
            PMEStoragePointers[i] = NULL;
        }
        if (size > 0)
        {
            if (debugMemoryPrint)
                printf("asked to alloc %d", size);
            size = size * 1.02; //yupinov overalloc
            if (debugMemoryPrint)
                printf(", actually allocating %d\n", size);
            if (location == ML_DEVICE)
            {
                stat = hipMalloc((void **) &PMEStoragePointers[i], size);
                CU_RET_ERR(stat, "PME hipMalloc error");
            }
            else
            {
                PMEStoragePointers[i] = new T[size / sizeof(T)]; //yupinov cudaHostMalloc?
            }
            PMEStorageSizes[i] = size;
        }
    }
    return (T *) PMEStoragePointers[i];
}

real *PMEFetchRealArray(PMEDataID id, int unusedTag, int size, MemLocType location)
{
    return PMEFetch<real>(id, unusedTag, size, location);
}

t_complex *PMEFetchComplexArray(PMEDataID id, int unusedTag, int size, MemLocType location)
{
    return PMEFetch<t_complex>(id, unusedTag, size, location);
}

int *PMEFetchIntegerArray(PMEDataID id, int unusedTag, int size, MemLocType location)
{
    return PMEFetch<int>(id, unusedTag, size, location);
}

template <typename T>
T *PMEFetchAndCopy(PMEDataID id, int unusedTag, void *src, int size, MemLocType location, hipStream_t s)
{
    T *result = PMEFetch<T>(id, unusedTag, size, location);
    PMECopy(result, src, size, location, s);
    return result;
}

t_complex *PMEFetchAndCopyComplexArray(PMEDataID id, int unusedTag, void *src, int size, MemLocType location, hipStream_t s)
{
    return PMEFetchAndCopy<t_complex>(id, unusedTag, src, size, location, s);
}

real *PMEFetchAndCopyRealArray(PMEDataID id, int unusedTag, void *src, int size, MemLocType location, hipStream_t s)
{
    return PMEFetchAndCopy<real>(id, unusedTag, src, size, location, s);
}

int *PMEFetchAndCopyIntegerArray(PMEDataID id, int unusedTag, void *src, int size, MemLocType location, hipStream_t s)
{
    return PMEFetchAndCopy<int>(id, unusedTag, src, size, location, s);
}

void PMECopy(void *dest, void *src, int size, MemLocType destination, hipStream_t s) //yupinov move everything onto this function - or not
{
    if (destination == ML_DEVICE)
    {
        //hipError_t stat = hipMemcpy(dest, src, size, hipMemcpyHostToDevice);
        hipError_t stat = hipMemcpyAsync(dest, src, size, hipMemcpyHostToDevice, s);
        CU_RET_ERR(stat, "PME hipMemcpyHostToDevice error");
    }
    else
    {
        //hipError_t stat = hipMemcpy(dest, src, size, hipMemcpyDeviceToHost);
        hipError_t stat = hipMemcpyAsync(dest, src, size, hipMemcpyDeviceToHost, s);
        CU_RET_ERR(stat, "PME hipMemcpyDeviceToHost error");
    }
}

