#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <vector>

#include "gromacs/gpu_utils/cudautils.cuh"

#include "gromacs/utility/smalloc.h"

// for GPU init
#include "gromacs/gpu_utils/gpu_utils.h"
#include "gromacs/utility/cstringutil.h"
#include "gromacs/hardware/hw_info.h"
#include "gromacs/utility/logger.h"

#include "pme-cuda.cuh"
#include "pme-gpu.h"

void pme_gpu_update_flags(
        gmx_pme_gpu_t *pmeGPU,
        gmx_bool keepGPUDataBetweenSpreadAndR2C,
        gmx_bool keepGPUDataBetweenR2CAndSolve,
        gmx_bool keepGPUDataBetweenSolveAndC2R,
        gmx_bool keepGPUDataBetweenC2RAndGather
        )
{
    pmeGPU->keepGPUDataBetweenSpreadAndR2C = keepGPUDataBetweenSpreadAndR2C;
    pmeGPU->keepGPUDataBetweenR2CAndSolve = keepGPUDataBetweenR2CAndSolve;
    pmeGPU->keepGPUDataBetweenSolveAndC2R = keepGPUDataBetweenSolveAndC2R;
    pmeGPU->keepGPUDataBetweenC2RAndGather = keepGPUDataBetweenC2RAndGather;
}

void pme_gpu_step_reinit(gmx_pme_t *pme)
{
    // this is ran at the end of MD step + at the DD init
    const int grid_index = 0; //!
    pme_gpu_clear_grid(pme, grid_index);
    pme_gpu_clear_energy_virial(pme, grid_index);
}

void pme_gpu_init(gmx_pme_gpu_t **pmeGPU, gmx_pme_t *pme, const gmx_hw_info_t *hwinfo,
                  const gmx_gpu_opt_t *gpu_opt)
{
    // this is ran in the beginning/on DD
    if (!pme->bGPU) //yupinov fix this
        return;

    gmx_bool firstInit = !*pmeGPU;
    if (firstInit) // first init
    {
        *pmeGPU = new gmx_pme_gpu_t;
        hipError_t stat;

        // GPU selection copied from non-bondeds
        const int PMEGPURank = pme->nodeid;
        char gpu_err_str[STRLEN];
        assert(hwinfo->gpu_info.gpu_dev);
        assert(gpu_opt->dev_use);
        (*pmeGPU)->deviceInfo = &hwinfo->gpu_info.gpu_dev[gpu_opt->dev_use[PMEGPURank]];
        const gmx::MDLogger temp;
        if (!init_gpu(temp, PMEGPURank, gpu_err_str, &hwinfo->gpu_info, gpu_opt))
            gmx_fatal(FARGS, "Could not select GPU %d for PME rank %d\n", (*pmeGPU)->deviceInfo->id, PMEGPURank);
        // fallback instead?
        // first init and either of the hw structures NULL => should also fall back to CPU

        // permanent settings

        (*pmeGPU)->doOutOfPlaceFFT = true;
        // this should give better performance, according to the cuFFT documentation
        // performance seems to be the same though
        // perhaps the limiting factor is using paddings/overlaps in the grid, which is also frowned upon
        // PME should also try to pick up nice grid sizes (with factors of 2, 3, 5, 7)

        (*pmeGPU)->doTime = (getenv("GMX_DISABLE_CUDA_TIMING") == NULL);
        // this should check for PP GPU being launched
        // just like NB should check for PME GPU

        (*pmeGPU)->useTextureObjects = ((*pmeGPU)->deviceInfo->prop.major >= 3);
        // if false, texture references are used instead

        // internal storage
        size_t pointerStorageSize = ML_END_INVALID * PME_ID_END_INVALID;
        (*pmeGPU)->StorageSizes.assign(pointerStorageSize, 0);
        (*pmeGPU)->StoragePointers.assign(pointerStorageSize, NULL);

        // creating a PME stream
#if GMX_CUDA_VERSION >= 5050
        int highest_priority;
        int lowest_priority;
        stat = hipDeviceGetStreamPriorityRange(&lowest_priority, &highest_priority);
        CU_RET_ERR(stat, "PME hipDeviceGetStreamPriorityRange failed");
        stat = hipStreamCreateWithPriority(&(*pmeGPU)->pmeStream,
                                                //hipStreamNonBlocking,
                                                hipStreamDefault,
                                                highest_priority);

        CU_RET_ERR(stat, "hipStreamCreateWithPriority on PME stream failed");
#else
        stat = hipStreamCreate(&(*pmeGPU)->pmeStream);
        CU_RET_ERR(stat, "PME hipStreamCreate error");
#endif
        // creating synchronization events
        stat = hipEventCreateWithFlags(&(*pmeGPU)->syncEnerVirH2D, hipEventDisableTiming);
        CU_RET_ERR(stat, "hipEventCreate on syncEnerVirH2D failed");
        stat = hipEventCreateWithFlags(&(*pmeGPU)->syncForcesH2D, hipEventDisableTiming);
        CU_RET_ERR(stat, "hipEventCreate on syncForcesH2D failed");

        if ((pme->gpu)->doTime)
            pme_gpu_init_timings(pme);

        pme_gpu_update_flags(*pmeGPU, false, false, false, false);
    }

    // all these functions should only be called when the grid size changes (e.g. DD)
    const int grid_index = 0;
    pme_gpu_copy_wrap_zones(pme);
    pme_gpu_copy_calcspline_constants(pme);
    pme_gpu_copy_bspline_moduli(pme);
    pme_gpu_alloc_gather_forces(pme);
    pme_gpu_alloc_grids(pme, grid_index);
    pme_gpu_alloc_energy_virial(pme, grid_index);

    if (pme->bGPUFFT) //copied from gmx_pme_init
    {
        ivec ndata;
        ndata[0]    = pme->nkx;
        ndata[1]    = pme->nky;
        ndata[2]    = pme->nkz;
        snew((*pmeGPU)->pfft_setup_gpu, pme->ngrids);
        for (int i = 0; i < pme->ngrids; ++i)
        {
            gmx_parallel_3dfft_init_gpu(&(*pmeGPU)->pfft_setup_gpu[i], ndata, pme);
        }
    }

    pme_gpu_step_reinit(pme);

    if (debug)
        fprintf(debug, "PME GPU %s\n", firstInit ? "init" : "reinit");
}

void pme_gpu_deinit(//gmx_pme_gpu_t **pmeGPU,
                    gmx_pme_t **pme)
{
    // this is ran at the end of run

    if (!(*pme)->bGPU) // we're assuming this boolean doesn't change during the run
        return;

    stopGpuProfiler();

    hipError_t stat;

    // these are all the GPU/host pointers allocated through PMEMemoryFetch - grids included
    // it's a temporary cleanup solution
    for (unsigned int id = 0; id < PME_ID_END_INVALID; id++)
        for (unsigned int location = 0; location < ML_END_INVALID; location++)
        {
            PMEMemoryFree(*pme, (PMEDataID)id, (MemLocType)location);
        }

    // FFT
    for (int i = 0; i < (*pme)->ngrids; i++)
        gmx_parallel_3dfft_destroy_gpu((*pme)->gpu->pfft_setup_gpu[i]);
    sfree((*pme)->gpu->pfft_setup_gpu);

    // destroy synchronization events
    stat = hipEventDestroy((*pme)->gpu->syncEnerVirH2D);
    CU_RET_ERR(stat, "hipEventDestroy failed on syncEnerVirH2D");
    stat = hipEventDestroy((*pme)->gpu->syncForcesH2D);
    CU_RET_ERR(stat, "hipEventDestroy failed on syncForcesH2D");

    // destroy the stream
    stat = hipStreamDestroy((*pme)->gpu->pmeStream);
    CU_RET_ERR(stat, "PME hipStreamDestroy error");

    // delete the structure itself
    delete ((*pme)->gpu);
    (*pme)->gpu = NULL;
}


void pme_gpu_step_init(gmx_pme_t *pme)
{
    // this is ran at the beginning of MD step
    // should ideally be empty
    if (!pme->bGPU)
        return;

    pme_gpu_copy_recipbox(pme); //yupinov test changing box

    pme_gpu_copy_coordinates(pme);
}

void pme_gpu_step_end(gmx_pme_t *pme, const gmx_bool bCalcF, const gmx_bool bCalcEnerVir)
{
    // this is ran at the end of MD step
    if (!pme->bGPU)
        return;

    hipError_t stat = hipStreamSynchronize(pme->gpu->pmeStream); // needed for timings and for copy back events
    CU_RET_ERR(stat, "failed to synchronize the PME GPU stream!");

    if (bCalcF)
        pme_gpu_get_forces(pme);
    if (bCalcEnerVir)
        pme_gpu_get_energy_virial(pme);

    pme_gpu_update_timings(pme);

    pme_gpu_get_timings(pme); // no need to call every step

    pme_gpu_step_reinit(pme);
}

#if PME_EXTERN_CMEM
__constant__ __device__ int2 OVERLAP_SIZES[OVERLAP_ZONES];
__constant__ __device__ int OVERLAP_CELLS_COUNTS[OVERLAP_ZONES];
__constant__ __device__ float3 RECIPBOX[3];
#endif

//yupinov stuff more data into constants, like ewaldcoef, etc?

void pme_gpu_copy_recipbox(gmx_pme_t *pme)
{
    const float3 box[3] =
    {
        {pme->recipbox[XX][XX], pme->recipbox[YY][XX], pme->recipbox[ZZ][XX]},
        {                  0.0, pme->recipbox[YY][YY], pme->recipbox[ZZ][YY]},
        {                  0.0,                   0.0, pme->recipbox[ZZ][ZZ]}
    };
    assert(pme->recipbox[XX][XX] != 0.0);
#if PME_EXTERN_CMEM
    PMECopyConstant(RECIPBOX, box, sizeof(box), s);
#else
    memcpy(pme->gpu->recipbox.box, box, sizeof(box));
#endif
}

void pme_gpu_copy_coordinates(gmx_pme_t *pme)
{
    const int n = pme->atc[0].n;

    // coordinates
    const size_t coordinatesSize = DIM * n * sizeof(real);
    float3 *coordinates_h = (float3 *)PMEMemoryFetch(pme, PME_ID_XPTR, coordinatesSize, ML_HOST);
    memcpy(coordinates_h, pme->atc[0].x, coordinatesSize);
    pme->gpu->coordinates = (float3 *)PMEMemoryFetch(pme, PME_ID_XPTR, coordinatesSize, ML_DEVICE);
    cu_copy_H2D_async(pme->gpu->coordinates, coordinates_h, coordinatesSize, pme->gpu->pmeStream);
    /*
    float4 *xptr_h = (float4 *)PMEMemoryFetch(pme, PME_ID_XPTR, 4 * n_blocked * sizeof(real), ML_HOST);
    memset(xptr_h, 0, 4 * n_blocked * sizeof(real));
    for (int i = 0; i < n; i++)
    {
       memcpy(xptr_h + i, atc->x + i, sizeof(rvec));
    }
    xptr_d = (float4 *)PMEMemoryFetch(pme, PME_ID_XPTR, 4 * n_blocked * sizeof(real), ML_DEVICE);
    PMECopy(pme->gpu->coordinates, xptr_h, 4 * n_blocked * sizeof(real), ML_DEVICE, pme->gpu->pmeStream);
    */
}

void pme_gpu_copy_charges(gmx_pme_t *pme)
{
    const int n = pme->atc[0].n;
    // coefficients - can be different for PME/LJ?
    const size_t coefficientSize = n * sizeof(real);
    real *coefficients_h = (real *)PMEMemoryFetch(pme, PME_ID_COEFFICIENT, coefficientSize, ML_HOST);
    memcpy(coefficients_h, pme->atc[0].coefficient, coefficientSize); // why not just register host memory?
    pme->gpu->coefficients = (real *)PMEMemoryFetch(pme, PME_ID_COEFFICIENT, coefficientSize, ML_DEVICE);
    cu_copy_H2D_async(pme->gpu->coefficients, coefficients_h, coefficientSize, pme->gpu->pmeStream);
}

void pme_gpu_copy_wrap_zones(gmx_pme_t *pme)
{
    const int nx = pme->nkx;
    const int ny = pme->nky;
    const int nz = pme->nkz;
    const int overlap = pme->pme_order - 1;

    // cell count in 7 parts of overlap
    const int3 zoneSizes_h[OVERLAP_ZONES] =
    {
        {     nx,        ny,   overlap},
        {     nx,   overlap,        nz},
        {overlap,        ny,        nz},
        {     nx,   overlap,   overlap},
        {overlap,        ny,   overlap},
        {overlap,   overlap,        nz},
        {overlap,   overlap,   overlap}
    };

    const int2 zoneSizesYZ_h[OVERLAP_ZONES] =
    {
        {     ny,   overlap},
        {overlap,        nz},
        {     ny,        nz},
        {overlap,   overlap},
        {     ny,   overlap},
        {overlap,        nz},
        {overlap,   overlap}
    };

    int cellsAccumCount_h[OVERLAP_ZONES];
    for (int i = 0; i < OVERLAP_ZONES; i++)
        cellsAccumCount_h[i] = zoneSizes_h[i].x * zoneSizes_h[i].y * zoneSizes_h[i].z;
    // accumulate
    for (int i = 1; i < OVERLAP_ZONES; i++)
    {
        cellsAccumCount_h[i] = cellsAccumCount_h[i] + cellsAccumCount_h[i - 1];
    }
#if PME_EXTERN_CMEM
    PMECopyConstant(OVERLAP_SIZES, zoneSizesYZ_h, sizeof(zoneSizesYZ_h), s);
    PMECopyConstant(OVERLAP_CELLS_COUNTS, cellsAccumCount_h, sizeof(cellsAccumCount_h), s);
#else
    memcpy(pme->gpu->overlap.overlapSizes, zoneSizesYZ_h, sizeof(zoneSizesYZ_h));
    memcpy(pme->gpu->overlap.overlapCellCounts, cellsAccumCount_h, sizeof(cellsAccumCount_h));
#endif
}

static gmx_bool debugMemoryPrint = false;

void PMEMemoryFree(gmx_pme_t *pme, PMEDataID id, MemLocType location)
{
    hipError_t stat;
    size_t i = location * PME_ID_END_INVALID + id;
    if (pme->gpu->StoragePointers[i])
    {
        if (debugMemoryPrint)
            printf("free! %p %d %d\n", pme->gpu->StoragePointers[i], id, location);
        if (location == ML_DEVICE)
        {
            stat = hipFree(pme->gpu->StoragePointers[i]);
            CU_RET_ERR(stat, "PME hipFree error");
        }
        else
        {
            stat = hipHostFree(pme->gpu->StoragePointers[i]);
            CU_RET_ERR(stat, "PME hipHostFree error");
        }
        pme->gpu->StoragePointers[i] = NULL;
    }
}

void *PMEMemoryFetch(gmx_pme_t *pme, PMEDataID id, size_t size, MemLocType location)
{
    // size == 0 => just return a current pointer

    assert(pme->gpu);
    hipError_t stat = hipSuccess;
    size_t i = location * PME_ID_END_INVALID + id;

    if ((pme->gpu->StorageSizes[i] > 0) && (size > 0) && (size > pme->gpu->StorageSizes[i]))
        printf("asked to realloc %lu into %lu with ID %d\n", pme->gpu->StorageSizes[i], size, id);

    if (pme->gpu->StorageSizes[i] < size) // delete
    {
        PMEMemoryFree(pme, id, location);
        if (size > 0)
        {
            if (debugMemoryPrint)
                printf("asked to alloc %lu", size);
            size = size * 1.02; // slight overalloc for no apparent reason
            if (debugMemoryPrint)
                printf(", actually allocating %lu\n", size);
            if (location == ML_DEVICE)
            {
                stat = hipMalloc((void **)&pme->gpu->StoragePointers[i], size);
                CU_RET_ERR(stat, "PME hipMalloc error");
            }
            else
            {
                unsigned int allocFlags = hipHostMallocDefault;
                //allocFlags |= hipHostMallocWriteCombined;
                //yupinov try hipHostMallocWriteCombined for almost-constant global memory? do I even have that?
                // yes, I do: coordinates/coefficients and thetas/dthetas. should be helpful for spread being overwhelmed by L2 cache!
                stat = hipHostAlloc((void **)&pme->gpu->StoragePointers[i], size, allocFlags);
                CU_RET_ERR(stat, "PME hipHostAlloc error");
            }
            pme->gpu->StorageSizes[i] = size;
        }
    }
    return pme->gpu->StoragePointers[i];
}

void PMECopyConstant(const void *dest, void const *src, size_t size, hipStream_t s)
{
    assert(s != 0);
    hipError_t stat = hipMemcpyToSymbolAsync(HIP_SYMBOL(dest), src, size, 0, hipMemcpyHostToDevice, s);
    CU_RET_ERR(stat, "PME hipMemcpyToSymbolAsync");
}

