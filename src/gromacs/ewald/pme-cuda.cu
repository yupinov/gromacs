#include "hip/hip_runtime.h"
#include <vector>
#include <stdio.h>
#include "gromacs/gpu_utils/cudautils.cuh"

#include "pme-cuda.cuh"

#include <assert.h>

void pme_gpu_update_flags(
        gmx_pme_gpu_t *pmeGPU,
        gmx_bool keepGPUDataBetweenSpreadAndR2C,
        gmx_bool keepGPUDataBetweenR2CAndSolve,
        gmx_bool keepGPUDataBetweenSolveAndC2R,
        gmx_bool keepGPUDataBetweenC2RAndGather
        )
{
    pmeGPU->keepGPUDataBetweenSpreadAndR2C = keepGPUDataBetweenSpreadAndR2C;
    pmeGPU->keepGPUDataBetweenR2CAndSolve = keepGPUDataBetweenR2CAndSolve;
    pmeGPU->keepGPUDataBetweenSolveAndC2R = keepGPUDataBetweenSolveAndC2R;
    pmeGPU->keepGPUDataBetweenC2RAndGather = keepGPUDataBetweenC2RAndGather;
}

void pme_gpu_init(gmx_pme_gpu_t **pmeGPU)
{
    gmx_bool firstInit = !*pmeGPU;
    if (firstInit) // first init
    {
        *pmeGPU = new gmx_pme_gpu_t;
        hipError_t stat;
    //yupinov dealloc@

        // creating a PME stream
    #if GMX_CUDA_VERSION >= 5050
        int highest_priority;
        int lowest_priority;
        stat = hipDeviceGetStreamPriorityRange(&lowest_priority, &highest_priority);
        CU_RET_ERR(stat, "PME hipDeviceGetStreamPriorityRange failed");
        stat = hipStreamCreateWithPriority(&(*pmeGPU)->pmeStream,
                                                //hipStreamNonBlocking,
                                                hipStreamDefault,
                                                highest_priority);
        //yupinov: fighting with nbnxn non-local for highest priority - check on MPI!
        CU_RET_ERR(stat, "hipStreamCreateWithPriority on PME stream failed");
    #else
        stat = hipStreamCreate(&(*pme)->pmeStream);
        CU_RET_ERR(stat, "PME hipStreamCreate error");
    #endif

        // creating synchronization events
        stat = hipEventCreateWithFlags(&(*pmeGPU)->syncEnerVirH2D, hipEventDisableTiming);
        CU_RET_ERR(stat, "hipEventCreate on syncEnerVirH2D failed");
        stat = hipEventCreateWithFlags(&(*pmeGPU)->syncForcesH2D, hipEventDisableTiming);
        CU_RET_ERR(stat, "hipEventCreate on syncForcesH2D failed");
        //yupinov again dealloc
        /*
        stat = hipEventDestroy(nb->nonlocal_done);
        CU_RET_ERR(stat, "hipEventDestroy failed on timers->nonlocal_done");
        stat = hipEventDestroy(nb->misc_ops_and_local_H2D_done);
        CU_RET_ERR(stat, "hipEventDestroy failed on timers->misc_ops_and_local_H2D_done");
        */

        pme_gpu_update_flags(*pmeGPU, false, false, false, false);
    }

    if (debug)
        fprintf(debug, "PME GPU %s\n", firstInit ? "init" : "reinit");
}
#if PME_EXTERN_CMEM
__constant__ __device__ int2 OVERLAP_SIZES[OVERLAP_ZONES];
__constant__ __device__ int OVERLAP_CELLS_COUNTS[OVERLAP_ZONES];
__constant__ __device__ float3 RECIPBOX[3];

void pme_gpu_copy_recipbox(gmx_pme_t *pme)
{
    hipStream_t s = pme->gpu->pmeStream;
    const float3 recipbox_h[3] =
    {
        {pme->recipbox[XX][XX], pme->recipbox[YY][XX], pme->recipbox[ZZ][XX]},
        {                  0.0, pme->recipbox[YY][YY], pme->recipbox[ZZ][YY]},
        {                  0.0,                   0.0, pme->recipbox[ZZ][ZZ]}
    };
    /*
    void *testing;
    hipError_t stat = hipGetSymbolAddress(&testing, HIP_SYMBOL(RECIPBOX));
    CU_RET_ERR(stat, "stat");
    printf("copying %g to %p\n", pme->recipbox[0][0], testing);
    */

    PMECopyConstant(RECIPBOX, recipbox_h, sizeof(recipbox_h), s);
}

void pme_gpu_copy_overlap_zones(gmx_pme_t *pme)
{
    const int nx = pme->nkx;
    const int ny = pme->nky;
    const int nz = pme->nkz;

    // cell count in 7 parts of overlap
    const int3 zoneSizes_h[OVERLAP_ZONES] =
    {
        {     nx,        ny,   overlap},
        {     nx,   overlap,        nz},
        {overlap,        ny,        nz},
        {     nx,   overlap,   overlap},
        {overlap,        ny,   overlap},
        {overlap,   overlap,        nz},
        {overlap,   overlap,   overlap}
    };

    const int2 zoneSizesYZ_h[OVERLAP_ZONES] =
    {
        {     ny,   overlap},
        {overlap,        nz},
        {     ny,        nz},
        {overlap,   overlap},
        {     ny,   overlap},
        {overlap,        nz},
        {overlap,   overlap}
    };

    int cellsAccumCount_h[OVERLAP_ZONES];
    for (int i = 0; i < OVERLAP_ZONES; i++)
        cellsAccumCount_h[i] = zoneSizes_h[i].x * zoneSizes_h[i].y * zoneSizes_h[i].z;
    // accumulate
    for (int i = 1; i < OVERLAP_ZONES; i++)
    {
        cellsAccumCount_h[i] = cellsAccumCount_h[i] + cellsAccumCount_h[i - 1];
    }

    PMECopyConstant(OVERLAP_SIZES, zoneSizesYZ_h, sizeof(zoneSizesYZ_h), s);
    PMECopyConstant(OVERLAP_CELLS_COUNTS, cellsAccumCount_h, sizeof(cellsAccumCount_h), s);
    //other constants
}

#else

void pme_gpu_copy_recipbox(gmx_pme_t *pme)
{
    const float3 box[3] =
    {
        {pme->recipbox[XX][XX], pme->recipbox[YY][XX], pme->recipbox[ZZ][XX]},
        {                  0.0, pme->recipbox[YY][YY], pme->recipbox[ZZ][YY]},
        {                  0.0,                   0.0, pme->recipbox[ZZ][ZZ]}
    };

    memcpy(pme->gpu->recipbox.box, box, sizeof(box));
}

void pme_gpu_copy_overlap_zones(gmx_pme_t *pme)
{
    const int nx = pme->nkx;
    const int ny = pme->nky;
    const int nz = pme->nkz;
    const int overlap = pme->pme_order - 1;

    // cell count in 7 parts of overlap
    const int3 zoneSizes_h[OVERLAP_ZONES] =
    {
        {     nx,        ny,   overlap},
        {     nx,   overlap,        nz},
        {overlap,        ny,        nz},
        {     nx,   overlap,   overlap},
        {overlap,        ny,   overlap},
        {overlap,   overlap,        nz},
        {overlap,   overlap,   overlap}
    };

    const int2 zoneSizesYZ_h[OVERLAP_ZONES] =
    {
        {     ny,   overlap},
        {overlap,        nz},
        {     ny,        nz},
        {overlap,   overlap},
        {     ny,   overlap},
        {overlap,        nz},
        {overlap,   overlap}
    };

    memcpy(pme->gpu->overlap.overlapSizes, zoneSizesYZ_h, sizeof(zoneSizesYZ_h));

    int cellsAccumCount_h[OVERLAP_ZONES];
    for (int i = 0; i < OVERLAP_ZONES; i++)
        cellsAccumCount_h[i] = zoneSizes_h[i].x * zoneSizes_h[i].y * zoneSizes_h[i].z;
    // accumulate
    for (int i = 1; i < OVERLAP_ZONES; i++)
    {
        cellsAccumCount_h[i] = cellsAccumCount_h[i] + cellsAccumCount_h[i - 1];
    }

    memcpy(pme->gpu->overlap.overlapCellCounts, cellsAccumCount_h, sizeof(cellsAccumCount_h));
}

#endif



#define MAXTAGS 1

static std::vector<int> PMEStorageSizes(ML_END_INVALID * PME_ID_END_INVALID * MAXTAGS);
static std::vector<void *> PMEStoragePointers(ML_END_INVALID * PME_ID_END_INVALID * MAXTAGS);

static bool debugMemoryPrint = false;

template <typename T>
T *PMEFetch(PMEDataID id, int unusedTag, int size, MemLocType location)
{
    //yupinov grid resize mistake!
    assert(unusedTag == 0);
    hipError_t stat;
    int i = (location * PME_ID_END_INVALID + id) * MAXTAGS + unusedTag;

    if ((PMEStorageSizes[i] > 0) && (size > 0) && (size > PMEStorageSizes[i]))
        printf("asked to realloc %d into %d with ID %d\n", PMEStorageSizes[i], size, id);

    if (PMEStorageSizes[i] < size || size == 0) //delete
    {
        if (PMEStoragePointers[i])
        {
            if (debugMemoryPrint)
                fprintf(stderr, "free! %p\n", PMEStoragePointers[i]);
            if (location == ML_DEVICE)
            {
                stat = hipFree(PMEStoragePointers[i]);
                CU_RET_ERR(stat, "PME hipFree error");
            }
            else
            {
                delete[] (T *) PMEStoragePointers[i];
            }
            PMEStoragePointers[i] = NULL;
        }
        if (size > 0)
        {
            if (debugMemoryPrint)
                printf("asked to alloc %d", size);
            size = size * 1.02; //yupinov overalloc
            if (debugMemoryPrint)
                printf(", actually allocating %d\n", size);
            if (location == ML_DEVICE)
            {
                stat = hipMalloc((void **) &PMEStoragePointers[i], size);
                CU_RET_ERR(stat, "PME hipMalloc error");
            }
            else
            {
                PMEStoragePointers[i] = new T[size / sizeof(T)]; //yupinov cudaHostMalloc?
            }
            PMEStorageSizes[i] = size;
        }
    }
    return (T *) PMEStoragePointers[i];
}

real *PMEFetchRealArray(PMEDataID id, int unusedTag, int size, MemLocType location)
{
    return PMEFetch<real>(id, unusedTag, size, location);
}

t_complex *PMEFetchComplexArray(PMEDataID id, int unusedTag, int size, MemLocType location)
{
    return PMEFetch<t_complex>(id, unusedTag, size, location);
}

int *PMEFetchIntegerArray(PMEDataID id, int unusedTag, int size, MemLocType location)
{
    return PMEFetch<int>(id, unusedTag, size, location);
}

template <typename T>
T *PMEFetchAndCopy(PMEDataID id, int unusedTag, void *src, int size, MemLocType location, hipStream_t s, gmx_bool sync = false)
{
    T *result = PMEFetch<T>(id, unusedTag, size, location);
    PMECopy(result, src, size, location, s, sync);
    return result;
}

t_complex *PMEFetchAndCopyComplexArray(PMEDataID id, int unusedTag, void *src, int size, MemLocType location, hipStream_t s)
{
    return PMEFetchAndCopy<t_complex>(id, unusedTag, src, size, location, s);
}

real *PMEFetchAndCopyRealArray(PMEDataID id, int unusedTag, void *src, int size, MemLocType location, hipStream_t s, gmx_bool sync)
{
    return PMEFetchAndCopy<real>(id, unusedTag, src, size, location, s, sync);
}

int *PMEFetchAndCopyIntegerArray(PMEDataID id, int unusedTag, void *src, int size, MemLocType location, hipStream_t s)
{
    return PMEFetchAndCopy<int>(id, unusedTag, src, size, location, s);
}

void PMECopy(void *dest, void *src, int size, MemLocType destination, hipStream_t s, gmx_bool sync) //yupinov move everything onto this function - or not
{
    // synchronous copies are not used anywhere currently, I think
    assert(s != 0);
    hipError_t stat;
    if (destination == ML_DEVICE)
    {
        if (sync)
            stat = hipMemcpy(dest, src, size, hipMemcpyHostToDevice);
        else
            stat = hipMemcpyAsync(dest, src, size, hipMemcpyHostToDevice, s);
        CU_RET_ERR(stat, "PME hipMemcpyHostToDevice error");
    }
    else
    {
        if (sync)
            stat = hipMemcpy(dest, src, size, hipMemcpyDeviceToHost);
        else
            stat = hipMemcpyAsync(dest, src, size, hipMemcpyDeviceToHost, s);
        CU_RET_ERR(stat, "PME hipMemcpyDeviceToHost error");
    }
}

void PMECopyConstant(const void *dest, void const *src, size_t size, hipStream_t s)
{
    assert(s != 0);
    hipError_t stat = hipMemcpyToSymbolAsync(HIP_SYMBOL(dest), src, size, 0, hipMemcpyHostToDevice, s);
    CU_RET_ERR(stat, "PME hipMemcpyToSymbolAsync");
}

int PMEGetAllocatedSize(PMEDataID id, int unusedTag, MemLocType location)
{
    int i = (location * PME_ID_END_INVALID + id) * MAXTAGS + unusedTag;
    return PMEStorageSizes[i];
}

