#include <vector>
#include <stdio.h>
#include "gromacs/gpu_utils/cudautils.cuh"

#include "pme-cuda.cuh"

#include <assert.h>

void pme_gpu_update_flags(
        gmx_pme_gpu_t *pmeGPU,
        gmx_bool keepGPUDataBetweenSpreadAndR2C,
        gmx_bool keepGPUDataBetweenR2CAndSolve,
        gmx_bool keepGPUDataBetweenSolveAndC2R,
        gmx_bool keepGPUDataBetweenC2RAndGather
        )
{
    pmeGPU->keepGPUDataBetweenSpreadAndR2C = keepGPUDataBetweenSpreadAndR2C;
    pmeGPU->keepGPUDataBetweenR2CAndSolve = keepGPUDataBetweenR2CAndSolve;
    pmeGPU->keepGPUDataBetweenSolveAndC2R = keepGPUDataBetweenSolveAndC2R;
    pmeGPU->keepGPUDataBetweenC2RAndGather = keepGPUDataBetweenC2RAndGather;
}

void pme_gpu_init(gmx_pme_gpu_t **pmeGPU)
{
    gmx_bool firstInit = !*pmeGPU;
    if (firstInit) // first init
    {
        *pmeGPU = new gmx_pme_gpu_t;
        hipError_t stat;
    //yupinov dealloc@

        // creating a PME stream
    #if GMX_CUDA_VERSION >= 5050
        int highest_priority;
        int lowest_priority;
        stat = hipDeviceGetStreamPriorityRange(&lowest_priority, &highest_priority);
        CU_RET_ERR(stat, "PME hipDeviceGetStreamPriorityRange failed");
        stat = hipStreamCreateWithPriority(&(*pmeGPU)->pmeStream,
                                                //hipStreamNonBlocking,
                                                hipStreamDefault,
                                                highest_priority);
        //yupinov: fighting with nbnxn non-local for highest priority - check on MPI!
        CU_RET_ERR(stat, "hipStreamCreateWithPriority on PME stream failed");
    #else
        stat = hipStreamCreate(&(*pme)->pmeStream);
        CU_RET_ERR(stat, "PME hipStreamCreate error");
    #endif

        // creating synchronization events
        stat = hipEventCreateWithFlags(&(*pmeGPU)->syncEnerVirH2D, hipEventDisableTiming);
        CU_RET_ERR(stat, "hipEventCreate on syncEnerVirH2D failed");
        stat = hipEventCreateWithFlags(&(*pmeGPU)->syncForcesH2D, hipEventDisableTiming);
        CU_RET_ERR(stat, "hipEventCreate on syncForcesH2D failed");
        //yupinov again dealloc
        /*
        stat = hipEventDestroy(nb->nonlocal_done);
        CU_RET_ERR(stat, "hipEventDestroy failed on timers->nonlocal_done");
        stat = hipEventDestroy(nb->misc_ops_and_local_H2D_done);
        CU_RET_ERR(stat, "hipEventDestroy failed on timers->misc_ops_and_local_H2D_done");
        */

        pme_gpu_update_flags(*pmeGPU, false, false, false, false);
    }
    if (debug)
        fprintf(debug, "PME GPU %s\n", firstInit ? "init" : "reinit");
}

#define MAXTAGS 1

static std::vector<int> PMEStorageSizes(ML_END_INVALID * PME_ID_END_INVALID * MAXTAGS);
static std::vector<void *> PMEStoragePointers(ML_END_INVALID * PME_ID_END_INVALID * MAXTAGS);

static bool debugMemoryPrint = false;

template <typename T>
T *PMEFetch(PMEDataID id, int unusedTag, int size, MemLocType location)
{
    //yupinov grid resize mistake!
    assert(unusedTag == 0);
    hipError_t stat;
    int i = (location * PME_ID_END_INVALID + id) * MAXTAGS + unusedTag;

    if ((PMEStorageSizes[i] > 0) && (size > 0) && (size > PMEStorageSizes[i]))
        printf("asked to realloc %d into %d with ID %d\n", PMEStorageSizes[i], size, id);

    if (PMEStorageSizes[i] < size || size == 0) //delete
    {
        if (PMEStoragePointers[i])
        {
            if (debugMemoryPrint)
                fprintf(stderr, "free! %p\n", PMEStoragePointers[i]);
            if (location == ML_DEVICE)
            {
                stat = hipFree(PMEStoragePointers[i]);
                CU_RET_ERR(stat, "PME hipFree error");
            }
            else
            {
                delete[] (T *) PMEStoragePointers[i];
            }
            PMEStoragePointers[i] = NULL;
        }
        if (size > 0)
        {
            if (debugMemoryPrint)
                printf("asked to alloc %d", size);
            size = size * 1.02; //yupinov overalloc
            if (debugMemoryPrint)
                printf(", actually allocating %d\n", size);
            if (location == ML_DEVICE)
            {
                stat = hipMalloc((void **) &PMEStoragePointers[i], size);
                CU_RET_ERR(stat, "PME hipMalloc error");
            }
            else
            {
                PMEStoragePointers[i] = new T[size / sizeof(T)]; //yupinov cudaHostMalloc?
            }
            PMEStorageSizes[i] = size;
        }
    }
    return (T *) PMEStoragePointers[i];
}

real *PMEFetchRealArray(PMEDataID id, int unusedTag, int size, MemLocType location)
{
    return PMEFetch<real>(id, unusedTag, size, location);
}

t_complex *PMEFetchComplexArray(PMEDataID id, int unusedTag, int size, MemLocType location)
{
    return PMEFetch<t_complex>(id, unusedTag, size, location);
}

int *PMEFetchIntegerArray(PMEDataID id, int unusedTag, int size, MemLocType location)
{
    return PMEFetch<int>(id, unusedTag, size, location);
}

template <typename T>
T *PMEFetchAndCopy(PMEDataID id, int unusedTag, void *src, int size, MemLocType location, hipStream_t s, gmx_bool sync = false)
{
    T *result = PMEFetch<T>(id, unusedTag, size, location);
    PMECopy(result, src, size, location, s, sync);
    return result;
}

t_complex *PMEFetchAndCopyComplexArray(PMEDataID id, int unusedTag, void *src, int size, MemLocType location, hipStream_t s)
{
    return PMEFetchAndCopy<t_complex>(id, unusedTag, src, size, location, s);
}

real *PMEFetchAndCopyRealArray(PMEDataID id, int unusedTag, void *src, int size, MemLocType location, hipStream_t s, gmx_bool sync)
{
    return PMEFetchAndCopy<real>(id, unusedTag, src, size, location, s, sync);
}

int *PMEFetchAndCopyIntegerArray(PMEDataID id, int unusedTag, void *src, int size, MemLocType location, hipStream_t s)
{
    return PMEFetchAndCopy<int>(id, unusedTag, src, size, location, s);
}

void PMECopy(void *dest, void *src, int size, MemLocType destination, hipStream_t s, gmx_bool sync) //yupinov move everything onto this function - or not
{
    // synchronous copies are not used anywhere currently, I think
    assert(s != 0);
    hipError_t stat;
    if (destination == ML_DEVICE)
    {
        if (sync)
            stat = hipMemcpy(dest, src, size, hipMemcpyHostToDevice);
        else
            stat = hipMemcpyAsync(dest, src, size, hipMemcpyHostToDevice, s);
        CU_RET_ERR(stat, "PME hipMemcpyHostToDevice error");
    }
    else
    {
        if (sync)
            stat = hipMemcpy(dest, src, size, hipMemcpyDeviceToHost);
        else
            stat = hipMemcpyAsync(dest, src, size, hipMemcpyDeviceToHost, s);
        CU_RET_ERR(stat, "PME hipMemcpyDeviceToHost error");
    }
}

void PMECopyConstant(const void *dest, void const *src, size_t size, hipStream_t s)
{
    assert(s != 0);
    hipError_t stat = hipMemcpyToSymbolAsync(HIP_SYMBOL(dest), src, size, 0, hipMemcpyHostToDevice, s);
    CU_RET_ERR(stat, "PME hipMemcpyToSymbolAsync");
}

int PMEGetAllocatedSize(PMEDataID id, int unusedTag, MemLocType location)
{
    int i = (location * PME_ID_END_INVALID + id) * MAXTAGS + unusedTag;
    return PMEStorageSizes[i];
}

