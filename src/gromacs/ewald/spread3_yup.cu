#include "hip/hip_runtime.h"
/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 1991-2000, University of Groningen, The Netherlands.
 * Copyright (c) 2001-2004, The GROMACS development team.
 * Copyright (c) 2013-2015, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */
//yupinov unused file!
#include "pme.h"
#include "pme-internal.h"

#include "gromacs/utility/basedefinitions.h"
#include "gromacs/utility/real.h"
#include "gromacs/math/vectypes.h"
#include "check.h"
#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/gpu_utils/cuda_arch_utils.cuh"
#include <hip/hip_runtime.h>

typedef real *splinevec[DIM];
#ifdef DEBUG_PME_GPU
extern gpu_flags spread_gpu_flags;
extern gpu_flags spread_bunching_gpu_flags;
#endif
#ifdef DEBUG_PME_TIMINGS_GPU
extern gpu_events gpu_events_spread;
#endif
#include "thread_mpi/mutex.h"

#include "th-a.cuh"

static tMPI::mutex print_mutex;



/* This has to be a macro to enable full compiler optimization with xlC (and probably others too) */

#define DO_BSPLINE(order)                                         \
    _Pragma("unroll")                                                    \
    if (globalParticleIndex == 0) \
        printf("%g coefficient\n", coefficient[globalParticleIndex]); \
    for (ithx = 0; (ithx < order); ithx++)                    \
    {                                                             \
        index_x = (i0 + ithx) * pny * pnz;                    \
        valx = coefficient[globalParticleIndex] * thx[ithx];                      \
        if (globalParticleIndex == 0) \
        {    printf("%d index_x\n", index_x); \
         printf("%d %d %d %d\n", i0, ithx, pny, pnz);} \
        _Pragma("unroll")                                                         \
        for (ithy = 0; (ithy < order); ithy++)                \
        {                                                         \
            valxy    = valx*thy[ithy];                       \
            index_xy = index_x+(j0+ithy)*pnz;                 \
             _Pragma("unroll")                                                     \
            for (ithz = 0; (ithz < order); ithz++)            \
            {                                                     \
                index_xyz        = index_xy+(k0+ithz);        \
                /*grid[index_xyz] += valxy*thz[ithz];*/               \
                atomicAdd(grid + index_xyz, valxy*thz[ithz]);      \
                if (globalParticleIndex == 0) \
                    printf("contributing %g to %i\n", valxy*thz[ithz], index_xyz); \
            }                                                     \
        }                                                         \
    }




//template <int order, int N, int K, int D>
// K is particles per block?
template <int order, int particlesPerBlock>
__global__ void spread3_kernel
(int nx, int ny, int nz,
 int start_ix, int start_iy, int start_iz,
 real rxx, real ryx, real ryy, real rzx, real rzy, real rzz,
 //int *g2tx, int *g2ty, int *g2tz,
 real *fshx, real *fshy,
 int *nnx, int *nny, int *nnz,
 real *xptr, real *yptr, real *zptr,
 real *coefficient,
 real *grid,
 int n)
{
/*

    pnx = pmegrid->s[XX];
    pny = pmegrid->s[YY];
    pnz = pmegrid->s[ZZ];

    offx = pmegrid->offset[XX];
    offy = pmegrid->offset[YY];
    offz = pmegrid->offset[ZZ];

*/

    const int offx = 0, offy = 0, offz = 0;
    const int pnx = nx + order - 1, pny = ny + order - 1, pnz = nz + order - 1; //yupinov fix me!

    //const int B = K / D / order / order;

    __shared__ int idxxptr[particlesPerBlock];
    __shared__ int idxyptr[particlesPerBlock];
    __shared__ int idxzptr[particlesPerBlock];
    __shared__ real fxptr[particlesPerBlock];
    __shared__ real fyptr[particlesPerBlock];
    __shared__ real fzptr[particlesPerBlock];

    __shared__ real theta[3 * order * particlesPerBlock];
    __shared__ real dtheta[3 * order * particlesPerBlock];
    //printf("%d %d %d %d %d %d\n", blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z);

    // so I have particlesPerBlock to process with warp_size threads?

    int ithx, index_x, ithy, index_xy, ithz, index_xyz;
    real valx, valxy;


    int localParticleIndex = threadIdx.x;  //yupinov
    int globalParticleIndex = blockIdx.x * particlesPerBlock + localParticleIndex;
    if (globalParticleIndex < n)
    //yupinov - this is a single particle work!
    {
        // INTERPOL_IDX

        /* Fractional coordinates along box vectors, add 2.0 to make 100% sure we are positive for triclinic boxes */
        real tx, ty, tz;
        tx = nx * ( xptr[globalParticleIndex] * rxx + yptr[globalParticleIndex] * ryx + zptr[globalParticleIndex] * rzx + 2.0 );
        ty = ny * (                                   yptr[globalParticleIndex] * ryy + zptr[globalParticleIndex] * rzy + 2.0 );
        tz = nz * (                                                                     zptr[globalParticleIndex] * rzz + 2.0 );

        int tix, tiy, tiz;
        tix = (int)(tx);
        tiy = (int)(ty);
        tiz = (int)(tz);
        if (globalParticleIndex == 0)
        for (int rr = 0; rr < 5 * nx; rr++)
            printf("%d test %d\n", rr, nnx[rr]);
        if (globalParticleIndex == 0)
            printf("%d %d tix\n", tix, nnx[tix]);
        /* Because decomposition only occurs in x and y,
        * we never have a fraction correction in z.
        */

        fxptr[localParticleIndex] = tx - tix + fshx[tix];
        fyptr[localParticleIndex] = ty - tiy + fshy[tiy];
        fzptr[localParticleIndex] = tz - tiz;

        idxxptr[localParticleIndex] = nnx[tix];
        idxyptr[localParticleIndex] = nny[tiy];
        idxzptr[localParticleIndex] = nnz[tiz];

        // CALCSPLINE

        if (coefficient[globalParticleIndex] != 0.0) //yupinov weak
        {
            real dr, div;
            real data[order];

            _Pragma("unroll")
            for (int j = 0; j < DIM; j++)
            {
                //dr  = fractx[i*DIM + j];
                dr = j == 0 ? fxptr[localParticleIndex] : (j == 1 ? fyptr[localParticleIndex] : fzptr[localParticleIndex]);

                /* dr is relative offset from lower cell limit */
                data[order - 1] = 0;
                data[1]         = dr;
                data[0]         = 1 - dr;

                _Pragma("unroll")
                for (int k = 3; k < order; k++)
                {
                    div         = 1.0 / (k - 1.0);
                    data[k - 1] = div * dr * data[k - 2];
                    _Pragma("unroll")
                    for (int l = 1; l < (k - 1); l++)
                    {
                        data[k - l - 1] = div * ((dr + l) * data[k - l - 2] + (k - l - dr) * data[k - l - 1]);
                    }
                    data[0] = div * (1 - dr) * data[0];
                }
                /* differentiate */
                int thetaOffset = (j * particlesPerBlock + localParticleIndex) * order;
                dtheta[thetaOffset] = -data[0];

                _Pragma("unroll")
                for (int k = 1; k < order; k++)
                {
                    dtheta[thetaOffset + k] = data[k - 1] - data[k];
                }

                div             = 1.0 / (order - 1);
                data[order - 1] = div * dr * data[order - 2];
                _Pragma("unroll")
                for (int l = 1; l < (order - 1); l++)
                {
                    data[order - l - 1] = div * ((dr + l) * data[order - l - 2] + (order - l - dr) * data[order - l - 1]);
                }
                data[0] = div * (1 - dr) * data[0];

                _Pragma("unroll")
                for (int k = 0; k < order; k++)
                {
                    theta[thetaOffset + k] = data[k];
                }
            }
        }


        if (coefficient[globalParticleIndex] != 0.0) //yupinov weak
        {
            int i0   = idxxptr[localParticleIndex] - offx; //?
            int j0   = idxyptr[localParticleIndex] - offy;
            int k0   = idxzptr[localParticleIndex] - offz;

            real *thx = theta + (0 * particlesPerBlock + localParticleIndex) * order;
            real *thy = theta + (1 * particlesPerBlock + localParticleIndex) * order;
            real *thz = theta + (2 * particlesPerBlock + localParticleIndex) * order;

           // switch (order)
            DO_BSPLINE(order);
        }
    }
    //yupinov start spread copypaste



      //yupinov end spread copypaste





    /*
#ifdef DEBUG_PME_TIMINGS_GPU
    events_record_stop(gpu_events_calcspline, ewcsPME_CALCSPLINE, 0);
#endif
    */


    //yupinov
    /*
    hipFree(theta_d);
    hipFree(dtheta_d);
    hipFree(fractx_d);
    hipFree(coefficient_d);
    free(fractx_h);
    free(coefficient_h);
    */




    //int block_i0 = blockIdx.x * N;
    //int idxspline_i = (threadIdx.z * order + threadIdx.y) * order + threadIdx.x; // 0 -> 31 -> index of first or second particle contrib....
/*
    for (int block_i = 0; block_i < N; block_i += K) //256, += 32 => each iter is a warp
    {
        //int i = block_i0 + block_i + idxspline_i; //block_i0 + block_i is a particle index
        //if ((blockIdx.x == 1) && (idxspline_i == 1))
        //    printf("%d %d %d %d %d\n", blockIdx.x, threadIdx.x, threadIdx.y, threadIdx.z, i);
        //so why why would you add thread id?
       // if (i < N)  //it is always less than N, isn't it?
        {
            //printf("%d %d %d %d %d %d %d\n", blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z, i);
            ;

        }*/
    /*
        if (particlesPerBlock > 32) //???
        {
            __syncthreads();
        }
        */
        //printf("hello2 %d\n", idxspline_i);
        // SPREAD
        /*
        for (int spread_i0 = 0; spread_i0 < K; spread_i0 += B) // B is 1 now = stride?
        {
            int i = block_i0 + block_i + spread_i0 + threadIdx.z / D;
            if (i < n)
            {
                int *i0 = idxxptr;
                int *j0 = idxyptr;
                int *k0 = idxzptr;
                real *thx = theta;
                real *thy = thx + order*N;
                real *thz = thy + order*N;

                int ithz = threadIdx.x;
                int ithy = threadIdx.y;
                int i = blockIdx.z * blockDim.z + threadIdx.z;
                if (i < n)
                {
                    if (coefficient[i])
                    {
                        _Pragma("unroll")
                        for (int ithx0 = 0; ithx0 < order; ithx0 += D)
                        {
                            int ithx = ithx0 + threadIdx.z % D;
                            int index_x = (i0[i]+ithx)*ny*nz;
                            real valx    = coefficient[i]*thx[i*order+ithx];

                            real valxy    = valx*thy[i*order+ithy];
                            int index_xy = index_x+(j0[i]+ithy)*nz;

                            int index_xyz        = index_xy+(k0[i]+ithz);
                            //grid[index_xyz] += valxy*thz[i*order+ithz];
                            atomicAdd(&grid[index_xyz], valxy*thz[i*order+ithz]);
                        }
                    }
                }
            }
        }
        */
    //}
}


void spread3_yup_gpu(struct gmx_pme_t *pme, pme_atomcomm_t *atc,
         int grid_index,
         pmegrid_t *pmegrid)
{
  int nx = pme->nkx, ny = pme->nky, nz = pme->nkz;
  //int nx = pmegrid->s[XX], ny = pmegrid->s[YY], nz = pmegrid->s[ZZ];
  real *grid = pmegrid->grid;
  const int order = pmegrid->order;
  int thread = 0;


    printf("%d second\n", pme->nnx[201]);

  int n = atc->n;
  int n_blocked = (n + warp_size - 1) / warp_size * warp_size;
  int ndatatot = nx*ny*nz;
  int size_grid = ndatatot * sizeof(real);

#ifdef DEBUG_PME_GPU
  // GRID CHECK


  real *grid_check;
  if (check_vs_cpu_j(spread_gpu_flags, 3))
  {
    grid_check = th_a(TH_ID_GRID, thread, size_grid, TH_LOC_HOST);
    memcpy(grid_check, pmegrid, ndatatot * sizeof(real));
  }
#endif
  // G2T
  /*
  int *g2tx_h = pme->pmegrid[grid_index].g2t[XX];
  int *g2ty_h = pme->pmegrid[grid_index].g2t[YY];
  int *g2tz_h = pme->pmegrid[grid_index].g2t[ZZ];
  int *g2tx_d = th_i(TH_ID_G2T, thread, 3 * n32 * sizeof(int), TH_LOC_CUDA);
  int *g2ty_d = g2tx_d + n32;
  int *g2tz_d = g2ty_d + n32;
  hipMemcpy(g2tx_d, g2tx_h, n * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(g2ty_d, g2ty_h, n * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(g2tz_d, g2tz_h, n * sizeof(int), hipMemcpyHostToDevice);
  */

  // FSH
  real *fshx_d = th_a(TH_ID_FSH, thread, 5 * (nx + ny) * sizeof(real), TH_LOC_CUDA);
  real *fshy_d = fshx_d + 5 * nx;
  hipMemcpy(fshx_d, pme->fshx, 5 * nx * sizeof(real), hipMemcpyHostToDevice);
  hipMemcpy(fshy_d, pme->fshy, 5 * ny * sizeof(real), hipMemcpyHostToDevice);

  // NN
  int *nnx_d = th_i(TH_ID_NN, thread, 5 * (nx + ny + nz) * sizeof(int), TH_LOC_CUDA);
  int *nny_d = nnx_d + 5 * nx;
  int *nnz_d = nny_d + 5 * ny;
  hipMemcpy(nnx_d, pme->nnx, 5 * nx * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(nny_d, pme->nny, 5 * ny * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(nnz_d, pme->nnz, 5 * nz * sizeof(int), hipMemcpyHostToDevice);

  // XPTR
  real *xptr_h = th_a(TH_ID_XPTR, thread, 3 * n_blocked * sizeof(real), TH_LOC_HOST);
  real *xptr_d = th_a(TH_ID_XPTR, thread, 3 * n_blocked * sizeof(real), TH_LOC_CUDA);
  real *yptr_d = xptr_d + n_blocked;
  real *zptr_d = yptr_d + n_blocked;
  {
    int ix = 0, iy = n_blocked, iz = 2 * n_blocked;
    for (int i = 0; i < n; i++)
    {
      real *xptr = atc->x[i];
      xptr_h[ix++] = xptr[XX];
      xptr_h[iy++] = xptr[YY];
      xptr_h[iz++] = xptr[ZZ];
    }
  }
  hipMemcpy(xptr_d, xptr_h, 3 * n_blocked * sizeof(real), hipMemcpyHostToDevice);

  // COEFFICIENT
  real *coefficient_d = th_a(TH_ID_COEFFICIENT, thread, n * sizeof(real), TH_LOC_CUDA);
  hipMemcpy(coefficient_d, atc->coefficient, n * sizeof(real), hipMemcpyHostToDevice);

  // GRID

  for (int i = 0; i < ndatatot; i++)
    {
      // FIX clear grid on device instead
      grid[i] = 0;
    }

  hipError_t stat;
  real *grid_d = th_a(TH_ID_GRID, thread, size_grid, TH_LOC_CUDA);
  //stat = hipMemcpy(grid_d, grid, size_grid, hipMemcpyHostToDevice);
  //CU_RET_ERR(stat, "hipMemcpy spread error");
  stat = hipMemset(grid_d, 0, size_grid);
  CU_RET_ERR(stat, "hipMemset spread error");
      printf("%d third\n", pme->nnx[201]);
#ifdef DEBUG_PME_TIMINGS_GPU
  events_record_start(gpu_events_spread);
#endif
  /*
  const int N = 256;
  const int D = 2;
  int n_blocks = (n + N - 1) / N;
  dim3 dimGrid(n_blocks, 1, 1);
  dim3 dimBlock(order, order, D);
  */
  const int particlesPerBlock = warp_size;
  //const int D = 2;
  dim3 nBlocks((n + particlesPerBlock - 1) / particlesPerBlock, 1, 1);
  //dim3 dimBlock(order, order, D); //each block has 32 threads now to hand 32 particlesPerBlock
  dim3 dimBlock(particlesPerBlock, 1, 1); //yupinov heavy
  switch (order)
  {
  case 4:
      /*
    const int O = 4;
    const int B = 1;
    const int K = B * D * O * O;
    */
    //spread3_kernel<4, N, K, D><<<dimGrid, dimBlock>>>
      spread3_kernel<4, particlesPerBlock><<<nBlocks, dimBlock>>>
      (nx, ny, nz,
       pme->pmegrid_start_ix, pme->pmegrid_start_iy, pme->pmegrid_start_iz,
       pme->recipbox[XX][XX],
       pme->recipbox[YY][XX],
       pme->recipbox[YY][YY],
       pme->recipbox[ZZ][XX],
       pme->recipbox[ZZ][YY],
       pme->recipbox[ZZ][ZZ],
       //g2tx_d, g2ty_d, g2tz_d,
       fshx_d, fshy_d,
       nnx_d, nny_d, nnz_d,
       xptr_d, yptr_d, zptr_d,
       coefficient_d,
       grid_d,
       n);
    //yupinov orders
  }
  CU_LAUNCH_ERR("spread3_kernel");

#ifdef DEBUG_PME_TIMINGS_GPU
  events_record_stop(gpu_events_spread, ewcsPME_SPREAD, 3);
#endif
#ifdef DEBUG_PME_GPU
  if (check_vs_cpu_j(spread_gpu_flags, 3)) {
    print_mutex.lock();
    fprintf(stderr, "Check %d  (%d x %d x %d)\n",
        thread, nx, ny, nz);
    for (int i = 0; i < ndatatot; ++i) {
      real diff = grid_check[i];
      real cpu_v = grid_check[i];
      hipMemcpy(&grid_check[i], &grid_d[i], sizeof(real), hipMemcpyDeviceToHost);
      diff -= grid_check[i];
      real gpu_v = grid_check[i];
      if (diff != 0) {
    real absdiff = fabs(diff) / fabs(cpu_v);
    if (absdiff > .000001) {
      fprintf(stderr, "%dppm", (int) (absdiff * 1e6));
      if (absdiff > .0001) {
        fprintf(stderr, " value %f ", cpu_v);
      }
    } else {
      fprintf(stderr, "~");
    }
    //fprintf(stderr, "(%f - %f)", cpu_v, gpu_v);
      } else {
    if (gpu_v == 0) {
      fprintf(stderr, "0");
    } else {
      fprintf(stderr, "=");
    }
      }
      if ((i + 1) % nz == 0) {
    fprintf(stderr, "\n");
      }
    }
    print_mutex.unlock();
  }
  #endif
  stat = hipMemcpy(grid, grid_d, size_grid, hipMemcpyDeviceToHost); //yupinov part of check?
  CU_RET_ERR(stat, "hipMemcpy spread3 error");
}

