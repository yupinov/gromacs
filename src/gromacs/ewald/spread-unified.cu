#include "hip/hip_runtime.h"
/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 1991-2000, University of Groningen, The Netherlands.
 * Copyright (c) 2001-2004, The GROMACS development team.
 * Copyright (c) 2013-2015, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */
//yupinov unused file!
#include "pme.h"
#include "pme-internal.h"

#include "gromacs/utility/basedefinitions.h"
#include "gromacs/utility/real.h"
#include "gromacs/math/vectypes.h"
#include "check.h"
#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/gpu_utils/cuda_arch_utils.cuh"
#include <hip/hip_runtime.h>

typedef real *splinevec[DIM];
#ifdef DEBUG_PME_GPU
extern gpu_flags spread_gpu_flags;
extern gpu_flags spread_bunching_gpu_flags;
#endif
#ifdef DEBUG_PME_TIMINGS_GPU
extern gpu_events gpu_events_spread;
#endif
#include "thread_mpi/mutex.h"

#include "pme-cuda.h"
#include "th-a.cuh"


static tMPI::mutex print_mutex; //yupinov



/* This has to be a macro to enable full compiler optimization with xlC (and probably others too) */

#define DO_BSPLINE(order)                                         \
    _Pragma("unroll")                                                    \
    for (ithx = 0; (ithx < order); ithx++)                    \
    {                                                             \
        index_x = (i0 + ithx) * pny * pnz;                    \
        valx = coefficient[globalParticleIndex] * thx[ithx];                      \
        _Pragma("unroll")                                                         \
        for (ithy = 0; (ithy < order); ithy++)                \
        {                                                         \
            valxy    = valx*thy[ithy];                       \
            index_xy = index_x+(j0+ithy)*pnz;                 \
             _Pragma("unroll")                                                     \
            for (ithz = 0; (ithz < order); ithz++)            \
            {                                                     \
                index_xyz        = index_xy+(k0+ithz);        \
                atomicAdd(grid + index_xyz, valxy*thz[ithz]);    \
            }                                                     \
        }                                                         \
    }


//template <int order, int N, int K, int D>
// K is particles per block?
template <int order, int particlesPerBlock>
__global__ void spread3_kernel
(int nx, int ny, int nz,
 int start_ix, int start_iy, int start_iz,
 real rxx, real ryx, real ryy, real rzx, real rzy, real rzz,
 //int *g2tx, int *g2ty, int *g2tz,
 real *fshx, real *fshy,
 int *nnx, int *nny, int *nnz,
 real *xptr, real *yptr, real *zptr,
 real *coefficient,
 real *grid, real *theta, real *dtheta, int *idx, //yupinov
 int n)
{
/*

    pnx = pmegrid->s[XX];
    pny = pmegrid->s[YY];
    pnz = pmegrid->s[ZZ];

    offx = pmegrid->offset[XX];
    offy = pmegrid->offset[YY];
    offz = pmegrid->offset[ZZ];

*/

    const int offx = 0, offy = 0, offz = 0;
    const int pny = ny + order - 1, pnz = nz + order - 1; //yupinov fix me!

    //const int B = K / D / order / order;

    __shared__ int idxxptr[particlesPerBlock];
    __shared__ int idxyptr[particlesPerBlock];
    __shared__ int idxzptr[particlesPerBlock];
    __shared__ real fxptr[particlesPerBlock];
    __shared__ real fyptr[particlesPerBlock];
    __shared__ real fzptr[particlesPerBlock];

    __shared__ real theta_shared[3 * order * particlesPerBlock];
    __shared__ real dtheta_shared[3 * order * particlesPerBlock];
    //printf("%d %d %d %d %d %d\n", blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z);

    // so I have particlesPerBlock to process with warp_size threads?

    int ithx, index_x, ithy, index_xy, ithz, index_xyz;
    real valx, valxy;


    int localParticleIndex = threadIdx.x;  //yupinov
    int globalParticleIndex = blockIdx.x * particlesPerBlock + localParticleIndex;
    if (globalParticleIndex < n)
    //yupinov - this is a single particle work!
        //yup bDoSplines!
    {
        // INTERPOL_IDX

        /* Fractional coordinates along box vectors, add 2.0 to make 100% sure we are positive for triclinic boxes */
        real tx, ty, tz;
        tx = nx * ( xptr[globalParticleIndex] * rxx + yptr[globalParticleIndex] * ryx + zptr[globalParticleIndex] * rzx + 2.0 );
        ty = ny * (                                   yptr[globalParticleIndex] * ryy + zptr[globalParticleIndex] * rzy + 2.0 );
        tz = nz * (                                                                     zptr[globalParticleIndex] * rzz + 2.0 );

        int tix, tiy, tiz;
        tix = (int)(tx);
        tiy = (int)(ty);
        tiz = (int)(tz);
        /* Because decomposition only occurs in x and y,
        * we never have a fraction correction in z.
        */

        fxptr[localParticleIndex] = tx - tix + fshx[tix];
        fyptr[localParticleIndex] = ty - tiy + fshy[tiy];
        fzptr[localParticleIndex] = tz - tiz;

        idxxptr[localParticleIndex] = nnx[tix];
        idxyptr[localParticleIndex] = nny[tiy];
        idxzptr[localParticleIndex] = nnz[tiz];

        // CALCSPLINE

        if (coefficient[globalParticleIndex] != 0.0) //yupinov weak
        {
            real dr, div;
            real data[order];

            _Pragma("unroll")
            for (int j = 0; j < DIM; j++)
            {
                //dr  = fractx[i*DIM + j];
                dr = j == 0 ? fxptr[localParticleIndex] : (j == 1 ? fyptr[localParticleIndex] : fzptr[localParticleIndex]);

                /* dr is relative offset from lower cell limit */
                data[order - 1] = 0;
                data[1]         = dr;
                data[0]         = 1 - dr;

                _Pragma("unroll")
                for (int k = 3; k < order; k++)
                {
                    div         = 1.0 / (k - 1.0);
                    data[k - 1] = div * dr * data[k - 2];
                    _Pragma("unroll")
                    for (int l = 1; l < (k - 1); l++)
                    {
                        data[k - l - 1] = div * ((dr + l) * data[k - l - 2] + (k - l - dr) * data[k - l - 1]);
                    }
                    data[0] = div * (1 - dr) * data[0];
                }
                /* differentiate */
                int thetaOffset = (j * particlesPerBlock + localParticleIndex) * order;
                dtheta_shared[thetaOffset] = -data[0];

                _Pragma("unroll")
                for (int k = 1; k < order; k++)
                {
                    dtheta_shared[thetaOffset + k] = data[k - 1] - data[k];
                }

                div             = 1.0 / (order - 1);
                data[order - 1] = div * dr * data[order - 2];
                _Pragma("unroll")
                for (int l = 1; l < (order - 1); l++)
                {
                    data[order - l - 1] = div * ((dr + l) * data[order - l - 2] + (order - l - dr) * data[order - l - 1]);
                }
                data[0] = div * (1 - dr) * data[0];

                _Pragma("unroll")
                for (int k = 0; k < order; k++)
                {
                    theta_shared[thetaOffset + k] = data[k];
                }
            }

            //yupinov store to global
            _Pragma("unroll")
            for (int j = 0; j < DIM; j++)
            {
                int thetaOffset = (j * particlesPerBlock + localParticleIndex) * order;
                int thetaGlobalOffset = (j * n + globalParticleIndex) * order;
                _Pragma("unroll")
                for (int z = 0; z < order; z++)
                {
                    theta[thetaGlobalOffset + z] = theta_shared[thetaOffset + z];
                    dtheta[thetaGlobalOffset + z] = dtheta_shared[thetaOffset + z];
                }
            }
            idx[globalParticleIndex * DIM + 0] = idxxptr[localParticleIndex];
            idx[globalParticleIndex * DIM + 1] = idxyptr[localParticleIndex];
            idx[globalParticleIndex * DIM + 2] = idxzptr[localParticleIndex];

            // SPREAD


            int i0   = idxxptr[localParticleIndex] - offx; //?
            int j0   = idxyptr[localParticleIndex] - offy;
            int k0   = idxzptr[localParticleIndex] - offz;

            real *thx = theta_shared + (0 * particlesPerBlock + localParticleIndex) * order;
            real *thy = theta_shared + (1 * particlesPerBlock + localParticleIndex) * order;
            real *thz = theta_shared + (2 * particlesPerBlock + localParticleIndex) * order;

            // switch (order)
            DO_BSPLINE(order);
        }
    }
}


void spread_on_grid_gpu(struct gmx_pme_t *pme, pme_atomcomm_t *atc,
         int grid_index,
         pmegrid_t *pmegrid)//yupinov, gmx_bool bCalcSplines, gmx_bool bSpread, gmx_bool bDoSplines)
//yupinov templating!
//real *fftgrid
//added:, gmx_wallcycle_t wcycle)
{
    hipError_t stat;
    hipStream_t s = pme->gpu->pmeStream;

    atc->spline[0].n = atc->n; //yupinov - without it, the conserved energy went down by 0.5%! used in gather or sometwhere else?

    int nx = pme->nkx, ny = pme->nky, nz = pme->nkz;
    //int nx = pmegrid->s[XX], ny = pmegrid->s[YY], nz = pmegrid->s[ZZ];
    real *grid = pmegrid->grid;
    const int order = pmegrid->order;
    int thread = 0;

    const int pnx = nx + order - 1, pny = ny + order - 1, pnz = nz + order - 1; //yupinov fix me!

    int n = atc->n;
    int n_blocked = (n + warp_size - 1) / warp_size * warp_size;
    int ndatatot = pnx*pny*pnz;
    int size_grid = ndatatot * sizeof(real);

    int size_order = order * n * sizeof(real);
    int size_order_dim = size_order * DIM;
    real *theta_d = th_a(TH_ID_THETA, thread, size_order_dim, TH_LOC_CUDA);
    real *dtheta_d = th_a(TH_ID_DTHETA, thread, size_order_dim, TH_LOC_CUDA);

    // G2T
    /*
    int *g2tx_h = pme->pmegrid[grid_index].g2t[XX];
    int *g2ty_h = pme->pmegrid[grid_index].g2t[YY];
    int *g2tz_h = pme->pmegrid[grid_index].g2t[ZZ];
    int *g2tx_d = th_i(TH_ID_G2T, thread, 3 * n32 * sizeof(int), TH_LOC_CUDA);
    int *g2ty_d = g2tx_d + n32;
    int *g2tz_d = g2ty_d + n32;
    hipMemcpy(g2tx_d, g2tx_h, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(g2ty_d, g2ty_h, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(g2tz_d, g2tz_h, n * sizeof(int), hipMemcpyHostToDevice);
    */

    // IDXPTR
    int idx_size = n * DIM * sizeof(int);
    int *idx_d = th_i(TH_ID_IDXPTR, thread, idx_size, TH_LOC_CUDA); //why is it not stored?

    // FSH
    real *fshx_d = th_a(TH_ID_FSH, thread, 5 * (nx + ny) * sizeof(real), TH_LOC_CUDA);
    real *fshy_d = fshx_d + 5 * nx;
    th_cpy(fshx_d, pme->fshx, 5 * nx * sizeof(real), TH_LOC_CUDA, s);
    th_cpy(fshy_d, pme->fshy, 5 * ny * sizeof(real), TH_LOC_CUDA, s);

    // NN
    int *nnx_d = th_i(TH_ID_NN, thread, 5 * (nx + ny + nz) * sizeof(int), TH_LOC_CUDA);
    int *nny_d = nnx_d + 5 * nx;
    int *nnz_d = nny_d + 5 * ny;
    th_cpy(nnx_d, pme->nnx, 5 * nx * sizeof(int), TH_LOC_CUDA, s);
    th_cpy(nny_d, pme->nny, 5 * ny * sizeof(int), TH_LOC_CUDA, s);
    th_cpy(nnz_d, pme->nnz, 5 * nz * sizeof(int), TH_LOC_CUDA, s);

    // XPTR
    real *xptr_h = th_a(TH_ID_XPTR, thread, 3 * n_blocked * sizeof(real), TH_LOC_HOST);
    real *xptr_d = th_a(TH_ID_XPTR, thread, 3 * n_blocked * sizeof(real), TH_LOC_CUDA);
    real *yptr_d = xptr_d + n_blocked;
    real *zptr_d = yptr_d + n_blocked;
    {
        int ix = 0, iy = n_blocked, iz = 2 * n_blocked;
        for (int i = 0; i < n; i++)
        {
          real *xptr = atc->x[i];
          xptr_h[ix++] = xptr[XX];
          xptr_h[iy++] = xptr[YY];
          xptr_h[iz++] = xptr[ZZ];
        }
    }
    th_cpy(xptr_d, xptr_h, 3 * n_blocked * sizeof(real), TH_LOC_CUDA, s);

    // COEFFICIENT
    real *coefficient_d = th_a_cpy(TH_ID_COEFFICIENT, thread, atc->coefficient, n * sizeof(real), TH_LOC_CUDA, s); //yupinov compact here as weel?

    // GRID
    /*
    for (int i = 0; i < ndatatot; i++)
    {
      // FIX clear grid on device instead
      grid[i] = 0;
    }
    */

    real *grid_d = th_a(TH_ID_GRID, thread, size_grid, TH_LOC_CUDA);
    stat = hipMemsetAsync(grid_d, 0, size_grid, s); //yupinov
    CU_RET_ERR(stat, "hipMemsetAsync spread error");
    #ifdef DEBUG_PME_TIMINGS_GPU
    events_record_start(gpu_events_spread, s);
    #endif
    /*
    const int N = 256;
    const int D = 2;
    int n_blocks = (n + N - 1) / N;
    dim3 dimGrid(n_blocks, 1, 1);
    dim3 dimBlock(order, order, D);
    */
    const int particlesPerBlock = warp_size;
    //const int D = 2;
    dim3 nBlocks((n + particlesPerBlock - 1) / particlesPerBlock, 1, 1);
    //dim3 dimBlock(order, order, D); //each block has 32 threads now to hand 32 particlesPerBlock
    dim3 dimBlock(particlesPerBlock, 1, 1); //yupinov heavy
    switch (order)
    {
      case 4:
          /*
    const int O = 4;
    const int B = 1;
    const int K = B * D * O * O;
    */
          //spread3_kernel<4, N, K, D><<<dimGrid, dimBlock>>>
          spread3_kernel<4, particlesPerBlock><<<nBlocks, dimBlock, 0, s>>>
                                                                    (nx, ny, nz,
                                                                     pme->pmegrid_start_ix, pme->pmegrid_start_iy, pme->pmegrid_start_iz,
                                                                     pme->recipbox[XX][XX],
                                                                     pme->recipbox[YY][XX],
                                                                     pme->recipbox[YY][YY],
                                                                     pme->recipbox[ZZ][XX],
                                                                     pme->recipbox[ZZ][YY],
                                                                     pme->recipbox[ZZ][ZZ],
                                                                     //g2tx_d, g2ty_d, g2tz_d,
                                                                     fshx_d, fshy_d,
                                                                     nnx_d, nny_d, nnz_d,
                                                                     xptr_d, yptr_d, zptr_d,
                                                                     coefficient_d,
                                                                     grid_d, theta_d, dtheta_d, idx_d,
                                                                     n);
          //yupinov orders
    }
    CU_LAUNCH_ERR("spread3_kernel");

#ifdef DEBUG_PME_TIMINGS_GPU
  events_record_stop(gpu_events_spread, s, ewcsPME_SPREAD, 3);
#endif
  th_cpy(grid, grid_d, size_grid, TH_LOC_HOST, s);
  for (int j = 0; j < DIM; ++j)
  {
      th_cpy(atc->spline[thread].dtheta[j], dtheta_d + j * n * order, size_order, TH_LOC_HOST, s);
      th_cpy(atc->spline[thread].theta[j], theta_d + j * n * order, size_order, TH_LOC_HOST, s);
  }
  th_cpy(atc->idx, idx_d, idx_size, TH_LOC_HOST, s);
//yupinov free, keep allocated
  /*
  hipFree(theta_d);
  hipFree(dtheta_d);
  hipFree(fractx_d);
  hipFree(coefficient_d);
  free(fractx_h);
  free(coefficient_h);
  */
}

