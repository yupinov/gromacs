#include <hip/hip_runtime.h>

#include "gromacs/utility/basedefinitions.h"
#include "gromacs/utility/real.h"

#ifdef DEBUG_PME_TIMINGS_GPU

#include "gromacs/timing/gpu_timing.h"
#include "gromacs/timing/wallcycle.h"


struct gpu_events
{
  bool created;
  hipEvent_t event_start, event_stop;
  gpu_events() : created(false) { }
};

gpu_events gpu_events_interpol_idx;
gpu_events gpu_events_calcspline;
gpu_events gpu_events_spread;
gpu_events gpu_events_fft_r2c;
gpu_events gpu_events_solve;
gpu_events gpu_events_fft_c2r;
gpu_events gpu_events_gather;

void events_record_start(gpu_events &events)
{
  if (!events.created) {
    hipEventCreate(&events.event_start);
    hipEventCreate(&events.event_stop);
    events.created = true;
  }
  hipEventRecord(events.event_start);
}

void events_record_stop(gpu_events &events, int ewcsn, int j) {
  hipEventRecord(events.event_stop);
  hipEventSynchronize(events.event_stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, events.event_start, events.event_stop);

  int idx = ewcsn - ewcsPME_INTERPOL_IDX;
  gmx_wallclock_gpu_pme.pme_time[idx][j].t += milliseconds;
  ++gmx_wallclock_gpu_pme.pme_time[idx][j].c;
}
#endif



#ifdef DEBUG_PME_GPU

#include "thread_mpi/mutex.h"

const bool check_verbose = false;
static tMPI::mutex print_mutex;

template <typename T>
void check(const char *name, T *data, T *expected, int size, gmx_bool bDevice, gmx_bool bPrintGrid = false)
{
    gmx_bool print1Char = bPrintGrid;
    gmx_bool printAlways = print1Char; //|=
    print_mutex.lock();
    bool bDiff = false;
    for (int i = 0; i < size; ++i) 
    {
        T cpu_v = expected[i];
        T gpu_v;
        if (bDevice) 
            hipMemcpy(&gpu_v, &data[i], sizeof(T), hipMemcpyDeviceToHost);
        else 
            gpu_v = data[i];
        T diff = gpu_v - cpu_v;
        if (check_verbose) 
          fprintf(stderr, " %d:%f(%f)", i, (double) cpu_v, (double) diff);
        if (diff != 0) 
        {
            if (!bDiff && name && !print1Char)
            {
                fprintf(stderr, "%s:\n", name);
                bDiff = true;
            }
            T absdiff = diff > 0 ? diff : -diff;
            T abscpu_v = cpu_v > 0 ? cpu_v : -cpu_v;
            T reldiff = absdiff / (abscpu_v > 1e-11 ? abscpu_v : 1e-11);
            if (reldiff > .000001)
            {
                if (print1Char)
                    fprintf(stderr, "&");
                else
                    fprintf(stderr, "%.0fppm", (double) (reldiff * 1e6));
                if (reldiff > .0001)
                    if (print1Char)
                        fprintf(stderr, "!");
                    else
                        fprintf(stderr, " value %f vs %f ", (double) cpu_v, (double) gpu_v);
            }
            else
                fprintf(stderr, "~");
        }
        else if (printAlways)
        {
            if (gpu_v == 0)
            {
                fprintf(stderr, "0");
            }
            else
            {
                fprintf(stderr, "=");
            }
        }
    }
    if (bDiff || printAlways)
        fprintf(stderr, "\n");
    print_mutex.unlock();
}

void check_int(const char *name, int *data, int *expected, int size, gmx_bool bDevice, gmx_bool bPrintGrid)
{
  check(name, data, expected, size, bDevice, bPrintGrid);
}

void check_real(const char *name, real *data, real *expected, int size, gmx_bool bDevice, gmx_bool bPrintGrid)
{
  check(name, data, expected, size, bDevice, bPrintGrid);
}

void print_lock() {
  print_mutex.lock();
}

void print_unlock() {
  print_mutex.lock();
}
#endif
