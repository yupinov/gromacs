#include <hip/hip_runtime.h>

#include "pme.h"
#include "pme-cuda.cuh"
#include "pme-timings.cuh"
#include "gromacs/timing/gpu_timing.h"
#include "gromacs/timing/wallcycle.h"
#include "gromacs/utility/gmxassert.h"
#include "gromacs/utility/smalloc.h"
#include "gromacs/gpu_utils/cudautils.cuh"

pme_gpu_timing::pme_gpu_timing()
{
    initialized = false;
    reset();
}

pme_gpu_timing::~pme_gpu_timing()
{
    if (initialized)
    {
        hipError_t stat;
        stat = hipEventDestroy(event_start);
        CU_RET_ERR(stat, "PME timing hipEventDestroy fail");
        stat = hipEventDestroy(event_stop);
        CU_RET_ERR(stat, "PME timing hipEventDestroy fail");
        initialized = false;
    }
}

void pme_gpu_timing::enable()
{
    if (!initialized)
    {
        hipError_t stat;
        stat = hipEventCreate(&event_start, hipEventDefault);
        CU_RET_ERR(stat, "PME timing hipEventCreate fail");
        stat = hipEventCreate(&event_stop, hipEventDefault);
        CU_RET_ERR(stat, "PME timing hipEventCreate fail");
        initialized = true;
    }
}

void pme_gpu_timing::start_recording(hipStream_t s)
{
    if (initialized)
    {
        hipError_t stat = hipEventRecord(event_start, s);
        CU_RET_ERR(stat, "PME timing hipEventRecord fail");
    }
}

void pme_gpu_timing::stop_recording(hipStream_t s)
{
    if (initialized)
    {
        hipError_t stat = hipEventRecord(event_stop, s);
        CU_RET_ERR(stat, "PME timing hipEventRecord fail");
        call_count++;
    }
}

void pme_gpu_timing::reset()
{
    total_milliseconds = 0.0;
    call_count = 0;
}

void pme_gpu_timing::update()
{
    if (initialized && (call_count > 0)) // only touched events needed
    {
        real milliseconds = 0.0;
        hipError_t stat = hipEventElapsedTime(&milliseconds, event_start, event_stop);
        CU_RET_ERR(stat, "PME timing hipEventElapsedTime fail");
        total_milliseconds += milliseconds;
    }
}

real pme_gpu_timing::get_total_time_milliseconds()
{
    return total_milliseconds;
}

unsigned int pme_gpu_timing::get_call_count()
{
    return call_count;
}

// general functions

void pme_gpu_timing_start(gmx_pme_t *pme, int PMEStageId)
{
    pme->gpu->timingEvents[PMEStageId]->start_recording(pme->gpu->pmeStream);
}

void pme_gpu_timing_stop(gmx_pme_t *pme, int PMEStageId)
{
    pme->gpu->timingEvents[PMEStageId]->stop_recording(pme->gpu->pmeStream);
}

void pme_gpu_get_timings(gmx_wallclock_gpu_t **timings, gmx_pme_t *pme)
{
    if (pme_gpu_enabled(pme))
    {
        GMX_ASSERT(timings, "Null GPU timing pointer");
        if (!*timings)
        {
            // alloc for PME-only run
            snew(*timings, 1);
            // init_timings(*timings);
            // frankly, it's just memset..
        }
        (*timings)->pme.timing.resize(pme->gpu->timingEvents.size());
        for (size_t i = 0; i < pme->gpu->timingEvents.size(); i++)
        {
            (*timings)->pme.timing[i].t = pme->gpu->timingEvents[i]->get_total_time_milliseconds();
            (*timings)->pme.timing[i].c = pme->gpu->timingEvents[i]->get_call_count();
        }
    }
}

void pme_gpu_update_timings(gmx_pme_t *pme)
{
    if (pme_gpu_enabled(pme))
    {
        for (size_t i = 0; i < pme->gpu->timingEvents.size(); i++)
            pme->gpu->timingEvents[i]->update();
    }
}

void pme_gpu_init_timings(gmx_pme_t *pme)
{
    if (pme_gpu_enabled(pme))
    {
        hipStreamSynchronize(pme->gpu->pmeStream);
        for (size_t i = 0; i < ewcsPME_END_INVALID; i++)
        {
            pme->gpu->timingEvents.push_back(new pme_gpu_timing());
            pme->gpu->timingEvents[i]->enable();
        }
    }
}

void pme_gpu_destroy_timings(gmx_pme_t *pme)
{
    if (pme_gpu_enabled(pme))
    {
        for (size_t i = 0; i < pme->gpu->timingEvents.size(); i++)
            delete pme->gpu->timingEvents[i];
        pme->gpu->timingEvents.resize(0);
    }
}

void pme_gpu_reset_timings(gmx_pme_t *pme)
{
    if (pme_gpu_enabled(pme))
    {
        for (size_t i = 0; i < pme->gpu->timingEvents.size(); i++)
            pme->gpu->timingEvents[i]->reset();
    }
}
