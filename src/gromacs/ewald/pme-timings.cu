#include "gromacs/utility/basedefinitions.h"
#include <hip/hip_runtime.h>
#include "pme-cuda.cuh"
#include "pme-timings.cuh"
#include "gromacs/timing/gpu_timing.h"
#include "gromacs/timing/wallcycle.h"
#include "gromacs/gpu_utils/cudautils.cuh"

pme_gpu_timing::pme_gpu_timing()
{
    initialized = false;
    reset();
}

pme_gpu_timing::~pme_gpu_timing()
{
#if PME_GPU_TIMINGS
    if (initialized)
    {
        hipError_t stat;
        stat = hipEventDestroy(event_start);
        CU_RET_ERR(stat, "PME timing hipEventDestroy fail");
        stat = hipEventDestroy(event_stop);
        CU_RET_ERR(stat, "PME timing hipEventDestroy fail");
        initialized = false;
    }
#endif
}

void pme_gpu_timing::check_init()
{
#if PME_GPU_TIMINGS
    if (!initialized)
    {
        hipError_t stat;
        stat = hipEventCreate(&event_start, hipEventDefault);
        CU_RET_ERR(stat, "PME timing hipEventCreate fail");
        stat = hipEventCreate(&event_stop, hipEventDefault);
        CU_RET_ERR(stat, "PME timing hipEventCreate fail");
        initialized = true;
    }
#endif
}

void pme_gpu_timing::start_recording(hipStream_t s)
{
    check_init();
#if PME_GPU_TIMINGS
    hipError_t stat = hipEventRecord(event_start, s);
    CU_RET_ERR(stat, "PME timing hipEventRecord fail");
#endif
}

void pme_gpu_timing::stop_recording(hipStream_t s)
{
#if PME_GPU_TIMINGS
    hipError_t stat = hipEventRecord(event_stop, s);
    CU_RET_ERR(stat, "PME timing hipEventRecord fail");
    call_count++;
#endif
}

void pme_gpu_timing::reset()
{
    total_milliseconds = 0.0;
    call_count = 0;
}

void pme_gpu_timing::update()
{
#if PME_GPU_TIMINGS
    real milliseconds = 0.0;
    if (initialized)
    {
        hipError_t stat = hipEventElapsedTime(&milliseconds, event_start, event_stop);
        CU_RET_ERR(stat, "PME timing hipEventElapsedTime fail");
    }
    total_milliseconds += milliseconds;
#endif
}

real pme_gpu_timing::get_total_time_milliseconds()
{
    return total_milliseconds;
}

unsigned int pme_gpu_timing::get_call_count()
{
    return call_count;
}

void pme_gpu_timing_start(gmx_pme_t *pme, int ewcsn)
{
    const int i = ewcsn - ewcsPME_INTERPOL_IDX;
    pme->gpu->timingEvents[i].start_recording(pme->gpu->pmeStream);
}

void pme_gpu_timing_stop(gmx_pme_t *pme, int ewcsn)
{
    const int i = ewcsn - ewcsPME_INTERPOL_IDX;
    pme->gpu->timingEvents[i].stop_recording(pme->gpu->pmeStream);
}

void pme_gpu_get_timing(gmx_pme_t *pme)
{
    if (pme && pme->bGPU)
    {
        for (int i = 0; i < PME_GPU_STAGES; i++)
        {
            gmx_wallclock_gpu_pme.pme_time[i].t = pme->gpu->timingEvents[i].get_total_time_milliseconds();
            gmx_wallclock_gpu_pme.pme_time[i].c = pme->gpu->timingEvents[i].get_call_count();
        }
    }
}

void pme_gpu_update_timing(gmx_pme_t *pme)
{
    if (pme && pme->bGPU)
    {
        for (int i = 0; i < PME_GPU_STAGES; i++)
            pme->gpu->timingEvents[i].update();
    }
}

void pme_gpu_reset_timings(gmx_pme_t *pme)
{
    if (pme && pme->bGPU)
    {
        for (int i = 0; i < PME_GPU_STAGES; i++)
            pme->gpu->timingEvents[i].reset();
    }
}
