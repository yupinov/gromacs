#include "hip/hip_runtime.h"
#include "pme.h"

#include "gromacs/utility/basedefinitions.h"
#include "gromacs/utility/real.h"
#include "gromacs/math/vectypes.h"
#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/gpu_utils/cuda_arch_utils.cuh"
#include <hip/hip_runtime.h>

#include "pme-timings.cuh"

#include "pme-internal.h"
#include "pme-cuda.cuh"

#include <assert.h>

void pme_gpu_alloc_gather_forces(gmx_pme_t *pme)
{
    const int tag = 0;
    const int n = pme->atc[0].n; //?
    assert(n > 0);
    const int forcesSize = DIM * n * sizeof(real);
    pme->gpu->forces = (real *)PMEMemoryFetch(PME_ID_FORCES, tag, forcesSize, ML_DEVICE);
}

void pme_gpu_get_forces(gmx_pme_t *pme)
{
    hipStream_t s = pme->gpu->pmeStream;
    hipError_t stat = hipStreamWaitEvent(s, pme->gpu->syncForcesH2D, 0);
    CU_RET_ERR(stat, "error while waiting for PME forces");

    const int tag = 0;
    const int n = pme->atc[0].n;
    const int forcesSize = DIM * n * sizeof(real);
    real *forces = (real *)PMEMemoryFetch(PME_ID_FORCES, tag, forcesSize, ML_HOST);
    memcpy(pme->atc[0].f, forces, forcesSize);

    /*
    if (PME_SKIP_ZEROES)
    {
        const int thread = 0;
        const int size_forces = DIM * n * sizeof(real);
        const int size_indices = n * sizeof(int);
        real *atc_f_h = (real *)PMEMemoryFetch(PME_ID_FORCES, thread, size_forces, ML_HOST);
        int *atc_i_compacted_h = (int *)PMEMemoryFetch(PME_ID_NONZERO_INDICES, thread, size_indices, ML_HOST);
        for (int iCompacted = 0; iCompacted < n; iCompacted++)  // iterating over compacted particles
        {
            int i = atc_i_compacted_h[iCompacted]; //index of uncompacted particle
            forces[i][XX] = atc_f_h[iCompacted * DIM + XX];
            forces[i][YY] = atc_f_h[iCompacted * DIM + YY];
            forces[i][ZZ] = atc_f_h[iCompacted * DIM + ZZ];
        }
    }
    */
}


//yupinov - texture memory?
template <
        const int order,
        const int particlesPerBlock,
        const gmx_bool bOverwriteForces
        >
__launch_bounds__(4 * warp_size, 16)
__global__ void pme_gather_kernel
(const real * __restrict__ gridGlobal, const int n,
 const float3 nXYZ, const int pnx, const int pny, const int pnz,
 const real * __restrict__ thetaGlobal,
 const real * __restrict__ dthetaGlobal,
 real * __restrict__ forcesGlobal, const real * __restrict__ coefficientGlobal,
 #if !PME_EXTERN_CMEM
  const struct pme_gpu_recipbox_t RECIPBOX,
 #endif
 const int * __restrict__ idxGlobal
 )
{
    /* sum forces for local particles */

    // these are particle indices - in shared and global memory
    const int localIndex = threadIdx.z;
    const int globalIndex = blockIdx.x * blockDim.z + threadIdx.z;

    const int particleDataSize = order * order;
    const int blockSize = particlesPerBlock * particleDataSize; //1 line per thread
    // should the array size aligned by warp size for shuffle?

    const int thetaStride = particlesPerBlock * DIM; // a global size dependency with spread!
    const int thetaSize = thetaStride * order;
    const int idxSize = thetaStride;
    __shared__ int idx[idxSize];
    __shared__ real theta[thetaSize];
    __shared__ real dtheta[thetaSize];


    // spline Y/Z coordinates
    const int ithy = threadIdx.y;
    const int ithz = threadIdx.x;
    // these are spline contribution indices in shared memory
    const int splineIndex = threadIdx.y * blockDim.x + threadIdx.x;   // relative to the current particle
    const int lineIndex = (threadIdx.z * (blockDim.x * blockDim.y)) + splineIndex; // and to all the block's particles


    int threadLocalId = (threadIdx.z * (blockDim.x * blockDim.y))
            + (threadIdx.y * blockDim.x)
            + threadIdx.x;

    if (threadLocalId < idxSize)
    {
        idx[threadLocalId] = idxGlobal[blockIdx.x * idxSize + threadLocalId];
    }
    if ((threadLocalId < thetaSize))
    {
        theta[threadLocalId] = thetaGlobal[blockIdx.x * thetaSize + threadLocalId];
        dtheta[threadLocalId] = dthetaGlobal[blockIdx.x * thetaSize + threadLocalId];
    }

    //locality?
    __syncthreads();

    real fx = 0.0f;
    real fy = 0.0f;
    real fz = 0.0f;

    if (globalIndex < n)
    {
        const int thetaOffsetY = localIndex * DIM + ithy * thetaStride + YY;
        const int thetaOffsetZ = localIndex * DIM + ithz * thetaStride + ZZ;
        const real ty = theta[thetaOffsetY];
        const real tz = theta[thetaOffsetZ];
        const real dy = dtheta[thetaOffsetY];
        const real dz = dtheta[thetaOffsetZ];
        //yupinov need to reorder theta when transferring thetas to and from CPU!
        for (int ithx = 0; (ithx < order); ithx++)
        {
            const int index_x = (idx[localIndex * DIM + XX] + ithx) * pny * pnz;
            const int index_xy = index_x + (idx[localIndex * DIM + YY] + ithy) * pnz;
            const real gridValue = gridGlobal[index_xy + (idx[localIndex * DIM + ZZ] + ithz)];
            const int thetaOffsetX = localIndex * DIM + ithx * thetaStride + XX;
            const real tx = theta[thetaOffsetX];
            const real dx = dtheta[thetaOffsetX];
            const real fxy1 = tz * gridValue;
            const real fz1  = dz * gridValue;
            fx += dx * ty * fxy1;
            fy += tx * dy * fxy1;
            fz += tx * ty * fz1;
            /*
            atomicAdd(fx + localIndex, dx * ty * fxy1);
            atomicAdd(fy + localIndex, tx * dy * fxy1);
            atomicAdd(fz + localIndex, tx * ty * fz1);
            */
        }
    }
    __syncthreads(); // breaking globalIndex condition?

    // now particlesPerBlock particles have to reduce order^2 contributions each

    __shared__ float3 fSumArray[particlesPerBlock];

#if (GMX_PTX_ARCH >= 300)
    if (!(order & (order - 1))) // only for orders of power of 2
    {
        // a tricky shuffle reduction inspired by reduce_force_j_warp_shfl

        assert(order == 4); // confused about others and the best data layout so far :(
        assert(particleDataSize <= warp_size);
        const int width = particleDataSize;
        // have to rework for particleDataSize > warp_size (order 8 or larger...)

        fx += __shfl_down(fx, 1, width);
        fy += __shfl_up  (fy, 1, width);
        fz += __shfl_down(fz, 1, width);

        if (splineIndex & 1)
        {
            fx = fy;
        }

        fx += __shfl_down(fx, 2, width);
        fz += __shfl_up  (fz, 2, width);

        if (splineIndex & 2)
        {
            fx = fz;
        }

        // by now fx contains intermediate sums of all 3 components in groups of 4:
        // splineIndex    0            1            2 and 3      4            5            6 and 7      8...
        // sum of...      fx0 to fx3   fy0 to fy3   fz0 to fz3   fx4 to fx7   fy4 to fy7   fz4 to fz7   etc.

        // we have to just further reduce those groups of 4
        for (int delta = 4; delta < particleDataSize; delta <<= 1)
        {
            fx += __shfl_down(fx, delta, width);
        }

        // a single operation for all 3 components!
        if (splineIndex < 3)
            *((real *)(&fSumArray[localIndex]) + splineIndex) = fx * ((real *)&nXYZ)[splineIndex];
    }
    else
#endif
    {
        // lazy 3-thread reduction in shared memory inspired by reduce_force_j_generic
        __shared__ real fSharedArray[DIM * blockSize];
        fSharedArray[lineIndex] = fx;
        fSharedArray[lineIndex + blockSize] = fy;
        fSharedArray[lineIndex + 2 * blockSize] = fz;

        if (splineIndex < 3)
        {
            float f = 0.0f;
            for (int j = localIndex * particleDataSize; j < (localIndex + 1) * particleDataSize; j++)
            {
                f += fSharedArray[blockSize * splineIndex + j];
            }
            *((real *)(&fSumArray[localIndex]) + splineIndex) = f * ((real *)&nXYZ)[splineIndex];
        }
    }
    __syncthreads();

    //reduce by components, again
    if (threadLocalId < DIM * particlesPerBlock)
    {
        // new, different particle indices
        const int localIndexFinal = threadLocalId / DIM;
        const int dimIndex = threadLocalId - localIndexFinal * DIM;

        const float3 fSum = fSumArray[localIndexFinal];
        const int globalIndexFinal = blockIdx.x * particlesPerBlock + localIndexFinal;
        const real coefficient = coefficientGlobal[globalIndexFinal];

        real contrib;
        // by columns!
        switch (dimIndex)
        {
            case XX:
            contrib = RECIPBOX.box[XX].x * fSum.x /*+ RECIPBOX.box[YY].x * fSum.y + RECIPBOX.box[ZZ].x * fSum.z*/;
            break;

            case YY:
            contrib = RECIPBOX.box[XX].y * fSum.x + RECIPBOX.box[YY].y * fSum.y /* + RECIPBOX.box[ZZ].y * fSum.z*/;
            break;

            case ZZ:
            contrib = RECIPBOX.box[XX].z * fSum.x + RECIPBOX.box[YY].z * fSum.y + RECIPBOX.box[ZZ].z * fSum.z;
            break;
        }
        contrib *= -coefficient;

        if (bOverwriteForces)
            forcesGlobal[blockIdx.x * particlesPerBlock * DIM + threadLocalId] = contrib;
        else
            forcesGlobal[blockIdx.x * particlesPerBlock * DIM + threadLocalId] += contrib;
    }
}


// a quick dirty copy of pme_wrap_kernel
template <
    const int order
    >
__global__ void pme_unwrap_kernel
    (const int nx, const int ny, const int nz,
     const int pny, const int pnz,
 #if !PME_EXTERN_CMEM
     const struct pme_gpu_overlap_t OVERLAP,
 #endif
     real * __restrict__ grid
     )
{
    // UNWRAP
    int blockId = blockIdx.x
                 + blockIdx.y * gridDim.x
                 + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                  + (threadIdx.z * (blockDim.x * blockDim.y))
                  + (threadIdx.y * blockDim.x)
                  + threadIdx.x;

    //should use ldg.128

    if (threadId < OVERLAP.overlapCellCounts[OVERLAP_ZONES - 1])
    {
        int zoneIndex = -1;
        do
        {
            zoneIndex++;
        }
        while (threadId >= OVERLAP.overlapCellCounts[zoneIndex]);
        const int2 zoneSizeYZ = OVERLAP.overlapSizes[zoneIndex];
        // this is the overlapped cells's index relative to the current zone
        const int cellIndex = (zoneIndex > 0) ? (threadId - OVERLAP.overlapCellCounts[zoneIndex - 1]) : threadId;

        // replace integer division/modular arithmetics - a big performance hit
        // try int_fastdiv?
        const int ixy = cellIndex / zoneSizeYZ.y; //yupinov check expensive integer divisions everywhere!
        const int iz = cellIndex - zoneSizeYZ.y * ixy;
        const int ix = ixy / zoneSizeYZ.x;
        const int iy = ixy - zoneSizeYZ.x * ix;
        const int sourceIndex = (ix * pny + iy) * pnz + iz;

        int targetOffset = 0;

        // stage those bits in constant memory as well
        const int overlapZ = ((zoneIndex == 0) || (zoneIndex == 3) || (zoneIndex == 4) || (zoneIndex == 6)) ? 1 : 0;
        const int overlapY = ((zoneIndex == 1) || (zoneIndex == 3) || (zoneIndex == 5) || (zoneIndex == 6)) ? 1 : 0;
        const int overlapX = ((zoneIndex == 2) || (zoneIndex > 3)) ? 1 : 0;
        if (overlapZ)
        {
            targetOffset = nz;
        }
        if (overlapY)
        {
            targetOffset += ny * pnz;
        }
        if (overlapX)
        {
            targetOffset += nx * pny * pnz;
        }
        const int targetIndex = sourceIndex + targetOffset;
        grid[targetIndex] = grid[sourceIndex];
    }
}

void gather_f_bsplines_gpu(struct gmx_pme_t *pme, real *grid,
                   pme_atomcomm_t *atc,
                   splinedata_t *spline,
                   real scale)
{
    //yupinov bClearf!
    int n = spline->n;
    if (!n)
        return;

    const int thread = 0;

    const gmx_bool bOverwriteForces = true;
    // false: we use some other GPU forces buffer for the final reduction, so we want to add to that
    // in that case, maybe we want to replace + with atomicAdd at the end of kernel?
    // true: we have our own buffer, so just write directly into that


    const int *spline_ind = spline->ind;
    const splinevec *spline_theta = &spline->theta;
    const splinevec *spline_dtheta = &spline->dtheta;

    hipStream_t s = pme->gpu->pmeStream;

    //pme_atomcomm_t atc = pme->atc[0];
    real *atc_coefficient = atc->coefficient;
    ivec *atc_idx = atc->idx;


    const int order = pme->pme_order;
    /*
    gmx_parallel_3dfft_real_limits_wrapper(pme, grid_index, local_ndata, local_offset, local_size);
    const int pnx = local_size[XX];
    const int pny = local_size[YY];
    const int pnz = local_size[ZZ];
    const int nx = local_ndata[XX];
    const int ny = local_ndata[YY];
    const int nz = local_ndata[ZZ];
    */

    /*
    const int pnx = pmegrid->n[XX];
    const int pny = pmegrid->n[YY];
    const int pnz = pmegrid->n[ZZ];
    */
    const int pnx   = pme->pmegrid_nx;
    const int pny   = pme->pmegrid_ny;
    const int pnz   = pme->pmegrid_nz;
    const int nx = pme->nkx;
    const int ny = pme->nky;
    const int nz = pme->nkz;


    const int ndatatot = pnx * pny * pnz;
    const int gridSize = ndatatot * sizeof(real);
    if (!pme->gpu->keepGPUDataBetweenC2RAndGather)
        cu_copy_H2D_async(pme->gpu->grid, grid, gridSize, s);

    if (pme->bGPUSingle)
    {
        if (order == 4)
        {
            const int blockSize = 4 * warp_size; //yupinov thsi is everywhere! and architecture-specific
            const int overlap = order - 1;

            const int overlappedCells = (nx + overlap) * (ny + overlap) * (nz + overlap) - nx * ny * nz;
            const int nBlocks = (overlappedCells + blockSize - 1) / blockSize;

            pme_gpu_timing_start(pme, ewcsPME_UNWRAP);

            pme_unwrap_kernel<4> <<<nBlocks, blockSize, 0, s>>>(nx, ny, nz, pny, pnz,
#if !PME_EXTERN_CMEM
                                                                pme->gpu->overlap,
#endif
                                                                pme->gpu->grid);

            CU_LAUNCH_ERR("pme_unwrap_kernel");

            pme_gpu_timing_stop(pme, ewcsPME_UNWRAP);

        }
        else
            gmx_fatal(FARGS, "gather: orders other than 4 untested!");
    }

    int forcesSize = DIM * n * sizeof(real);
    int size_indices = n * sizeof(int);
    int size_splines = order * n * sizeof(int);
    int size_coefficients = n * sizeof(real);

    real *atc_f_h = (real *)PMEMemoryFetch(PME_ID_FORCES, thread, forcesSize, ML_HOST);
    ivec *idx_h = NULL;

    real *coefficients_h = NULL;

    real *theta_x_h = NULL, *theta_y_h = NULL, *theta_z_h = NULL;
    real *dtheta_x_h = NULL, *dtheta_y_h = NULL, *dtheta_z_h = NULL;

    /*
    int *i0_h = NULL, *j0_h = NULL, *k0_h = NULL;
    i0_h = (int *)PMEFetch(PME_ID_I0, thread, size_indices, ML_HOST);
    j0_h = (int *)PMEFetch(PME_ID_J0, thread, size_indices, ML_HOST);
    k0_h = (int *)PMEFetch(PME_ID_K0, thread, size_indices, ML_HOST);
    */

    int *atc_i_compacted_h = NULL;

    // compact data (might be broken)
    if (PME_SKIP_ZEROES)
    {
        atc_i_compacted_h = (int *)PMEMemoryFetch(PME_ID_NONZERO_INDICES, thread, size_indices, ML_HOST);

        // thetas
        theta_x_h = (real *)PMEMemoryFetch(PME_ID_THX, thread, size_splines, ML_HOST);
        theta_y_h = (real *)PMEMemoryFetch(PME_ID_THY, thread, size_splines, ML_HOST);
        theta_z_h = (real *)PMEMemoryFetch(PME_ID_THZ, thread, size_splines, ML_HOST);
        dtheta_x_h = (real *)PMEMemoryFetch(PME_ID_DTHX, thread, size_splines, ML_HOST);
        dtheta_y_h = (real *)PMEMemoryFetch(PME_ID_DTHY, thread, size_splines, ML_HOST);
        dtheta_z_h = (real *)PMEMemoryFetch(PME_ID_DTHZ, thread, size_splines, ML_HOST);

        // indices
        idx_h = (ivec *)(int *)PMEMemoryFetch(PME_ID_IDXPTR, thread, DIM * size_indices, ML_HOST);

        // coefficients
        coefficients_h = (real *)PMEMemoryFetch(PME_ID_COEFFICIENT, thread, size_coefficients, ML_HOST);

        int iCompacted = 0;
        for (int ii = 0; ii < n; ii++)
        {
            int iOriginal = spline_ind[ii]; //should be just 1 : 1

            // coefficients
            real coefficient_i = scale * atc_coefficient[iOriginal]; //yupinov mutiply coefficients on device!

            if (coefficient_i != 0.0f)
            {
                coefficients_h[iCompacted] = coefficient_i;

                //indices
                /*
                int *idxptr = atc_idx[iOriginal];
                i0_h[iCompacted] = idxptr[XX];
                j0_h[iCompacted] = idxptr[YY];
                k0_h[iCompacted] = idxptr[ZZ];
                */
                memcpy(idx_h + iCompacted, atc_idx + iOriginal, sizeof(ivec));

                // thetas
                int iiorder = ii * order;
                int ooorder = iCompacted * order;
                for (int o = 0; o < order; ++o)
                {
                    theta_x_h[ooorder + o] = (*spline_theta)[XX][iiorder + o];
                    theta_y_h[ooorder + o] = (*spline_theta)[YY][iiorder + o];
                    theta_z_h[ooorder + o] = (*spline_theta)[ZZ][iiorder + o];
                    dtheta_x_h[ooorder + o] = (*spline_dtheta)[XX][iiorder + o];
                    dtheta_y_h[ooorder + o] = (*spline_dtheta)[YY][iiorder + o];
                    dtheta_z_h[ooorder + o] = (*spline_dtheta)[ZZ][iiorder + o];
                }

                // indices of uncompacted particles stored in a compacted array
                atc_i_compacted_h[iCompacted] = iOriginal;

                iCompacted++;
            }
        }
        // adjust sizes for device allocation
        n = iCompacted;
        size_coefficients = n * sizeof(real);
        size_splines = order * n * sizeof(int);
        size_indices = n * sizeof(int);
        forcesSize = DIM * n * sizeof(real);
    }
    else
    {
        for (int i = 0; i < n; i++)
        {
            // indices
            /*
            i0_h[i] = atc_idx[i][XX];
            j0_h[i] = atc_idx[i][YY];
            k0_h[i] = atc_idx[i][ZZ];
            */

            // coefficients
            atc_coefficient[i] *= scale;
        }

        // indices
        idx_h = atc_idx;
        // coefficients
        coefficients_h = atc_coefficient;
        // thetas
        theta_x_h = (*spline_theta)[XX];
        theta_y_h = (*spline_theta)[YY];
        theta_z_h = (*spline_theta)[ZZ];
        dtheta_x_h = (*spline_dtheta)[XX];
        dtheta_y_h = (*spline_dtheta)[YY];
        dtheta_z_h = (*spline_dtheta)[ZZ];
    }

    // thetas
    /*
    real *theta_x_d = (real *)PMEFetchAndCopy(PME_ID_THX, thread, theta_x_h, size_splines, s);
    real *theta_y_d = (real *)PMEFetchAndCopy(PME_ID_THY, thread, theta_y_h, size_splines, s);
    real *theta_z_d = (real *)PMEFetchAndCopy(PME_ID_THZ, thread, theta_z_h, size_splines, s);
    real *dtheta_x_d = (real *)PMEFetchAndCopy(PME_ID_DTHX, thread, dtheta_x_h, size_splines, s);
    real *dtheta_y_d = (real *)PMEFetchAndCopy(PME_ID_DTHY, thread, dtheta_y_h, size_splines, s);
    real *dtheta_z_d = (real *)PMEFetchAndCopy(PME_ID_DTHZ, thread, dtheta_z_h, size_splines, s);
    */
    real *theta_d = (real *)PMEMemoryFetch(PME_ID_THETA, thread, DIM * size_splines, ML_DEVICE);
    real *theta_x_d = theta_d + 0 * order * n;
    real *theta_y_d = theta_d + 1 * order * n;
    real *theta_z_d = theta_d + 2 * order * n;

    real *dtheta_d = (real *)PMEMemoryFetch(PME_ID_DTHETA, thread, DIM * size_splines, ML_DEVICE);
    real *dtheta_x_d = dtheta_d + 0 * order * n;
    real *dtheta_y_d = dtheta_d + 1 * order * n;
    real *dtheta_z_d = dtheta_d + 2 * order * n;

    // coefficients
    //real *coefficients_d = (real *)PMEMemoryFetch(PME_ID_COEFFICIENT, thread, size_coefficients, ML_DEVICE);
    //yupinov

    // indices
    int *idx_d = (int *)PMEMemoryFetch(PME_ID_IDXPTR, thread, DIM * size_indices, ML_DEVICE);

    if (!pme->gpu->keepGPUDataBetweenC2RAndGather) // compare with spread and compacting
    {
        cu_copy_H2D_async(theta_x_d, theta_x_h, size_splines, s);
        cu_copy_H2D_async(theta_y_d, theta_y_h, size_splines, s);
        cu_copy_H2D_async(theta_z_d, theta_z_h, size_splines, s);

        cu_copy_H2D_async(dtheta_x_d, dtheta_x_h, size_splines, s);
        cu_copy_H2D_async(dtheta_y_d, dtheta_y_h, size_splines, s);
        cu_copy_H2D_async(dtheta_z_d, dtheta_z_h, size_splines, s);

        //yupinov cu_copy_H2D_async(pme->gpu->coefficients, coefficients_h, size_coefficients, s);

        cu_copy_H2D_async(idx_d, idx_h, DIM * size_indices, s);
    }

    const float3 nXYZ = {(real)nx, (real)ny, (real)nz};


    const int blockSize = 4 * warp_size;
    const int particlesPerBlock = blockSize / order / order;
    dim3 nBlocks((n + blockSize - 1) / blockSize * order * order); //yupinov what does this mean?
    dim3 dimBlock(order, order, particlesPerBlock);

    pme_gpu_timing_start(pme, ewcsPME_GATHER);

    if (order == 4) //yupinov
        if (bOverwriteForces)
            pme_gather_kernel<4, blockSize / 4 / 4, TRUE> <<<nBlocks, dimBlock, 0, s>>>
              (pme->gpu->grid,
               n,
               nXYZ, pnx, pny, pnz,
               theta_d, dtheta_d,
               pme->gpu->forces, pme->gpu->coefficients,
#if !PME_EXTERN_CMEM
               pme->gpu->recipbox,
#endif
               idx_d);
        else
            pme_gather_kernel<4, blockSize / 4 / 4, FALSE> <<<nBlocks, dimBlock, 0, s>>>
              (pme->gpu->grid,
               n,
               nXYZ, pnx, pny, pnz,
               theta_d, dtheta_d,
               pme->gpu->forces, pme->gpu->coefficients,
#if !PME_EXTERN_CMEM
               pme->gpu->recipbox,
#endif
               idx_d);
    else
        gmx_fatal(FARGS, "gather: orders other than 4 untested!");
    CU_LAUNCH_ERR("pme_gather_kernel");

    pme_gpu_timing_stop(pme, ewcsPME_GATHER);

    cu_copy_D2H_async(atc_f_h, pme->gpu->forces, forcesSize, s);
    hipError_t stat = hipEventRecord(pme->gpu->syncForcesH2D, s);
    CU_RET_ERR(stat, "PME gather forces sync fail");
}

