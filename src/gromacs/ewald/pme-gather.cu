#include "hip/hip_runtime.h"
#include "pme.h"

#include "gromacs/utility/basedefinitions.h"
#include "gromacs/utility/real.h"
#include "gromacs/math/vectypes.h"
#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/gpu_utils/cuda_arch_utils.cuh"
#include <hip/hip_runtime.h>

#include "pme-timings.cuh"

#include "pme-internal.h"
#include "pme-cuda.cuh"

#include <assert.h>

#define SHARED_MEMORY_REDUCTION 1

//yupinov - texture memory?
template <
        const int order,
        const int particlesPerBlock
        >
__launch_bounds__(4 * warp_size, 16)
static __global__ void pme_gather_kernel
(const real * __restrict__ grid, const int n,
 const int nx, const int ny, const int nz, const int pnx, const int pny, const int pnz,
 const real rxx, const real ryx, const real ryy, const real rzx, const real rzy, const real rzz,
 const real * __restrict__ thx, const real * __restrict__ thy, const real * __restrict__ thz,
 const real * __restrict__ dthx, const real * __restrict__ dthy, const real * __restrict__ dthz,
 real * __restrict__ atc_f, const real * __restrict__ coefficient_v,
 //const int * __restrict__ i0, const int * __restrict__ j0, const int * __restrict__ k0,
 const int * __restrict__ idx
 )
{
    /* sum forces for local particles */

    // these are particle indices - in shared and global memory
    const int localIndex = threadIdx.z;
    const int globalIndex = blockIdx.x * blockDim.z + threadIdx.z;

    const int particleDataSize = order * order;
    const int blockSize = particlesPerBlock * particleDataSize; //1 line per thread
    // with odd orders something might break here?


    // spline Y/Z coordinates
    const int ithy = threadIdx.y;
    const int ithz = threadIdx.x;
    // these are spline contribution indices in shared memory
    const int splineIndex = threadIdx.y * blockDim.x + threadIdx.x;   // relative to the current particle
    const int lineIndex = (threadIdx.z * (blockDim.x * blockDim.y)) + splineIndex; // and to all the block's particles

    const int idxSize = DIM * particlesPerBlock;
    __shared__ int sharedIdx[idxSize];

    int blockId = blockIdx.x
                 + blockIdx.y * gridDim.x
                 + gridDim.x * gridDim.y * blockIdx.z;
    /*
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                  + (threadIdx.z * (blockDim.x * blockDim.y))
                  + (threadIdx.y * blockDim.x)
                  + threadIdx.x;
                  */
    int threadLocalId = (threadIdx.z * (blockDim.x * blockDim.y))
            + (threadIdx.y * blockDim.x)
            + threadIdx.x;
    if (threadLocalId < idxSize)
    {
        sharedIdx[threadLocalId] = idx[blockIdx.x * idxSize + threadLocalId];
    }//locality?
    __syncthreads();


#if SHARED_MEMORY_REDUCTION
    __shared__ real fx[blockSize];
    __shared__ real fy[blockSize];
    __shared__ real fz[blockSize];
    fx[lineIndex] = 0.0f;
    fy[lineIndex] = 0.0f;
    fz[lineIndex] = 0.0f;
#else
    real fx = 0.0f;
    real fy = 0.0f;
    real fz = 0.0f;
#endif

    __shared__ real coefficient[particlesPerBlock];

    if (globalIndex < n)
    {
        const int thetaOffset = globalIndex * order;

        for (int ithx = 0; (ithx < order); ithx++)
        {
            //const int index_x = (i0[globalIndex] + ithx) * pny * pnz;
            //const int index_x = (idx[globalIndex * DIM + XX] + ithx) * pny * pnz;
            const int index_x = (sharedIdx[localIndex * DIM + XX] + ithx) * pny * pnz;
            //if (blockId == 1)
            //    printf("%d %d\n", idx[globalIndex * DIM + XX], sharedIdx[localIndex * DIM + XX]);

            const real tx = thx[thetaOffset + ithx];
            const real dx = dthx[thetaOffset + ithx];

            //for (int ithy = 0; (ithy < order); ithy++)
            {
                //const int index_xy = index_x + (j0[globalIndex] + ithy) * pnz;
                //const int index_xy = index_x + (idx[globalIndex * DIM + YY] + ithy) * pnz;
                const int index_xy = index_x + (sharedIdx[localIndex * DIM + YY] + ithy) * pnz;
                const real ty = thy[thetaOffset + ithy];
                const real dy = dthy[thetaOffset + ithy];
                real fxy1 = 0.0f;
                real fz1 = 0.0f;

                /*for (int ithz = 0; (ithz < order); ithz++)    */
                /*   gridValue[particlesPerBlock * ithz + localIndex] = grid[index_xy + (k0[globalIndex] + ithz)];*/
                //for (int ithz = 0; (ithz < order); ithz++)
                {
                    /*printf(" INDEX %d %d %d\n", (i0[i] + ithx), (j0[i]+ithy), (k0[i]+ithz));*/
                    /*gridValue[localIndex] = grid[index_xy+(k0[globalIndex]+ithz)]; */
                    /*fxy1 += thz[thetaOffset + ithz] * gridValue[particlesPerBlock * ithz + localIndex];  */
                    /*fz1  += dthz[thetaOffset + ithz] * gridValue[particlesPerBlock * ithz + localIndex];    */
                    //const real gridValue = grid[index_xy + (k0[globalIndex] + ithz)];
                    //const real gridValue = grid[index_xy + (idx[globalIndex * DIM + ZZ] + ithz)];
                    const real gridValue = grid[index_xy + (sharedIdx[localIndex * DIM + ZZ] + ithz)];
                    fxy1 += thz[thetaOffset + ithz] * gridValue;
                    fz1  += dthz[thetaOffset + ithz] * gridValue;
                }
                //yupinov do a normal reduction here and below
#if SHARED_MEMORY_REDUCTION
                fx[lineIndex] += dx * ty * fxy1;
                fy[lineIndex] += tx * dy * fxy1;
                fz[lineIndex] += tx * ty * fz1;
#else
                fx += dx * ty * fxy1;
                fy += tx * dy * fxy1;
                fz += tx * ty * fz1;
#endif
                /*
                atomicAdd(fx + localIndex, dx * ty * fxy1);
                atomicAdd(fy + localIndex, tx * dy * fxy1);
                atomicAdd(fz + localIndex, tx * ty * fz1);
                */
                /*
                fx[localIndex] += dx * ty * fxy1;
                fy[localIndex] += tx * dy * fxy1;
                fz[localIndex] += tx * ty * fz1;
                */
            }
        }
    }
    __syncthreads(); // breaking globalIndex condition?

    // now particlesPerBlock have to sum order^2 contributions each

    // do a simple reduction in shared mem
    for (unsigned int s = 1; s < particleDataSize; s *= 2)//<<= 1)
    {
        if ((splineIndex % (2 * s) == 0) && (splineIndex + s < particleDataSize))
        {
            // order = 5 => splineIndex 24 (the last one) will get neighbour element without the second conditional
            // unroll for different orders?
            fx[lineIndex] += fx[lineIndex + s];
            fy[lineIndex] += fy[lineIndex + s];
            fz[lineIndex] += fz[lineIndex + s];
        }
        __syncthreads();
    }
    // skip shared memory,
    //  do a shuffle loop stopping before last step for order 4



    // below is the failed modified reduction #6
    // from http://developer.download.nvidia.com/compute/cuda/1.1-Beta/x86_website/projects/reduction/doc/reduction.pdf
    // (they have even better #7!)
    /*
    if (particleDataSize >= 512)
    {
        if (splineIndex < 256)
        {
            fx[lineIndex] += fx[lineIndex + 256];
            fy[lineIndex] += fy[lineIndex + 256];
            fz[lineIndex] += fz[lineIndex + 256];
        }
        __syncthreads();
    }
    if (particleDataSize >= 256)
    {
        if (splineIndex < 128)
        {
            fx[lineIndex] += fx[lineIndex + 128];
            fy[lineIndex] += fy[lineIndex + 128];
            fz[lineIndex] += fz[lineIndex + 128];
        }
        __syncthreads();
    }
    if (particleDataSize >= 128)
    {
        if (splineIndex < 64)
        {
            fx[lineIndex] += fx[lineIndex + 64];
            fy[lineIndex] += fy[lineIndex + 64];
            fz[lineIndex] += fz[lineIndex + 64];
        }
        __syncthreads();
    }
    //if (splineIndex < 32) //yupinov this is inside-warp-magic to not sync threads anymore - brings me mistakes?
    {
        if ((particleDataSize >= 64) && (splineIndex < 32))
        {
            fx[lineIndex] += fx[lineIndex + 32];
            fy[lineIndex] += fy[lineIndex + 32];
            fz[lineIndex] += fz[lineIndex + 32];
        }
        __syncthreads();
        if ((particleDataSize >= 32) && (splineIndex < 16))
        {
            fx[lineIndex] += fx[lineIndex + 16];
            fy[lineIndex] += fy[lineIndex + 16];
            fz[lineIndex] += fz[lineIndex + 16];
        }
        __syncthreads();
        if ((particleDataSize >= 16) && (splineIndex < 8))
        {
            fx[lineIndex] += fx[lineIndex +  8];
            fy[lineIndex] += fy[lineIndex +  8];
            fz[lineIndex] += fz[lineIndex +  8];
        }
        __syncthreads();
        if ((particleDataSize >=  8) && (splineIndex < 4))
        {
            fx[lineIndex] += fx[lineIndex +  4];
            fy[lineIndex] += fy[lineIndex +  4];
            fz[lineIndex] += fz[lineIndex +  4];
        }
        __syncthreads();
        if ((particleDataSize >=  4) && (splineIndex < 2))
        {
            fx[lineIndex] += fx[lineIndex +  2];
            fy[lineIndex] += fy[lineIndex +  2];
            fz[lineIndex] += fz[lineIndex +  2];
        }
        __syncthreads();
        if ((particleDataSize >=  2) && (splineIndex == 0))
        {
            fx[lineIndex] += fx[lineIndex +  1];
            fy[lineIndex] += fy[lineIndex +  1];
            fz[lineIndex] += fz[lineIndex +  1];
        }
        __syncthreads();
    }
    */

    if (splineIndex == 0) //yupinov stupid
    {
        coefficient[localIndex] = coefficient_v[globalIndex];
        const int idim = globalIndex * DIM;
        const int sumIndex = localIndex * particleDataSize;
        fx[sumIndex] *= (real) nx;
        fy[sumIndex] *= (real) ny;
        fz[sumIndex] *= (real) nz;
        atc_f[idim + XX] += -coefficient[localIndex] * ( fx[sumIndex] * rxx );
        atc_f[idim + YY] += -coefficient[localIndex] * ( fx[sumIndex] * ryx + fy[sumIndex] * ryy );
        atc_f[idim + ZZ] += -coefficient[localIndex] * ( fx[sumIndex] * rzx + fy[sumIndex] * rzy + fz[sumIndex] * rzz );
    }
}

template <
    const int order,
    const int stage
    >
__global__ void pme_unwrap_kernel
    (const int nx, const int ny, const int nz,
     const int pnx,const int pny, const int pnz,
     real * __restrict__ grid)
{
    //UNWRAP

    const int iz = blockIdx.x * blockDim.x + threadIdx.x;
    const int iy = blockIdx.y * blockDim.y + threadIdx.y;
    const int ix = blockIdx.z * blockDim.z + threadIdx.z;

    const int overlap = order - 1;

    int    ny_x;//, ix;

    //if (pme->nnodes_major == 1)
    if (stage & 4)
    {
        //ny_x = (pme->nnodes_minor == 1 ? ny : pme->pmegrid_ny);
        ny_x = ny;

        if (iz < nz)
        //for (ix = 0; ix < overlap; ix++)
        {
            //for (iy = 0; iy < ny_x; iy++)
            {
                //for (iz = 0; iz < nz; iz++)
                {
                    const int address = (ix * pny + iy) * pnz + iz;
                    const int offset_x = nx * pny * pnz;
                    grid[address + offset_x] = grid[address];
                }
            }
        }
    }

    //if (pme->nnodes_minor == 1)
    if (stage & 2)
    {
        if (iz < nz)
        //for (ix = 0; ix < pnx; ix++)
        {
            //int iy, iz;

            //for (iy = 0; iy < overlap; iy++)
            {
                //for (iz = 0; iz < nz; iz++)
                {
                    const int address = (ix * pny + iy) * pnz + iz;
                    const int offset_y = ny * pnz;
                    grid[address + offset_y] = grid[address];
                }
            }
        }
    }

    /* Copy periodic overlap in z */
    if (stage & 1)
    {
        //for (ix = 0; ix < pnx; ix++)
        if (iy < pny)
        {
            //int iy, iz;

            //for (iy = 0; iy < pny; iy++)
            {
                //for (iz = 0; iz < overlap; iz++)
                {
                    const int address = (ix * pny + iy) * pnz + iz;
                    const int offset_z = nz;
                    grid[address + offset_z] = grid[address];
                }
            }
        }
    }
}

void gather_f_bsplines_gpu
(real *grid, gmx_bool bClearF,
 const int order,
 int nx, int ny, int nz, int pnx, int pny, int pnz,
 real rxx, real ryx, real ryy, real rzx, real rzy, real rzz,
 int *spline_ind, int n,
 real *atc_coefficient, rvec *atc_f, ivec *atc_idx,
 splinevec *spline_theta, splinevec *spline_dtheta,
 real scale,
 gmx_pme_t *pme,
 int thread
 )
{
    hipStream_t s = pme->gpu->pmeStream;
    if (!n)
        return;

    const int ndatatot = pnx * pny * pnz;
    const int gridSize = ndatatot * sizeof(real);
    real *grid_d = PMEFetchRealArray(PME_ID_REAL_GRID, thread, gridSize, ML_DEVICE);
    if (!pme->gpu->keepGPUDataBetweenC2RAndGather)
        PMECopy(grid_d, grid, gridSize, ML_DEVICE, s);

    if (pme->bGPUSingle)
    {
        if (order == 4)
        {
            const int blockSize = 4 * warp_size; //yupinov thsi is everywhere! and arichitecture-specific
            const int overlap = order - 1; // all copied from pme-spread.cu
            int overlapLinesPerBlock = blockSize / overlap; //so there is unused padding in each block;

            dim3 blocks[] =
            {
                dim3(1, (pny + overlapLinesPerBlock - 1) / overlapLinesPerBlock, pnx),
                dim3((nz + overlapLinesPerBlock - 1) / overlapLinesPerBlock, 1, pnx),
                dim3((nz + overlapLinesPerBlock - 1) / overlapLinesPerBlock, ny, 1),
            };
            // low occupancy :(
            dim3 threads[] =
            {
                dim3(overlap, overlapLinesPerBlock, 1),
                dim3(overlapLinesPerBlock, overlap, 1),
                dim3(overlapLinesPerBlock, 1, overlap),
            };

            events_record_start(gpu_events_unwrap, s);

            pme_unwrap_kernel<4, 4> <<<blocks[2], threads[2], 0, s>>>(nx, ny, nz, pnx, pny, pnz, grid_d);
            pme_unwrap_kernel<4, 2> <<<blocks[1], threads[1], 0, s>>>(nx, ny, nz, pnx, pny, pnz, grid_d);
            pme_unwrap_kernel<4, 1> <<<blocks[0], threads[0], 0, s>>>(nx, ny, nz, pnx, pny, pnz, grid_d);

            CU_LAUNCH_ERR("pme_unwrap_kernel");

            events_record_stop(gpu_events_unwrap, s, ewcsPME_UNWRAP, 0);
        }
        else
            gmx_fatal(FARGS, "gather: orders other than 4 untested!");
    }

    int size_forces = DIM * n * sizeof(real); //yupinov!
    int size_indices = n * sizeof(int);
    int size_splines = order * n * sizeof(int);
    int size_coefficients = n * sizeof(real);

    real *atc_f_h = NULL;
    int *i0_h = NULL, *j0_h = NULL, *k0_h = NULL;
    real *coefficients_h = NULL;

    real *theta_x_h = NULL, *theta_y_h = NULL, *theta_z_h = NULL;
    real *dtheta_x_h = NULL, *dtheta_y_h = NULL, *dtheta_z_h = NULL;

    //indices - allocated here because maybe different sturcture?
    i0_h = PMEFetchIntegerArray(PME_ID_I0, thread, size_indices, ML_HOST);
    j0_h = PMEFetchIntegerArray(PME_ID_J0, thread, size_indices, ML_HOST);
    k0_h = PMEFetchIntegerArray(PME_ID_K0, thread, size_indices, ML_HOST);
    //yupinov broken!

    int *atc_i_compacted_h = NULL;


    // compact data
    if (PME_SKIP_ZEROES)
    {
        atc_i_compacted_h = PMEFetchIntegerArray(PME_ID_NONZERO_INDICES, thread, size_indices, ML_HOST);

        // fixed host allocation sizes - will only be smaller on GPU

        // forces
        atc_f_h = PMEFetchRealArray(PME_ID_F, thread, size_forces, ML_HOST);

        // thetas
        theta_x_h = PMEFetchRealArray(PME_ID_THX, thread, size_splines, ML_HOST);
        theta_y_h = PMEFetchRealArray(PME_ID_THY, thread, size_splines, ML_HOST);
        theta_z_h = PMEFetchRealArray(PME_ID_THZ, thread, size_splines, ML_HOST);
        dtheta_x_h = PMEFetchRealArray(PME_ID_DTHX, thread, size_splines, ML_HOST);
        dtheta_y_h = PMEFetchRealArray(PME_ID_DTHY, thread, size_splines, ML_HOST);
        dtheta_z_h = PMEFetchRealArray(PME_ID_DTHZ, thread, size_splines, ML_HOST);

        // coefficients
        coefficients_h = PMEFetchRealArray(PME_ID_COEFFICIENT, thread, size_coefficients, ML_HOST);

        int iCompacted = 0;
        for (int ii = 0; ii < n; ii++)
        {
            int iOriginal = spline_ind[ii]; //yupinov is there a point to this spline_ind? shoould be just 1 : 1

            assert(spline_ind[ii] == ii);

            // coefficients
            real coefficient_i = scale * atc_coefficient[iOriginal]; //yupinov mutiply coefficients on device!

            if (coefficient_i != 0.0f)
            {
                coefficients_h[iCompacted] = coefficient_i;

                //indices
                int *idxptr = atc_idx[iOriginal];
                i0_h[iCompacted] = idxptr[XX];
                j0_h[iCompacted] = idxptr[YY];
                k0_h[iCompacted] = idxptr[ZZ];

                //thetas
                int iiorder = ii * order;
                int ooorder = iCompacted * order;
                for (int o = 0; o < order; ++o)
                {
                    theta_x_h[ooorder + o] = (*spline_theta)[XX][iiorder + o];
                    theta_y_h[ooorder + o] = (*spline_theta)[YY][iiorder + o];
                    theta_z_h[ooorder + o] = (*spline_theta)[ZZ][iiorder + o];
                    dtheta_x_h[ooorder + o] = (*spline_dtheta)[XX][iiorder + o];
                    dtheta_y_h[ooorder + o] = (*spline_dtheta)[YY][iiorder + o];
                    dtheta_z_h[ooorder + o] = (*spline_dtheta)[ZZ][iiorder + o];
                }

                //forces
                if (!bClearF)
                {
                    atc_f_h[iCompacted * DIM + XX] = atc_f[iOriginal][XX];
                    atc_f_h[iCompacted * DIM + YY] = atc_f[iOriginal][YY];
                    atc_f_h[iCompacted * DIM + ZZ] = atc_f[iOriginal][ZZ];
                }

                // indices of uncompacted particles stored in a compacted array
                atc_i_compacted_h[iCompacted] = iOriginal;

                iCompacted++;
            }
        }
        // adjust sizes for device allocation
        n = iCompacted;
        size_coefficients = n * sizeof(real);
        size_splines = order * n * sizeof(int);
        size_indices = n * sizeof(int);
        size_forces = DIM * n * sizeof(real);
    }
    else
    {
        for (int i = 0; i < n; i++)
        {
            // indices
            i0_h[i] = atc_idx[i][XX]; //yupinov reorganize
            j0_h[i] = atc_idx[i][YY];
            k0_h[i] = atc_idx[i][ZZ];

            // coefficients
            atc_coefficient[i] *= scale;
        }

        // forces
        atc_f_h = (real *)atc_f;
        // coefficients
        coefficients_h = atc_coefficient;
        // thetas
        theta_x_h = (*spline_theta)[XX];
        theta_y_h = (*spline_theta)[YY];
        theta_z_h = (*spline_theta)[ZZ];
        dtheta_x_h = (*spline_dtheta)[XX];
        dtheta_y_h = (*spline_dtheta)[YY];
        dtheta_z_h = (*spline_dtheta)[ZZ];
    }

    // thetas
    /*
    real *theta_x_d = PMEFetchAndCopyRealArray(PME_ID_THX, thread, theta_x_h, size_splines, ML_DEVICE, s);
    real *theta_y_d = PMEFetchAndCopyRealArray(PME_ID_THY, thread, theta_y_h, size_splines, ML_DEVICE, s);
    real *theta_z_d = PMEFetchAndCopyRealArray(PME_ID_THZ, thread, theta_z_h, size_splines, ML_DEVICE, s);
    real *dtheta_x_d = PMEFetchAndCopyRealArray(PME_ID_DTHX, thread, dtheta_x_h, size_splines, ML_DEVICE, s);
    real *dtheta_y_d = PMEFetchAndCopyRealArray(PME_ID_DTHY, thread, dtheta_y_h, size_splines, ML_DEVICE, s);
    real *dtheta_z_d = PMEFetchAndCopyRealArray(PME_ID_DTHZ, thread, dtheta_z_h, size_splines, ML_DEVICE, s);
    */
    real *theta_d = PMEFetchRealArray(PME_ID_THETA, thread, DIM * size_splines, ML_DEVICE);
    real *theta_x_d = theta_d + 0 * order * n;
    real *theta_y_d = theta_d + 1 * order * n;
    real *theta_z_d = theta_d + 2 * order * n;

    real *dtheta_d = PMEFetchRealArray(PME_ID_DTHETA, thread, DIM * size_splines, ML_DEVICE);
    real *dtheta_x_d = dtheta_d + 0 * order * n;
    real *dtheta_y_d = dtheta_d + 1 * order * n;
    real *dtheta_z_d = dtheta_d + 2 * order * n;

    // coefficients
    real *coefficients_d = PMEFetchRealArray(PME_ID_COEFFICIENT, thread, size_coefficients, ML_DEVICE);

    int *idx_d = PMEFetchIntegerArray(PME_ID_IDXPTR, thread, DIM * size_indices, ML_DEVICE);

    if (!pme->gpu->keepGPUDataBetweenC2RAndGather) // compare with spread and compacting
    {
        PMECopy(theta_x_d, theta_x_h, size_splines, ML_DEVICE, s);
        PMECopy(theta_y_d, theta_y_h, size_splines, ML_DEVICE, s);
        PMECopy(theta_z_d, theta_z_h, size_splines, ML_DEVICE, s);

        PMECopy(dtheta_x_d, dtheta_x_h, size_splines, ML_DEVICE, s);
        PMECopy(dtheta_y_d, dtheta_y_h, size_splines, ML_DEVICE, s);
        PMECopy(dtheta_z_d, dtheta_z_h, size_splines, ML_DEVICE, s);

        PMECopy(coefficients_d, coefficients_h, size_coefficients, ML_DEVICE, s);

        PMECopy(idx_d, atc_idx, DIM * size_indices, ML_DEVICE, s);
    }

    //forces
    real *atc_f_d = PMEFetchRealArray(PME_ID_F, thread, size_forces, ML_DEVICE);
    if (bClearF)
    {
        hipError_t stat = hipMemsetAsync(atc_f_d, 0, size_forces, s);
        CU_RET_ERR(stat, "hipMemsetAsync gather forces error");
    }
    else
        PMECopy(atc_f_d, atc_f_h, size_forces, ML_DEVICE, s);

    //indices

    //yupinov
    /*
    int *i0_d = PMEFetchAndCopyIntegerArray(PME_ID_I0, thread, i0_h, size_indices, ML_DEVICE, s);
    int *j0_d = PMEFetchAndCopyIntegerArray(PME_ID_J0, thread, j0_h, size_indices, ML_DEVICE, s);
    int *k0_d = PMEFetchAndCopyIntegerArray(PME_ID_K0, thread, k0_h, size_indices, ML_DEVICE, s);
    */


    const int blockSize = 4 * warp_size;
    const int particlesPerBlock = blockSize / order / order;
    dim3 nBlocks((n + blockSize - 1) / blockSize * order * order, 1, 1); //yupinov what does this mean?
    dim3 dimBlock(order, order, particlesPerBlock);

    events_record_start(gpu_events_gather, s);

    if (order == 4) //yupinov
        pme_gather_kernel<4, blockSize / 4 / 4> <<<nBlocks, dimBlock, 0, s>>>
          (grid_d,
           n,
           nx, ny, nz, pnx, pny, pnz,
           rxx, ryx, ryy, rzx, rzy, rzz,
           theta_x_d, theta_y_d, theta_z_d,
           dtheta_x_d, dtheta_y_d, dtheta_z_d,
           atc_f_d, coefficients_d,
           //i0_d, j0_d, k0_d,
           idx_d);
    else
        gmx_fatal(FARGS, "gather: orders other than 4 untested!");
    CU_LAUNCH_ERR("pme_gather_kernel");

    events_record_stop(gpu_events_gather, s, ewcsPME_GATHER, 0);

    PMECopy(atc_f_h, atc_f_d, size_forces, ML_HOST, s);

    if (PME_SKIP_ZEROES)
        for (int ii = 0; ii < n; ii++)  // iterating over compacted particles
        {
            int i = atc_i_compacted_h[ii]; //index of uncompacted particle
            atc_f[i][XX] = atc_f_h[ii * DIM + XX];
            atc_f[i][YY] = atc_f_h[ii * DIM + YY];
            atc_f[i][ZZ] = atc_f_h[ii * DIM + ZZ];
        }
}
